#include "hip/hip_runtime.h"
#include "diagmask.cuh"

static __global__ void diag_mask_inf_f32(const float* x, float* dst, const int ncols, const int rows_per_channel, const int n_past) {
	const int col = blockDim.y * blockIdx.y + threadIdx.y;
	const int row = blockDim.x * blockIdx.x + threadIdx.x;

	if (col >= ncols) {
		return;
	}

	const int i = row * ncols + col;
	//dst[i] = col > (n_past + row % rows_per_channel) ? -INFINITY : x[i];
	//dst[i] = x[i] - (col > n_past + row % rows_per_channel) * INT_MAX; // equivalent within rounding error but slightly faster on GPU
	dst[i] = x[i] - (col > n_past + row % rows_per_channel) * FLT_MAX;
}

static void diag_mask_inf_f32_cuda(const float* x, float* dst, const int ncols_x, const int nrows_x, const int rows_per_channel, const int n_past, hipStream_t stream) {
	const dim3 block_dims(1, CUDA_DIAG_MASK_INF_BLOCK_SIZE, 1);
	const int block_num_x = (ncols_x + CUDA_DIAG_MASK_INF_BLOCK_SIZE - 1) / CUDA_DIAG_MASK_INF_BLOCK_SIZE;
	const dim3 block_nums(nrows_x, block_num_x, 1);
	diag_mask_inf_f32<<<block_nums, block_dims, 0, stream>>>(x, dst, ncols_x, rows_per_channel, n_past);
}

void oiml_cuda_op_diag_mask_inf(oiml_backend_cuda_context& ctx, oiml_tensor* dst) {
	const oiml_tensor* src0 = dst->src[0];
	const float* src0_d		= ( const float* )src0->data;
	float* dst_d			= ( float* )dst->data;
	hipStream_t stream		= ctx.stream();

	OIML_ASSERT(src0->type == oiml::oiml_representation_types::float_32);
	OIML_ASSERT(dst->type == oiml::oiml_representation_types::float_32);

	const int64_t ne00 = src0->ne[0];
	const int64_t ne01 = src0->ne[1];
	const int nrows0   = oiml_nrows(src0);

	const int n_past = (( int32_t* )dst->op_params)[0];

	diag_mask_inf_f32_cuda(src0_d, dst_d, ne00, nrows0, ne01, n_past, stream);
}
