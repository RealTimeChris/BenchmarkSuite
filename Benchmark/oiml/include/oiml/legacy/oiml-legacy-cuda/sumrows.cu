#include "hip/hip_runtime.h"
#include "sumrows.cuh"

static __global__ void k_sum_rows_f32(const float* x, float* dst, const int ncols) {
	const int row = blockIdx.x;
	const int col = threadIdx.x;

	float sum = 0.0f;
	for (int i = col; i < ncols; i += blockDim.x) {
		sum += x[row * ncols + i];
	}

	sum = warp_reduce_sum(sum);

	if (col == 0) {
		dst[row] = sum;
	}
}

void sum_rows_f32_cuda(const float* x, float* dst, const int ncols, const int nrows, hipStream_t stream) {
	const dim3 block_dims(WARP_SIZE, 1, 1);
	const dim3 block_nums(nrows, 1, 1);
	k_sum_rows_f32<<<block_nums, block_dims, 0, stream>>>(x, dst, ncols);
}

void oiml_cuda_op_sum_rows(oiml_backend_cuda_context& ctx, oiml_tensor* dst) {
	const oiml_tensor* src0 = dst->src[0];
	const float* src0_d		= ( const float* )src0->data;
	float* dst_d			= ( float* )dst->data;
	hipStream_t stream		= ctx.stream();

	OIML_ASSERT(src0->type == oiml::oiml_representation_types::float_32);
	OIML_ASSERT(dst->type == oiml::oiml_representation_types::float_32);
	OIML_ASSERT(oiml_is_contiguous(src0));

	const int64_t ncols = src0->ne[0];
	const int64_t nrows = oiml_nrows(src0);

	sum_rows_f32_cuda(src0_d, dst_d, ncols, nrows, stream);
}
