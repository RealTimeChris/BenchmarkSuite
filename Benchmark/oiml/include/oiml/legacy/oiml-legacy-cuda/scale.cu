#include "hip/hip_runtime.h"
#include "scale.cuh"

static __global__ void scale_f32(const float* x, float* dst, const float scale, const int k) {
	const int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i >= k) {
		return;
	}

	dst[i] = scale * x[i];
}

static void scale_f32_cuda(const float* x, float* dst, const float scale, const int k, hipStream_t stream) {
	const int num_blocks = (k + CUDA_SCALE_BLOCK_SIZE - 1) / CUDA_SCALE_BLOCK_SIZE;
	scale_f32<<<num_blocks, CUDA_SCALE_BLOCK_SIZE, 0, stream>>>(x, dst, scale, k);
}

void oiml_cuda_op_scale(oiml_backend_cuda_context& ctx, oiml_tensor* dst) {
	const oiml_tensor* src0 = dst->src[0];
	const float* src0_d		= ( const float* )src0->data;
	float* dst_d			= ( float* )dst->data;
	hipStream_t stream		= ctx.stream();

	OIML_ASSERT(src0->type == oiml::oiml_representation_types::float_32);
	OIML_ASSERT(dst->type == oiml::oiml_representation_types::float_32);

	float scale;
	memcpy(&scale, dst->op_params, sizeof(float));

	scale_f32_cuda(src0_d, dst_d, scale, oiml_nelements(src0), stream);
}
