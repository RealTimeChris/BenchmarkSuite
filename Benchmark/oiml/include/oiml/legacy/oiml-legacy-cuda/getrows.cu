#include "hip/hip_runtime.h"
#include "getrows.cuh"
#include "dequantize.cuh"

template<int qk, int qr, dequantize_kernel_t dequantize_kernel, typename dst_t> static __global__ void k_get_rows(const void* __restrict__ src0, const int32_t* __restrict__ src1,
	dst_t* __restrict__ dst, const int64_t ne00, /*const int64_t ne01, const int64_t ne02, const int64_t ne03,*/
	/*const int64_t ne10, const int64_t ne11,*/ const int64_t ne12, /*const int64_t ne13,*/
	/*const size_t s0,*/ const size_t s1, const size_t s2, const size_t s3,
	/*const size_t nb00,*/ const size_t nb01, const size_t nb02, const size_t nb03, const size_t s10, const size_t s11, const size_t s12 /*, const size_t s13*/) {
	const int i00 = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
	const int i10 = blockDim.y * blockIdx.y + threadIdx.y;
	const int i11 = (blockIdx.z * blockDim.z + threadIdx.z) / ne12;
	const int i12 = (blockIdx.z * blockDim.z + threadIdx.z) % ne12;

	if (i00 >= ne00) {
		return;
	}

	const int i01 = src1[i10 * s10 + i11 * s11 + i12 * s12];

	dst_t* dst_row		 = dst + i10 * s1 + i11 * s2 + i12 * s3;
	const void* src0_row = ( const char* )src0 + i01 * nb01 + i11 * nb02 + i12 * nb03;

	const int ib	   = i00 / qk;// block index
	const int iqs	   = (i00 % qk) / qr;// quant index
	const int iybs	   = i00 - i00 % qk;// dst block start index
	const int y_offset = qr == 1 ? 1 : qk / 2;

	// dequantize
	dfloat2 v;
	dequantize_kernel(src0_row, ib, iqs, v);

	dst_row[iybs + iqs + 0]		   = v.x;
	dst_row[iybs + iqs + y_offset] = v.y;
}

template<typename src0_t, typename dst_t> static __global__ void k_get_rows_float(const src0_t* __restrict__ src0, const int32_t* __restrict__ src1, dst_t* __restrict__ dst,
	const int64_t ne00, /*const int64_t ne01, const int64_t ne02, const int64_t ne03,*/
	/*const int64_t ne10, const int64_t ne11,*/ const int64_t ne12, /*const int64_t ne13,*/
	/*const size_t s0,*/ const size_t s1, const size_t s2, const size_t s3,
	/*const size_t nb00,*/ const size_t nb01, const size_t nb02, const size_t nb03, const size_t s10, const size_t s11, const size_t s12 /*, const size_t s13*/) {
	const int i00 = blockIdx.x * blockDim.x + threadIdx.x;
	const int i10 = blockDim.y * blockIdx.y + threadIdx.y;
	const int i11 = (blockIdx.z * blockDim.z + threadIdx.z) / ne12;
	const int i12 = (blockIdx.z * blockDim.z + threadIdx.z) % ne12;

	if (i00 >= ne00) {
		return;
	}

	const int i01 = src1[i10 * s10 + i11 * s11 + i12 * s12];

	dst_t* dst_row		   = dst + i10 * s1 + i11 * s2 + i12 * s3;
	const src0_t* src0_row = ( const src0_t* )(( const char* )src0 + i01 * nb01 + i11 * nb02 + i12 * nb03);

	dst_row[i00] = src0_row[i00];
}

template<typename grad_t, typename dst_t> static __global__ void k_get_rows_back_float(const grad_t* __restrict__ grad, const int32_t* __restrict__ rows, dst_t* __restrict__ dst,
	const int64_t ncols, const int64_t nrows_grad) {
	const int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (col >= ncols) {
		return;
	}

	const int dst_row = blockIdx.y * blockDim.y + threadIdx.y;

	float sum = 0.0f;

	for (int64_t i = 0; i < nrows_grad; ++i) {
		if (rows[i] != dst_row) {
			continue;
		}
		sum += grad[i * ncols + col];
	}

	dst[dst_row * ncols + col] = sum;
}

template<int qk, int qr, dequantize_kernel_t dq>
static void get_rows_cuda(const oiml_tensor* src0, const oiml_tensor* src1, oiml_tensor* dst, const void* src0_dd, const int32_t* src1_dd, float* dst_dd, hipStream_t stream) {
	OIML_TENSOR_BINARY_OP_LOCALS

	const dim3 block_dims(CUDA_GET_ROWS_BLOCK_SIZE, 1, 1);
	const int block_num_x = (ne00 + 2 * CUDA_GET_ROWS_BLOCK_SIZE - 1) / (2 * CUDA_GET_ROWS_BLOCK_SIZE);
	const dim3 block_nums(block_num_x, ne10, ne11 * ne12);

	// strides in elements
	//const size_t s0 = nb0 / oiml_element_size(dst);
	const size_t s1 = nb1 / oiml_element_size(dst);
	const size_t s2 = nb2 / oiml_element_size(dst);
	const size_t s3 = nb3 / oiml_element_size(dst);

	const size_t s10 = nb10 / oiml_element_size(src1);
	const size_t s11 = nb11 / oiml_element_size(src1);
	const size_t s12 = nb12 / oiml_element_size(src1);
	//const size_t s13 = nb13 / oiml_element_size(src1);

	OIML_ASSERT(ne00 % 2 == 0);

	k_get_rows<qk, qr, dq><<<block_nums, block_dims, 0, stream>>>(src0_dd, src1_dd, dst_dd, ne00, /*ne01, ne02, ne03,*/
		/*ne10, ne11,*/ ne12, /*ne13,*/
		/* s0,*/ s1, s2, s3,
		/* nb00,*/ nb01, nb02, nb03, s10, s11, s12 /*, s13*/);

	OIML_UNUSED(dst);
}

template<typename src0_t> static void get_rows_cuda_float(const oiml_tensor* src0, const oiml_tensor* src1, oiml_tensor* dst, const src0_t* src0_dd, const int32_t* src1_dd,
	float* dst_dd, hipStream_t stream) {
	OIML_TENSOR_BINARY_OP_LOCALS

	OIML_ASSERT(ne13 == 1);

	const dim3 block_dims(CUDA_GET_ROWS_BLOCK_SIZE, 1, 1);
	const int block_num_x = (ne00 + CUDA_GET_ROWS_BLOCK_SIZE - 1) / CUDA_GET_ROWS_BLOCK_SIZE;
	const dim3 block_nums(block_num_x, ne10, ne11 * ne12);

	// strides in elements
	//const size_t s0 = nb0 / oiml_element_size(dst);
	const size_t s1 = nb1 / oiml_element_size(dst);
	const size_t s2 = nb2 / oiml_element_size(dst);
	const size_t s3 = nb3 / oiml_element_size(dst);

	const size_t s10 = nb10 / oiml_element_size(src1);
	const size_t s11 = nb11 / oiml_element_size(src1);
	const size_t s12 = nb12 / oiml_element_size(src1);
	//const size_t s13 = nb13 / oiml_element_size(src1);

	k_get_rows_float<<<block_nums, block_dims, 0, stream>>>(src0_dd, src1_dd, dst_dd, ne00, /*ne01, ne02, ne03,*/
		/*ne10, ne11,*/ ne12, /*ne13,*/
		/* s0,*/ s1, s2, s3,
		/* nb00,*/ nb01, nb02, nb03, s10, s11, s12 /*, s13*/);

	OIML_UNUSED(dst);
}

void oiml_cuda_op_get_rows(oiml_backend_cuda_context& ctx, oiml_tensor* dst) {
	const oiml_tensor* src0 = dst->src[0];
	const oiml_tensor* src1 = dst->src[1];

	const void* src0_d	  = ( const void* )src0->data;
	const int32_t* src1_d = ( const int32_t* )src1->data;
	float* dst_d		  = ( float* )dst->data;

	hipStream_t stream = ctx.stream();

	OIML_ASSERT(src1->type == oiml::oiml_representation_types::int_32);
	OIML_ASSERT(dst->type == oiml::oiml_representation_types::float_32);

	OIML_ASSERT(src0->nb[0] == oiml_type_size(src0->type));
	OIML_ASSERT(src1->nb[0] == oiml_type_size(src1->type));
	OIML_ASSERT(dst->nb[0] == oiml_type_size(dst->type));

	switch (src0->type) {
		case oiml::oiml_representation_types::float_16:
			get_rows_cuda_float(src0, src1, dst, ( const half* )src0_d, src1_d, dst_d, stream);
			break;
		case oiml::oiml_representation_types::float_32:
			get_rows_cuda_float(src0, src1, dst, ( const float* )src0_d, src1_d, dst_d, stream);
			break;
		case oiml::oiml_representation_types::q8_0:
			get_rows_cuda<oiml::Q_SIZE, oiml::QR8_0, dequantize_q8_0>(src0, src1, dst, src0_d, src1_d, dst_d, stream);
			break;
		default:
			// TODO: k-quants
			OIML_ABORT("%s: unsupported type: %s\n", __func__, oiml_type_name(src0->type));
			break;
	}
}

void oiml_cuda_op_get_rows_back(oiml_backend_cuda_context& ctx, oiml_tensor* dst) {
	const oiml_tensor* src0 = dst->src[0];// gradients of forward pass output
	const oiml_tensor* src1 = dst->src[1];// src1 in forward pass

	OIML_TENSOR_BINARY_OP_LOCALS

	const float* src0_d	  = ( const float* )src0->data;
	const int32_t* src1_d = ( const int32_t* )src1->data;
	float* dst_d		  = ( float* )dst->data;

	hipStream_t stream = ctx.stream();

	OIML_ASSERT(src0->type == oiml::oiml_representation_types::float_32);
	OIML_ASSERT(src1->type == oiml::oiml_representation_types::int_32);
	OIML_ASSERT(dst->type == oiml::oiml_representation_types::float_32);

	OIML_ASSERT(oiml_is_contiguous(src0));
	OIML_ASSERT(oiml_is_contiguous(src1));
	OIML_ASSERT(oiml_is_contiguous(dst));

	OIML_ASSERT(ne02 * ne03 == 1);
	OIML_ASSERT(ne12 * ne13 == 1);
	OIML_ASSERT(ne2 * ne3 == 1);

	const dim3 block_dims(CUDA_GET_ROWS_BACK_BLOCK_SIZE, 1, 1);
	const int block_num_x = (ne00 + CUDA_GET_ROWS_BACK_BLOCK_SIZE - 1) / CUDA_GET_ROWS_BACK_BLOCK_SIZE;
	const dim3 block_nums(block_num_x, ne1, 1);

	k_get_rows_back_float<<<block_nums, block_dims, 0, stream>>>(src0_d, src1_d, dst_d, ne00, ne10);
}
