#include "hip/hip_runtime.h"
#include "argsort.cuh"

template<typename T> static inline __device__ void oiml_cuda_swap(T& a, T& b) {
	T tmp = a;
	a	  = b;
	b	  = tmp;
}

template<oiml_sort_order order> static __global__ void k_argsort_f32_i32(const float* x, int* dst, const int ncols, int ncols_pad) {
	// bitonic sort
	int col = threadIdx.x;
	int row = blockIdx.y;

	if (col >= ncols_pad) {
		return;
	}

	const float* x_row = x + row * ncols;
	extern __shared__ int dst_row[];

	// initialize indices
	dst_row[col] = col;

	__syncthreads();

	for (int k = 2; k <= ncols_pad; k *= 2) {
		for (int j = k / 2; j > 0; j /= 2) {
			int ixj = col ^ j;
			if (ixj > col) {
				if ((col & k) == 0) {
					if (dst_row[col] >= ncols ||
						(dst_row[ixj] < ncols && (order == OIML_SORT_ORDER_ASC ? x_row[dst_row[col]] > x_row[dst_row[ixj]] : x_row[dst_row[col]] < x_row[dst_row[ixj]]))) {
						oiml_cuda_swap(dst_row[col], dst_row[ixj]);
					}
				} else {
					if (dst_row[ixj] >= ncols ||
						(dst_row[col] < ncols && (order == OIML_SORT_ORDER_ASC ? x_row[dst_row[col]] < x_row[dst_row[ixj]] : x_row[dst_row[col]] > x_row[dst_row[ixj]]))) {
						oiml_cuda_swap(dst_row[col], dst_row[ixj]);
					}
				}
			}
			__syncthreads();
		}
	}

	// copy the result to dst without the padding
	if (col < ncols) {
		dst[row * ncols + col] = dst_row[col];
	}
}

static int next_power_of_2(int x) {
	int n = 1;
	while (n < x) {
		n *= 2;
	}
	return n;
}

static void argsort_f32_i32_cuda(const float* x, int* dst, const int ncols, const int nrows, oiml_sort_order order, hipStream_t stream) {
	// bitonic sort requires ncols to be power of 2
	const int ncols_pad = next_power_of_2(ncols);

	const dim3 block_dims(ncols_pad, 1, 1);
	const dim3 block_nums(1, nrows, 1);
	const size_t shared_mem = ncols_pad * sizeof(int);

	// FIXME: this limit could be raised by ~2-4x on Ampere or newer
	OIML_ASSERT(shared_mem <= oiml_cuda_info().devices[oiml_cuda_get_device()].smpb);

	if (order == OIML_SORT_ORDER_ASC) {
		k_argsort_f32_i32<OIML_SORT_ORDER_ASC><<<block_nums, block_dims, shared_mem, stream>>>(x, dst, ncols, ncols_pad);
	} else if (order == OIML_SORT_ORDER_DESC) {
		k_argsort_f32_i32<OIML_SORT_ORDER_DESC><<<block_nums, block_dims, shared_mem, stream>>>(x, dst, ncols, ncols_pad);
	} else {
		OIML_ABORT("fatal error");
	}
}

void oiml_cuda_op_argsort(oiml_backend_cuda_context& ctx, oiml_tensor* dst) {
	const oiml_tensor* src0 = dst->src[0];
	const float* src0_d		= ( const float* )src0->data;
	float* dst_d			= ( float* )dst->data;
	hipStream_t stream		= ctx.stream();

	OIML_ASSERT(src0->type == oiml::oiml_representation_types::float_32);
	OIML_ASSERT(dst->type == oiml::oiml_representation_types::int_32);
	OIML_ASSERT(oiml_is_contiguous(src0));

	const int64_t ncols = src0->ne[0];
	const int64_t nrows = oiml_nrows(src0);

	enum oiml_sort_order order = ( enum oiml_sort_order )dst->op_params[0];

	argsort_f32_i32_cuda(src0_d, ( int* )dst_d, ncols, nrows, order, stream);
}
