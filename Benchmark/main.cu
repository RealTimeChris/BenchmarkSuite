#include "hip/hip_runtime.h"
#include <BnchSwt/BenchmarkSuite.hpp>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

static constexpr uint64_t total_iterations{ 2 };
static constexpr uint64_t measured_iterations{ 1 };

template<auto multiple, typename value_01_type = decltype(multiple)> BNCH_SWT_INLINE constexpr value_01_type round_up_to_multiple(value_01_type value) noexcept {
	if constexpr ((multiple & (multiple - 1)) == 0) {
		constexpr value_01_type mulSub1{ multiple - 1 };
		constexpr value_01_type notMulSub1{ static_cast<value_01_type>(~mulSub1) };
		return (value + (mulSub1)) & notMulSub1;
	} else {
		const value_01_type remainder = value % multiple;
		return remainder == 0 ? value : value + (multiple - remainder);
	}
}

struct cuda_buffer {
	using value_type													= uint8_t;
	using pointer														= value_type*;
	using size_type														= uint64_t;
	BNCH_SWT_INLINE cuda_buffer() noexcept								= default;
	BNCH_SWT_INLINE cuda_buffer& operator=(const cuda_buffer&) noexcept = delete;
	BNCH_SWT_INLINE cuda_buffer(const cuda_buffer&) noexcept			= delete;

	BNCH_SWT_INLINE cuda_buffer& operator=(cuda_buffer&& other) noexcept {
		if (this != &other) {
			std::swap(data_val, other.data_val);
			std::swap(size_val, other.size_val);
		}
		return *this;
	}

	BNCH_SWT_INLINE cuda_buffer(cuda_buffer&& other) noexcept {
		*this = std::move(other);
	}

	BNCH_SWT_INLINE void init(uint64_t size) noexcept {
		if (data_val) {
			clear();
		}

		hipError_t result = hipMalloc(&data_val, size);
		if (result != hipSuccess) {
			data_val = nullptr;
			throw std::runtime_error{ "cuda_buffer - failed to allocate GPU memory" };
		}

		size_val = size;
	}

	BNCH_SWT_INLINE void deinit() noexcept {
		clear();
	}

	BNCH_SWT_INLINE uint64_t size() noexcept {
		return size_val;
	}

	BNCH_SWT_INLINE void* data() noexcept {
		return data_val;
	}

	BNCH_SWT_INLINE void* claim_memory(uint64_t offset_to_claim) noexcept {
		uint64_t aligned_amount = round_up_to_multiple<64>(offset_to_claim);
		if (aligned_amount > size_val) {
			throw std::runtime_error{ "cuda_buffer - not enough memory allocated!" };
		}
		pointer return_value = data_val + aligned_amount;
		return return_value;
	}

	BNCH_SWT_INLINE ~cuda_buffer() noexcept {
		clear();
	}

  protected:
	value_type* data_val{};
	uint64_t size_val{};

	BNCH_SWT_INLINE void clear() noexcept {
		if (data_val) {
			hipError_t result = hipFree(data_val);
			data_val		   = nullptr;
			size_val		   = 0;
		}
	}
};

using q8_quant = int8_t;

struct block_q8_0 {
	q8_quant quants[32]{};
	uint16_t scale{};
};

inline static uint16_t fp32_to_fp16(float f) {
	return static_cast<uint16_t>(_mm_extract_epi16(_mm_cvtps_ph(_mm_set_ss(f), _MM_FROUND_TO_NEAREST_INT), 0));
}

template<std::size_t count> inline std::vector<block_q8_0> generate_blocks() {
	std::vector<block_q8_0> result;
	result.reserve(count);

	for (std::size_t i = 0; i < count; ++i) {
		block_q8_0 block{};

		for (auto& q: block.quants) {
			q = static_cast<int8_t>((bnch_swt::random_generator::generateValue<uint8_t>() % 13) - 6);
		}

		float scale_float = (bnch_swt::random_generator::generateValue<float>() / std::numeric_limits<float>::max()) * 0.1f;

		block.scale = static_cast<uint16_t>(fp32_to_fp16(scale_float));

		result.emplace_back(block);
	}
	return result;
}

template<uint64_t iteration_count, std::size_t count> inline std::vector<std::vector<block_q8_0>> generate_blocks_final() {
	std::vector<std::vector<block_q8_0>> return_values{};
	for (uint64_t x = 0; x < iteration_count; ++x) {
		return_values.emplace_back(generate_blocks<count>());
	}
	return return_values;
}

template<std::size_t count> inline std::vector<float> generate_floats() {
	std::vector<float> result;
	result.reserve(count);

	for (std::size_t i = 0; i < count; ++i) {
		float raw_val	 = bnch_swt::random_generator::generateValue<float>();
		float normalized = (raw_val / std::numeric_limits<float>::max());
		result.emplace_back(normalized);
	}
	return result;
}

template<uint64_t iteration_count, std::size_t count> inline std::vector<std::vector<float>> generate_floats_final() {
	std::vector<std::vector<float>> return_values{};
	for (uint64_t x = 0; x < iteration_count; ++x) {
		return_values.emplace_back(generate_floats<count>());
	}
	return return_values;
}

BNCH_SWT_INLINE static constexpr float fp32_from_bits(uint32_t w) noexcept {
	return std::bit_cast<float>(w);
}

BNCH_SWT_INLINE static constexpr uint32_t fp32_to_bits(float f) noexcept {
	return std::bit_cast<uint32_t>(f);
}

BNCH_SWT_INLINE static float compute_fp16_to_fp32(half h) noexcept {
	const uint32_t w	 = static_cast<uint32_t>(h) << 16;
	const uint32_t sign	 = w & 0x80000000u;
	const uint32_t two_w = w + w;

	constexpr uint32_t exp_offset = 0xE0u << 23;
	constexpr float exp_scale	  = fp32_from_bits(0x7800000u);
	const float normalized_value  = fp32_from_bits((two_w >> 4) + exp_offset) * exp_scale;

	constexpr uint32_t magic_mask  = 126u << 23;
	constexpr float magic_bias	   = 0.5f;
	const float denormalized_value = fp32_from_bits((two_w >> 17) | magic_mask) - magic_bias;

	constexpr uint32_t denormalized_cutoff = 1u << 27;
	const uint32_t result				   = sign | (two_w < denormalized_cutoff ? fp32_to_bits(denormalized_value) : fp32_to_bits(normalized_value));
	return fp32_from_bits(result);
}

alignas(64) static float* __restrict fp16_to_fp32_array{ []() {
	alignas(64) static std::array<float, (1 << 16)> return_values_new{};
	for (uint64_t i = 0; i < (1 << 16); ++i) {
		return_values_new[i] = float{ compute_fp16_to_fp32(static_cast<uint16_t>(i)) };
	}
	return return_values_new.data();
}() };

BNCH_SWT_INLINE static float fp16_to_fp32(uint16_t f) {
	return fp16_to_fp32_array[f];
}

template<uint64_t matA_dim_00, uint64_t matA_dim_01, uint64_t block_size> struct reference_mul_mat {
	BNCH_SWT_INLINE static uint64_t impl(cuda_buffer& buffer, uint64_t& current_index, std::vector<std::vector<float>>& floats, std::vector<std::vector<block_q8_0>>& blocks,
		std::vector<std::vector<float>>& outputs, uint64_t matB_dim_01) {
		const auto& current_blocks = blocks[current_index];
		const auto& current_floats = floats[current_index];
		auto& current_outputs	   = outputs[current_index];

		for (uint64_t row = 0; row < matA_dim_00; ++row) {
			for (uint64_t col = 0; col < matB_dim_01; ++col) {
				float sum = 0.0f;

				for (uint64_t k = 0; k < matA_dim_01; ++k) {
					const uint64_t block_idx	 = (row * matA_dim_01 + k) / block_size;
					const uint64_t elem_in_block = (row * matA_dim_01 + k) % block_size;

					const auto& block  = current_blocks[block_idx];
					const float scale  = __half2float(*reinterpret_cast<const __half*>(&block.scale));
					const float a_elem = scale * static_cast<float>(block.quants[elem_in_block]);

					const float b_elem = current_floats[k * matB_dim_01 + col];

					sum += a_elem * b_elem;
				}

				current_outputs[row * matB_dim_01 + col] = sum;
			}
		}

		return current_outputs.size() * sizeof(float);
	}
};

template<uint64_t matA_dim_00, uint64_t matA_dim_01, uint64_t block_size> struct cuda_mul_mat_01_prep {
	BNCH_SWT_INLINE static uint64_t impl(cuda_buffer& buffer, uint64_t& current_index, std::vector<std::vector<float>>& floats, std::vector<std::vector<block_q8_0>>& blocks,
		std::vector<std::vector<float>>& outputs, uint64_t matB_dim_01) {
		constexpr uint64_t total_blocks_A = ((matA_dim_00 * matA_dim_01) + block_size - 1) / block_size;
		constexpr uint64_t blocks_size	  = total_blocks_A * sizeof(block_q8_0);
		const uint64_t floats_B_size	  = (matA_dim_01 * matB_dim_01) * sizeof(float);
		const uint64_t outputs_C_size	  = (matA_dim_00 * matB_dim_01) * sizeof(float);
		uint64_t offset					  = 0;
		block_q8_0* d_blocks			  = reinterpret_cast<block_q8_0*>(static_cast<uint8_t*>(buffer.data()) + offset);
		offset							  = round_up_to_multiple<64>(offset + blocks_size);

		float* d_floats = reinterpret_cast<float*>(static_cast<uint8_t*>(buffer.data()) + offset);
		offset			= round_up_to_multiple<64>(offset + floats_B_size);

		float* d_outputs = reinterpret_cast<float*>(static_cast<uint8_t*>(buffer.data()) + offset);

		//hipMemset(d_floats, 0, floats_B_size);
		//hipMemset(d_outputs, 0, outputs_C_size);
		if (current_index > 0) {
			auto& previous_outputs = outputs[current_index - 1];
			hipError_t err		   = hipMemcpy(previous_outputs.data(), d_outputs, outputs_C_size, hipMemcpyDeviceToHost);
			if (err != hipSuccess) {
				std::cerr << "Failed to copy previous outputs from device: " + std::string(hipGetErrorString(err)) << std::endl;
			}
		}

		const auto& current_blocks = blocks[current_index];
		const auto& current_floats = floats[current_index];

		hipError_t err = hipMemcpy(d_blocks, current_blocks.data(), blocks_size, hipMemcpyHostToDevice);
		if (err != hipSuccess) {
			std::cerr << "Failed to copy blocks to device: " + std::string(hipGetErrorString(err)) << std::endl;
		}

		err = hipMemcpy(d_floats, current_floats.data(), floats_B_size, hipMemcpyHostToDevice);
		if (err != hipSuccess) {
			std::cerr << "Failed to copy floats to device: " + std::string(hipGetErrorString(err)) << std::endl;
		}

		//err = hipMemset(d_outputs, 0, outputs_C_size);
		if (err != hipSuccess) {
			std::cerr << "Failed to zero output buffer: " + std::string(hipGetErrorString(err)) << std::endl;
		}

		return 0;
	}
};

template<uint64_t matA_dim_00, uint64_t matA_dim_01, uint64_t block_size>
__global__ void ggml_cuda_mul_mat_kernel(const float* input01, const block_q8_0* input02, float* output, uint64_t matB_dim_01) {
	const uint64_t row = blockIdx.y * blockDim.y + threadIdx.y;
	const uint64_t col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row >= matA_dim_00 || col >= matB_dim_01)
		return;

	float sum = 0.0f;

	const uint64_t k_end = matA_dim_01 & ~3;

	uint64_t k = 0;
	for (; k < k_end; k += 4) {
#pragma unroll
		for (uint64_t i = 0; i < 4; ++i) {
			const uint64_t k_idx		 = k + i;
			const uint64_t linear_idx	 = row * matA_dim_01 + k_idx;
			const uint64_t block_idx	 = linear_idx / block_size;
			const uint64_t elem_in_block = linear_idx % block_size;

			const block_q8_0& block = input02[block_idx];
			const float scale		= __half2float(*reinterpret_cast<const __half*>(&block.scale));
			const float a_elem		= scale * static_cast<float>(block.quants[elem_in_block]);
			const float b_elem		= input01[k_idx * matB_dim_01 + col];

			sum += a_elem * b_elem;
		}
	}

	for (; k < matA_dim_01; ++k) {
		const uint64_t linear_idx	 = row * matA_dim_01 + k;
		const uint64_t block_idx	 = linear_idx / block_size;
		const uint64_t elem_in_block = linear_idx % block_size;

		const block_q8_0& block = input02[block_idx];
		const float scale		= __half2float(*reinterpret_cast<const __half*>(&block.scale));
		const float a_elem		= scale * static_cast<float>(block.quants[elem_in_block]);
		const float b_elem		= input01[k * matB_dim_01 + col];

		sum += a_elem * b_elem;
	}

	output[row * matB_dim_01 + col] = sum;
}

template<uint64_t matA_dim_00, uint64_t matA_dim_01, uint64_t block_size> struct ggml_cuda_mul_mat {
	BNCH_SWT_INLINE static uint64_t impl(cuda_buffer& buffer, uint64_t& current_index, std::vector<std::vector<float>>& floats, std::vector<std::vector<block_q8_0>>& blocks,
		std::vector<std::vector<float>>& outputs, uint64_t matB_dim_01) {
		auto& current_outputs = outputs[current_index];

		static constexpr uint64_t total_blocks_A = ((matA_dim_00 * matA_dim_01) + block_size - 1) / block_size;
		static constexpr uint64_t blocks_size	 = total_blocks_A * sizeof(block_q8_0);
		const uint64_t floats_B_size			 = (matA_dim_01 * matB_dim_01) * sizeof(float);
		const uint64_t outputs_C_size			 = (matA_dim_00 * matB_dim_01) * sizeof(float);

		uint64_t offset			   = 0;
		const block_q8_0* d_blocks = reinterpret_cast<const block_q8_0*>(static_cast<uint8_t*>(buffer.data()) + offset);
		offset					   = round_up_to_multiple<64>(offset + blocks_size);

		const float* d_floats = reinterpret_cast<const float*>(static_cast<uint8_t*>(buffer.data()) + offset);
		offset				  = round_up_to_multiple<64>(offset + floats_B_size);

		float* d_outputs = reinterpret_cast<float*>(static_cast<uint8_t*>(buffer.data()) + offset);

		uint64_t block_dim_x, block_dim_y;
		if (matB_dim_01 <= 4) {
			block_dim_x = matB_dim_01;
			block_dim_y = 256 / block_dim_x;
		} else if (matA_dim_00 <= 16) {
			block_dim_x = 32;
			block_dim_y = 16;
		} else {
			block_dim_x = 16;
			block_dim_y = 32;
		}

		block_dim_x = std::min(block_dim_x, matB_dim_01);
		block_dim_y = std::min(block_dim_y, matA_dim_00);

		const uint64_t grid_dim_x = (matB_dim_01 + block_dim_x - 1) / block_dim_x;
		const uint64_t grid_dim_y = (matA_dim_00 + block_dim_y - 1) / block_dim_y;

		dim3 blockDim(static_cast<uint64_t>(block_dim_x), static_cast<uint64_t>(block_dim_y));
		dim3 gridDim(static_cast<uint64_t>(grid_dim_x), static_cast<uint64_t>(grid_dim_y));

		ggml_cuda_mul_mat_kernel<matA_dim_00, matA_dim_01, block_size><<<gridDim, blockDim>>>(d_floats, d_blocks, d_outputs, matB_dim_01);

		hipError_t err = hipGetLastError();
		if (err != hipSuccess) {
			std::cerr << "GGML CUDA kernel launch failed: " + std::string(hipGetErrorString(err)) << std::endl;
		}

		err = hipDeviceSynchronize();
		if (err != hipSuccess) {
			std::cerr << "GGML CUDA kernel execution failed: " + std::string(hipGetErrorString(err)) << std::endl;
		}

		++current_index;
		return current_outputs.size() * sizeof(float);
	}
};

enum class kernel_type_profiles : uint8_t {
	fp16_mha,
	fp16_moe,
	bf16_mha,
	bf16_gqa,
	q4_mha,
	q4_gqa,
	q4_moe,
	q8_mha,
	q8_gqa,
	q8_moe,
	mixed_fp16_fp32,
	mixed_bf16_fp32,
	count,
};

enum class mul_mat_types {
	q,
	k,
	v,
	kq,
	kqv,
	kqv_out,
	ffn_gate,
	ffn_up,
	ffn_out,
};

template<mul_mat_types core_type, kernel_type_profiles kernel_type_profile> struct mul_mat_params;

template<> struct mul_mat_params<mul_mat_types::q, kernel_type_profiles::q8_gqa> {
	static constexpr uint64_t tile_size{ 16 };
};

template<> struct mul_mat_params<mul_mat_types::ffn_up, kernel_type_profiles::q8_gqa> {
	static constexpr uint64_t tile_size{ 16 };
};

#include <hipblas.h>
#include <hipblaslt.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

template<typename value_type> using x_type = decltype(std::remove_cvref_t<value_type>::x);

template<typename value_type>
concept uint_cuda_types = std::is_unsigned_v<x_type<value_type>> && std::is_integral_v<x_type<value_type>>;

template<typename value_type>
concept int_cuda_types = std::is_signed_v<x_type<value_type>> && std::is_integral_v<x_type<value_type>> && !uint_cuda_types<value_type>;

template<typename value_type>
concept int8_cuda_types = int_cuda_types<x_type<value_type>> && sizeof(x_type<value_type>) == 1;

template<typename value_type>
concept int16_cuda_types = int_cuda_types<x_type<value_type>> && sizeof(x_type<value_type>) == 2;

template<typename value_type>
concept int32_cuda_types = int_cuda_types<x_type<value_type>> && sizeof(x_type<value_type>) == 4;

template<typename value_type>
concept int64_cuda_types = int_cuda_types<x_type<value_type>> && sizeof(x_type<value_type>) == 8;

template<typename value_type>
concept uint8_cuda_types = uint_cuda_types<x_type<value_type>> && sizeof(x_type<value_type>) == 1;

template<typename value_type>
concept uint16_cuda_types = uint_cuda_types<x_type<value_type>> && sizeof(x_type<value_type>) == 2;

template<typename value_type>
concept uint32_cuda_types = uint_cuda_types<x_type<value_type>> && sizeof(x_type<value_type>) == 4;

template<typename value_type>
concept uint64_cuda_types = uint_cuda_types<x_type<value_type>> && sizeof(x_type<value_type>) == 8;

template<typename value_type>
concept float_cuda_types = std::floating_point<x_type<value_type>>;

template<typename value_type>
concept float32_cuda_types = float_cuda_types<value_type> && sizeof(x_type<value_type>) == 4;

template<typename value_type>
concept float64_cuda_types = float_cuda_types<value_type> && sizeof(x_type<value_type>) == 8;

template<typename value_type>
concept r_value_reference_types = std::is_rvalue_reference_v<value_type>;

template<typename value_type> BNCH_SWT_INLINE __device__ constexpr value_type&& forward_device(value_type& arg) noexcept {
	return static_cast<value_type&&>(arg);
}

template<r_value_reference_types value_type> __device__ BNCH_SWT_INLINE constexpr value_type forward_device(value_type arg) noexcept {
	return arg;
}

enum class get_value_type_errors {
	invalid_type,
};

template<typename value_type>
concept dim04_types = requires() { std::remove_cvref_t<value_type>::w; };

template<typename value_type>
concept dim03_types = requires() { std::remove_cvref_t<value_type>::z; } && !dim04_types<value_type>;

template<typename value_type>
concept dim02_types = requires() { std::remove_cvref_t<value_type>::y; } && !dim03_types<value_type> && !dim04_types<value_type>;

template<typename value_type>
concept dim01_types = requires() { std::remove_cvref_t<value_type>::x; } && !dim02_types<value_type> && !dim03_types<value_type> && !dim04_types<value_type>;

template<typename value_type>
concept dim_types = requires() { std::remove_cvref_t<value_type>::x; };

template<typename value_type> struct get_value_type {
	template<typename... value_types> BNCH_SWT_INLINE __device__ static constexpr auto impl(value_types&&... args) {
		static_assert(false, "Failed to specialize this class!");
	}
};

template<int8_cuda_types value_type> struct get_value_type<value_type> {
	template<typename... value_types> BNCH_SWT_INLINE __device__ static constexpr auto impl(value_types&&... args) {
		if constexpr (dim01_types<value_type>) {
			return make_char1(forward_device<value_types>(args)...);
		} else if constexpr (dim02_types<value_type>) {
			return make_char2(forward_device<value_types>(args)...);
		} else if constexpr (dim03_types<value_type>) {
			return make_char3(forward_device<value_types>(args)...);
		} else if constexpr (dim04_types<value_type>) {
			return make_char4(forward_device<value_types>(args)...);
		}
	}
};

template<int16_cuda_types value_type> struct get_value_type<value_type> {
	template<typename... value_types> BNCH_SWT_INLINE __device__ static constexpr auto impl(value_types&&... args) {
		if constexpr (dim01_types<value_type>) {
			return make_short1(forward_device<value_types>(args)...);
		} else if constexpr (dim02_types<value_type>) {
			return make_short2(forward_device<value_types>(args)...);
		} else if constexpr (dim03_types<value_type>) {
			return make_short3(forward_device<value_types>(args)...);
		} else if constexpr (dim04_types<value_type>) {
			return make_short4(forward_device<value_types>(args)...);
		}
	}
};

template<int32_cuda_types value_type> struct get_value_type<value_type> {
	template<typename... value_types> BNCH_SWT_INLINE __device__ static constexpr auto impl(value_types&&... args) {
		if constexpr (dim01_types<value_type>) {
			return make_int1(forward_device<value_types>(args)...);
		} else if constexpr (dim02_types<value_type>) {
			return make_int2(forward_device<value_types>(args)...);
		} else if constexpr (dim03_types<value_type>) {
			return make_int3(forward_device<value_types>(args)...);
		} else if constexpr (dim04_types<value_type>) {
			return make_int4(forward_device<value_types>(args)...);
		}
	}
};

template<int64_cuda_types value_type> struct get_value_type<value_type> {
	template<typename... value_types> BNCH_SWT_INLINE __device__ static constexpr auto impl(value_types&&... args) {
		if constexpr (dim01_types<value_type>) {
			return make_long1(forward_device<value_types>(args)...);
		} else if constexpr (dim02_types<value_type>) {
			return make_long2(forward_device<value_types>(args)...);
		} else if constexpr (dim03_types<value_type>) {
			return make_long3(forward_device<value_types>(args)...);
		} else if constexpr (dim04_types<value_type>) {
			return make_long4(forward_device<value_types>(args)...);
		}
	}
};

template<uint8_cuda_types value_type> struct get_value_type<value_type> {
	template<typename... value_types> BNCH_SWT_INLINE __device__ static constexpr auto impl(value_types&&... args) {
		if constexpr (dim01_types<value_type>) {
			return make_uchar1(forward_device<value_types>(args)...);
		} else if constexpr (dim02_types<value_type>) {
			return make_uchar2(forward_device<value_types>(args)...);
		} else if constexpr (dim03_types<value_type>) {
			return make_uchar3(forward_device<value_types>(args)...);
		} else if constexpr (dim04_types<value_type>) {
			return make_uchar4(forward_device<value_types>(args)...);
		}
	}
};

template<uint16_cuda_types value_type> struct get_value_type<value_type> {
	template<typename... value_types> BNCH_SWT_INLINE __device__ static constexpr auto impl(value_types&&... args) {
		if constexpr (dim01_types<value_type>) {
			return make_ushort1(forward_device<value_types>(args)...);
		} else if constexpr (dim02_types<value_type>) {
			return make_ushort2(forward_device<value_types>(args)...);
		} else if constexpr (dim03_types<value_type>) {
			return make_ushort3(forward_device<value_types>(args)...);
		} else if constexpr (dim04_types<value_type>) {
			return make_ushort4(forward_device<value_types>(args)...);
		}
	}
};

template<uint32_cuda_types value_type> struct get_value_type<value_type> {
	template<typename... value_types> BNCH_SWT_INLINE __device__ static constexpr auto impl(value_types&&... args) {
		if constexpr (dim01_types<value_type>) {
			return make_uint1(forward_device<value_types>(args)...);
		} else if constexpr (dim02_types<value_type>) {
			return make_uint2(forward_device<value_types>(args)...);
		} else if constexpr (dim03_types<value_type>) {
			return make_uint3(forward_device<value_types>(args)...);
		} else if constexpr (dim04_types<value_type>) {
			return make_uint4(forward_device<value_types>(args)...);
		}
	}
};

template<uint64_cuda_types value_type> struct get_value_type<value_type> {
	template<typename... value_types> BNCH_SWT_INLINE __device__ static constexpr auto impl(value_types&&... args) {
		if constexpr (dim01_types<value_type>) {
			return make_ulong1(forward_device<value_types>(args)...);
		} else if constexpr (dim02_types<value_type>) {
			return make_ulong2(forward_device<value_types>(args)...);
		} else if constexpr (dim03_types<value_type>) {
			return make_ulong3(forward_device<value_types>(args)...);
		} else if constexpr (dim04_types<value_type>) {
			return make_ulong4(forward_device<value_types>(args)...);
		}
	}
};

template<float32_cuda_types value_type> struct get_value_type<value_type> {
	template<typename... value_types> BNCH_SWT_INLINE __device__ static constexpr auto impl(value_types&&... args) {
		if constexpr (dim01_types<value_type>) {
			return make_float1(forward_device<value_types>(args)...);
		} else if constexpr (dim02_types<value_type>) {
			return make_float2(forward_device<value_types>(args)...);
		} else if constexpr (dim03_types<value_type>) {
			return make_float3(forward_device<value_types>(args)...);
		} else if constexpr (dim04_types<value_type>) {
			return make_float4(forward_device<value_types>(args)...);
		}
	}
};

template<float64_cuda_types value_type> struct get_value_type<value_type> {
	template<typename... value_types> BNCH_SWT_INLINE __device__ static constexpr auto impl(value_types&&... args) {
		if constexpr (dim01_types<value_type>) {
			return make_double1(forward_device<value_types>(args)...);
		} else if constexpr (dim02_types<value_type>) {
			return make_double2(forward_device<value_types>(args)...);
		} else if constexpr (dim03_types<value_type>) {
			return make_double3(forward_device<value_types>(args)...);
		} else if constexpr (dim04_types<value_type>) {
			return make_double4(forward_device<value_types>(args)...);
		}
	}
};

enum class binary_op_types {
	add,
	sub,
	mul,
	div,
};

template<binary_op_types> struct binary_op_core;

template<> struct binary_op_core<binary_op_types::add> {
	template<typename value_type01, std::convertible_to<value_type01> value_type02>
	BNCH_SWT_INLINE static __device__ value_type01 impl(const value_type01& val01, value_type02&& val02) {
		return val01 + static_cast<value_type01>(val02);
	}
};

template<> struct binary_op_core<binary_op_types::sub> {
	template<typename value_type01, std::convertible_to<value_type01> value_type02>
	BNCH_SWT_INLINE static __device__ value_type01 impl(const value_type01& val01, value_type02&& val02) {
		return val01 - static_cast<value_type01>(val02);
	}
};

template<> struct binary_op_core<binary_op_types::mul> {
	template<typename value_type01, std::convertible_to<value_type01> value_type02>
	BNCH_SWT_INLINE static __device__ value_type01 impl(const value_type01& val01, value_type02&& val02) {
		return val01 * static_cast<value_type01>(val02);
	}
};

template<> struct binary_op_core<binary_op_types::div> {
	template<typename value_type01, std::convertible_to<value_type01> value_type02>
	BNCH_SWT_INLINE static __device__ value_type01 impl(const value_type01& val01, value_type02&& val02) {
		return val01 / static_cast<value_type01>(val02);
	}
};

template<binary_op_types binary_op_type> struct binary_op_base {
	using op_core_type = binary_op_core<binary_op_type>;
	template<typename value_type01, std::convertible_to<value_type01> value_type02>
	BNCH_SWT_INLINE static __device__ value_type01 impl_one(const value_type01& val01, value_type02&& val02) {
		return get_value_type<value_type01>::impl(op_core_type::impl(val01.x, forward_device<value_type02>(val02).x));
	}

	template<typename value_type01, std::convertible_to<value_type01> value_type02>
	BNCH_SWT_INLINE static __device__ value_type01 impl_two(const value_type01& val01, value_type02&& val02) {
		return get_value_type<value_type01>::impl(op_core_type::impl(val01.x, forward_device<value_type02>(val02).x),
			op_core_type::impl(val01.y, forward_device<value_type02>(val02).y));
	}

	template<typename value_type01, std::convertible_to<value_type01> value_type02>
	BNCH_SWT_INLINE static __device__ value_type01 impl_three(const value_type01& val01, value_type02&& val02) {
		return get_value_type<value_type01>::impl(op_core_type::impl(val01.x, forward_device<value_type02>(val02).x),
			op_core_type::impl(val01.y, forward_device<value_type02>(val02).y), op_core_type::impl(val01.z, forward_device<value_type02>(val02).z));
	}

	template<typename value_type01, std::convertible_to<value_type01> value_type02>
	BNCH_SWT_INLINE static __device__ value_type01 impl_four(const value_type01& val01, value_type02&& val02) {
		return get_value_type<value_type01>::impl(op_core_type::impl(val01.x, forward_device<value_type02>(val02).x),
			op_core_type::impl(val01.y, forward_device<value_type02>(val02).y), op_core_type::impl(val01.z, forward_device<value_type02>(val02).z),
			op_core_type::impl(val01.w, forward_device<value_type02>(val02).w));
	}
};

template<binary_op_types binary_op_type> struct binary_op {
	template<typename value_type01, std::convertible_to<value_type01> value_type02>
	BNCH_SWT_INLINE static __device__ value_type01 impl(const value_type01& val01, value_type02&& val02) {
		if constexpr (dim04_types<value_type01>) {
			return binary_op_base<binary_op_type>::impl_four(val01, forward_device<value_type02>(val02));
		} else if constexpr (dim03_types<value_type01>) {
			return binary_op_base<binary_op_type>::impl_three(val01, forward_device<value_type02>(val02));
		} else if constexpr (dim02_types<value_type01>) {
			return binary_op_base<binary_op_type>::impl_two(val01, forward_device<value_type02>(val02));
		} else {
			return binary_op_base<binary_op_type>::impl_one(val01, forward_device<value_type02>(val02));
		}
	}
};

template<dim_types value_type01, std::convertible_to<value_type01> value_type02>
BNCH_SWT_INLINE __device__ value_type01 operator+=(const value_type01& val01, value_type02&& val02) {
	return binary_op<binary_op_types::add>::impl(val01, val02);
}

template<dim_types value_type01, std::convertible_to<value_type01> value_type02>
BNCH_SWT_INLINE __device__ value_type01 operator+(const value_type01& val01, value_type02&& val02) {
	return binary_op<binary_op_types::add>::impl(val01, val02);
}

template<dim_types value_type01, std::convertible_to<value_type01> value_type02>
BNCH_SWT_INLINE __device__ value_type01 operator*=(const value_type01& val01, value_type02&& val02) {
	return binary_op<binary_op_types::mul>::impl(val01, val02);
}

template<dim_types value_type01, std::convertible_to<value_type01> value_type02>
BNCH_SWT_INLINE __device__ value_type01 operator*(const value_type01& val01, value_type02&& val02) {
	return binary_op<binary_op_types::mul>::impl(val01, val02);
}

template<dim_types value_type01, std::convertible_to<value_type01> value_type02>
BNCH_SWT_INLINE __device__ value_type01 operator-=(const value_type01& val01, value_type02&& val02) {
	return binary_op<binary_op_types::sub>::impl(val01, val02);
}

template<dim_types value_type01, std::convertible_to<value_type01> value_type02>
BNCH_SWT_INLINE __device__ value_type01 operator-(const value_type01& val01, value_type02&& val02) {
	return binary_op<binary_op_types::sub>::impl(val01, val02);
}

template<dim_types value_type01, std::convertible_to<value_type01> value_type02>
BNCH_SWT_INLINE __device__ value_type01 operator/=(const value_type01& val01, value_type02&& val02) {
	return binary_op<binary_op_types::div>::impl(val01, val02);
}

template<dim_types value_type01, std::convertible_to<value_type01> value_type02>
BNCH_SWT_INLINE __device__ value_type01 operator/(const value_type01& val01, value_type02&& val02) {
	return binary_op<binary_op_types::div>::impl(val01, val02);
}

struct gpu_properties {
	static constexpr uint64_t sm_count{ 70ull };
	static constexpr uint64_t max_threads_per_sm{ 1536ull };
	static constexpr uint64_t max_threads_per_block{ 1024ull };
	static constexpr uint64_t warp_size{ 32ull };
	static constexpr uint64_t l2_cache_size{ 50331648ull };
	static constexpr uint64_t shared_mem_per_block{ 49152ull };
	static constexpr uint64_t memory_bus_width{ 256ull };
	static constexpr uint64_t memory_clock_rate{ 14001000ull };
	static constexpr uint64_t major_compute_capability{ 12ull };
	static constexpr uint64_t minor_compute_capability{ 0ull };
	static constexpr uint64_t max_grid_size_x{ 2147483647ull };
	static constexpr uint64_t gpu_arch_index{ 4ull };
	static constexpr uint64_t total_threads{ 107520ull };
	static constexpr uint64_t optimal_block_size{ 512ull };
	static constexpr uint64_t optimal_grid_size{ 210ull };
};

template<uint64_t block_m, uint64_t block_n, uint64_t block_k, uint64_t warp_m_new, uint64_t warp_n_new, uint64_t thread_m_new, uint64_t thread_n_new> struct cuda_kernel_traits {
	static constexpr uint64_t block_tile_m		= block_m;
	static constexpr uint64_t block_tile_n		= block_n;
	static constexpr uint64_t block_tile_k		= block_k;
	static constexpr uint64_t warp_tile_m		= warp_m_new;
	static constexpr uint64_t warp_tile_n		= warp_n_new;
	static constexpr uint64_t thread_tile_m		= thread_m_new;
	static constexpr uint64_t thread_tile_n		= thread_n_new;
	static constexpr uint64_t warps_m			= block_m / warp_m_new;
	static constexpr uint64_t warps_n			= block_n / warp_n_new;
	static constexpr uint64_t threads_per_warp	= gpu_properties::warp_size;
	static constexpr uint64_t threads_per_block = warps_m * warps_n * threads_per_warp;

	static_assert(block_m > 0, "block_m must be greater than 0");
	static_assert(block_n > 0, "block_n must be greater than 0");
	static_assert(block_k > 0, "block_k must be greater than 0");
	static_assert(warp_m_new > 0, "warp_m must be greater than 0");
	static_assert(warp_n_new > 0, "warp_n must be greater than 0");
	static_assert(thread_m_new > 0, "thread_m must be greater than 0");
	static_assert(thread_n_new > 0, "thread_n must be greater than 0");

	static_assert(block_m % warp_m_new == 0, "block_m must be evenly divisible by warp_m");
	static_assert(block_n % warp_n_new == 0, "block_n must be evenly divisible by warp_n");

	static_assert(warp_m_new % thread_m_new == 0, "warp_m must be evenly divisible by thread_m");
	static_assert(warp_n_new % thread_n_new == 0, "warp_n must be evenly divisible by thread_n");

	static_assert((warp_m_new / thread_m_new) * (warp_n_new / thread_n_new) == gpu_properties::warp_size, "Warp configuration must result in exactly warp_size threads per warp");

	static_assert(threads_per_block <= gpu_properties::max_threads_per_block, "threads_per_block cannot exceed max_threads_per_block");
	static_assert(threads_per_block >= gpu_properties::warp_size, "threads_per_block must be at least warp_size");

	static_assert(block_m <= 512, "block_m should not exceed 512 for reasonable shared memory usage");
	static_assert(block_n <= 512, "block_n should not exceed 512 for reasonable shared memory usage");
	static_assert(block_k <= 64, "block_k should not exceed 64 for reasonable register usage");

	static_assert(block_k % 4 == 0, "block_k should be a multiple of 4 for vectorized loads");

	static_assert(thread_m_new <= 8, "thread_m should not exceed 8 for reasonable register usage");
	static_assert(thread_n_new <= 8, "thread_n should not exceed 8 for reasonable register usage");

	static_assert(warps_m > 0 && warps_n > 0, "Must have at least one warp in each dimension");
	static_assert(warps_m * warps_n <= 32, "Total warps per block should not exceed 32");

	static constexpr uint64_t shared_mem_usage = 2 * (block_m * block_k + block_k * block_n) * sizeof(float);
	static_assert(shared_mem_usage <= gpu_properties::shared_mem_per_block, "Estimated shared memory usage exceeds shared_mem_per_block limit");

	static_assert(threads_per_block % gpu_properties::warp_size == 0, "threads_per_block must be a multiple of warp_size");
};

using mul_mat_1_to_1024 = cuda_kernel_traits<32, 64, 16, 16, 32, 4, 4>;

template<uint64_t M, uint64_t K, typename traits>
__device__ __forceinline__ void load_smem_tile_A(float* smem_A, const block_q8_0* A_global, uint64_t N, uint64_t k_offset, uint64_t block_row) {
	constexpr uint64_t block_m			 = traits::block_tile_m;
	constexpr uint64_t block_k			 = traits::block_tile_k;
	constexpr uint64_t threads_per_block = traits::threads_per_block;
	const uint64_t tid					 = threadIdx.x;
	const uint64_t k_blocks				 = (K + 31) / 32;
	const uint64_t elements_per_block	 = block_m * block_k;
	const uint64_t vec4_elements		 = elements_per_block / 4;
	const uint64_t vec4_per_thread		 = (vec4_elements + threads_per_block - 1) / threads_per_block;
	for (uint64_t i = 0; i < vec4_per_thread; i++) {
		const uint64_t vec4_idx	  = tid + i * threads_per_block;
		const uint64_t linear_idx = vec4_idx * 4;
		const uint64_t row		  = linear_idx / block_k;
		const uint64_t col		  = linear_idx % block_k;
		const uint64_t global_row	= block_row + row;
		const uint64_t global_col	= k_offset + col;
		const uint64_t q8_block_row = global_row;
		const uint64_t q8_block_col = global_col / 32;
		const uint64_t q8_elem_idx	= global_col % 32;
		const uint64_t q8_block_idx = q8_block_row * k_blocks + q8_block_col;
		const block_q8_0& q8_block	= A_global[q8_block_idx];
		const float scale_raw		= __half2float(*reinterpret_cast<const __half*>(&q8_block.scale));
		const uint64_t smem_offset						 = row * block_k + col;
		*reinterpret_cast<float4*>(&smem_A[smem_offset]) = make_float4(static_cast<float>(q8_block.quants[q8_elem_idx]), static_cast<float>(q8_block.quants[q8_elem_idx + 1]),
															   static_cast<float>(q8_block.quants[q8_elem_idx + 2]), static_cast<float>(q8_block.quants[q8_elem_idx + 3])) *
			make_float4(scale_raw, scale_raw, scale_raw, scale_raw);
	}
}

template<uint64_t M, uint64_t K, typename traits>
__device__ __forceinline__ void load_smem_tile_B(float* smem_B, const float* B_global, uint64_t N, uint64_t k_offset, uint64_t block_col) {
	constexpr uint64_t block_n			 = traits::block_tile_n;
	constexpr uint64_t block_k			 = traits::block_tile_k;
	constexpr uint64_t threads_per_block = traits::threads_per_block;

	const uint64_t tid					 = threadIdx.x;
	const uint64_t vec4_cols_per_row	 = block_n / 4;
	const uint64_t total_vec4_loads		 = block_k * vec4_cols_per_row;
	const uint64_t vec4_loads_per_thread = (total_vec4_loads + threads_per_block - 1) / threads_per_block;

	for (uint64_t i = 0; i < vec4_loads_per_thread; i++) {
		const uint64_t vec4_idx = tid + i * threads_per_block;
		if (vec4_idx < total_vec4_loads) {
			const uint64_t row		= vec4_idx / vec4_cols_per_row;
			const uint64_t vec4_col = vec4_idx % vec4_cols_per_row;
			const uint64_t col		= vec4_col * 4;

			const uint64_t global_row = k_offset + row;
			const uint64_t global_col = block_col + col;

			if (global_row < K && global_col + 3 < N) {
				const uint64_t global_offset					 = global_row * N + global_col;
				const uint64_t smem_offset						 = row * block_n + col;
				*reinterpret_cast<float4*>(&smem_B[smem_offset]) = *reinterpret_cast<const float4*>(&B_global[global_offset]);
			} else {
				for (uint64_t elem = 0; elem < 4; elem++) {
					const uint64_t elem_global_col = global_col + elem;
					const uint64_t elem_col		   = col + elem;
					if (global_row < K && elem_global_col < N && elem_col < block_n) {
						smem_B[row * block_n + elem_col] = B_global[global_row * N + elem_global_col];
					}
				}
			}
		}
	}
}

template<typename traits> __device__ __forceinline__ void compute_warp_tile(float* smem_A, float* smem_B, float accumulator[traits::thread_tile_m][traits::thread_tile_n],
	uint64_t warp_row, uint64_t warp_col) {
	constexpr uint64_t warp_m	= traits::warp_tile_m;
	constexpr uint64_t warp_n	= traits::warp_tile_n;
	constexpr uint64_t thread_m = traits::thread_tile_m;
	constexpr uint64_t thread_n = traits::thread_tile_n;
	constexpr uint64_t block_k	= traits::block_tile_k;
	constexpr uint64_t block_n	= traits::block_tile_n;
	constexpr uint64_t block_m	= traits::block_tile_m;

	uint64_t lane_id		 = threadIdx.x % 32;
	uint64_t threads_per_row = warp_n / thread_n;
	uint64_t thread_row		 = lane_id / threads_per_row;
	uint64_t thread_col		 = lane_id % threads_per_row;

	float frag_A[thread_m];
	float frag_B[thread_n];

	for (uint64_t k = 0; k < block_k; k++) {
#pragma unroll
		for (uint64_t tm = 0; tm < thread_m; tm++) {
			uint64_t smem_row = warp_row + thread_row * thread_m + tm;
			frag_A[tm]		  = smem_A[smem_row * block_k + k];
		}

#pragma unroll
		for (uint64_t tn = 0; tn < thread_n; tn++) {
			uint64_t smem_col = warp_col + thread_col * thread_n + tn;
			frag_B[tn]		  = smem_B[k * block_n + smem_col];
		}

#pragma unroll
		for (uint64_t tm = 0; tm < thread_m; tm++) {
#pragma unroll
			for (uint64_t tn = 0; tn < thread_n; tn++) {
				accumulator[tm][tn] += frag_A[tm] * frag_B[tn];
			}
		}
	}
}

template<typename traits> __device__ __forceinline__ void store_output_tile(float* C_global, float accumulator[traits::thread_tile_m][traits::thread_tile_n], uint64_t M,
	uint64_t N, uint64_t block_row, uint64_t block_col, uint64_t warp_row, uint64_t warp_col) {
	constexpr uint64_t thread_m = traits::thread_tile_m;
	constexpr uint64_t thread_n = traits::thread_tile_n;
	constexpr uint64_t warp_n	= traits::warp_tile_n;

	uint64_t lane_id		 = threadIdx.x % 32;
	uint64_t threads_per_row = warp_n / thread_n;
	uint64_t thread_row		 = lane_id / threads_per_row;
	uint64_t thread_col		 = lane_id % threads_per_row;

#pragma unroll
	for (uint64_t tm = 0; tm < thread_m; tm++) {
#pragma unroll
		for (uint64_t tn = 0; tn < thread_n; tn++) {
			uint64_t global_row = block_row + warp_row + thread_row * thread_m + tm;
			uint64_t global_col = block_col + warp_col + thread_col * thread_n + tn;

			if (global_row < M && global_col < N) {
				C_global[global_row * N + global_col] = accumulator[tm][tn];
			}
		}
	}
}

template<uint64_t M, uint64_t K> __global__ void rt_tm_gemm_kernel(const block_q8_0* A, const float* B, float* C, uint64_t N) {
	using traits = mul_mat_1_to_1024;

	constexpr uint64_t block_m	= traits::block_tile_m;
	constexpr uint64_t block_n	= traits::block_tile_n;
	constexpr uint64_t block_k	= traits::block_tile_k;
	constexpr uint64_t warp_m	= traits::warp_tile_m;
	constexpr uint64_t warp_n	= traits::warp_tile_n;
	constexpr uint64_t thread_m = traits::thread_tile_m;
	constexpr uint64_t thread_n = traits::thread_tile_n;
	constexpr uint64_t warps_m	= traits::warps_m;
	constexpr uint64_t warps_n	= traits::warps_n;

	__shared__ float smem_A[2][block_m * block_k];
	__shared__ float smem_B[2][block_k * block_n];

	uint64_t block_row = blockIdx.y * block_m;
	uint64_t block_col = blockIdx.x * block_n;

	uint64_t warp_id  = threadIdx.x / 32;
	uint64_t warp_row = (warp_id / warps_n) * warp_m;
	uint64_t warp_col = (warp_id % warps_n) * warp_n;

	float accumulator[thread_m][thread_n];
#pragma unroll
	for (uint64_t tm = 0; tm < thread_m; tm++) {
#pragma unroll
		for (uint64_t tn = 0; tn < thread_n; tn++) {
			accumulator[tm][tn] = 0.0f;
		}
	}

	uint64_t smem_write_stage = 0;
	uint64_t smem_read_stage  = 0;

	load_smem_tile_A<M, K, traits>(smem_A[smem_write_stage], A, N, 0, block_row);
	load_smem_tile_B<M, K, traits>(smem_B[smem_write_stage], B, N, 0, block_col);
	__syncthreads();

	for (uint64_t k_tile = 0; k_tile < K; k_tile += block_k) {
		smem_read_stage	 = smem_write_stage;
		smem_write_stage = 1 - smem_write_stage;

		if (k_tile + block_k < K) {
			load_smem_tile_A<M, K, traits>(smem_A[smem_write_stage], A, N, k_tile + block_k, block_row);
			load_smem_tile_B<M, K, traits>(smem_B[smem_write_stage], B, N, k_tile + block_k, block_col);
		}

		compute_warp_tile<traits>(smem_A[smem_read_stage], smem_B[smem_read_stage], accumulator, warp_row, warp_col);

		__syncthreads();
	}

	store_output_tile<traits>(C, accumulator, M, N, block_row, block_col, warp_row, warp_col);
}

template<uint64_t matA_dim_00, uint64_t matA_dim_01, uint64_t block_size, mul_mat_types mul_mat_type> struct rt_tm_mul_mat {
	BNCH_SWT_INLINE static uint64_t impl(cuda_buffer& buffer, uint64_t& current_index, std::vector<std::vector<float>>& floats, std::vector<std::vector<block_q8_0>>& blocks,
		std::vector<std::vector<float>>& outputs, uint64_t matB_dim_01) {
		auto& current_outputs = outputs[current_index];

		static constexpr uint64_t total_blocks_A = ((matA_dim_00 * matA_dim_01) + block_size - 1) / block_size;
		static constexpr uint64_t blocks_size	 = total_blocks_A * sizeof(block_q8_0);
		const uint64_t floats_B_size			 = (matA_dim_01 * matB_dim_01) * sizeof(float);
		const uint64_t outputs_C_size			 = (matA_dim_00 * matB_dim_01) * sizeof(float);

		uint64_t offset			   = 0;
		const block_q8_0* d_blocks = reinterpret_cast<const block_q8_0*>(static_cast<uint8_t*>(buffer.data()) + offset);
		offset					   = round_up_to_multiple<64>(offset + blocks_size);

		const float* d_floats = reinterpret_cast<const float*>(static_cast<uint8_t*>(buffer.data()) + offset);
		offset				  = round_up_to_multiple<64>(offset + floats_B_size);

		float* d_outputs = reinterpret_cast<float*>(static_cast<uint8_t*>(buffer.data()) + offset);

		using traits = mul_mat_1_to_1024;

		dim3 grid{ static_cast<uint32_t>((matB_dim_01 + traits::block_tile_n - 1) / traits::block_tile_n),
			static_cast<uint32_t>((matA_dim_00 + traits::block_tile_m - 1) / traits::block_tile_m) };
		dim3 block{ traits::threads_per_block };
		rt_tm_gemm_kernel<matA_dim_00, matA_dim_01><<<grid, block>>>(d_blocks, d_floats, d_outputs, matB_dim_01);

		hipError_t err = hipGetLastError();
		if (err != hipSuccess) {
			std::cerr << "Kernel launch failed: " << hipGetErrorString(err) << std::endl;
		}

		err = hipDeviceSynchronize();
		if (err != hipSuccess) {
			std::cerr << "Kernel execution failed: " << hipGetErrorString(err) << std::endl;
		}

		++current_index;
		return current_outputs.size() * sizeof(float);
	}
};

__global__ void dequantize_q8_vectorized_kernel(const block_q8_0* input_blocks, float* output, uint64_t total_elements) {
	const uint64_t idx	  = blockIdx.x * blockDim.x + threadIdx.x;
	const uint64_t stride = blockDim.x * gridDim.x;

	for (uint64_t i = idx; i < total_elements; i += stride) {
		const uint64_t block_idx	 = i / 32;
		const uint64_t elem_in_block = i % 32;

		const block_q8_0& block = input_blocks[block_idx];
		const float scale		= __half2float(*reinterpret_cast<const __half*>(&block.scale));
		output[i]				= scale * static_cast<float>(block.quants[elem_in_block]);
	}
}

template<uint64_t matA_dim_00, uint64_t matA_dim_01, uint64_t block_size, mul_mat_types mul_mat_type> struct nvcuda_cublas_mul_mat_simple {
	inline static hipblasHandle_t cublas_handle;
	inline static bool handle_initialized;

	static void initialize_handle() {
		if (!handle_initialized) {
			hipblasCreate(&cublas_handle);
			hipblasSetMathMode(cublas_handle, HIPBLAS_TENSOR_OP_MATH);
			handle_initialized = true;
		}
	}

	BNCH_SWT_INLINE static uint64_t impl(cuda_buffer& buffer, uint64_t& current_index, std::vector<std::vector<float>>& floats, std::vector<std::vector<block_q8_0>>& blocks,
		std::vector<std::vector<float>>& outputs, uint64_t matB_dim_01) {
		initialize_handle();
		auto& current_outputs = outputs[current_index];

		static constexpr uint64_t total_blocks_A = ((matA_dim_00 * matA_dim_01) + block_size - 1) / block_size;
		static constexpr uint64_t blocks_size	 = total_blocks_A * sizeof(block_q8_0);
		const uint64_t floats_B_size			 = (matA_dim_01 * matB_dim_01) * sizeof(float);
		const uint64_t outputs_C_size			 = (matA_dim_00 * matB_dim_01) * sizeof(float);
		static constexpr uint64_t dequant_A_size = (matA_dim_00 * matA_dim_01) * sizeof(float);
		uint64_t offset							 = 0;
		const block_q8_0* d_blocks				 = reinterpret_cast<const block_q8_0*>(static_cast<uint8_t*>(buffer.data()) + offset);
		offset									 = round_up_to_multiple<64>(offset + blocks_size);

		const float* d_floats_B = reinterpret_cast<const float*>(static_cast<uint8_t*>(buffer.data()) + offset);
		offset					= round_up_to_multiple<64>(offset + floats_B_size);

		float* d_outputs = reinterpret_cast<float*>(static_cast<uint8_t*>(buffer.data()) + offset);
		offset			 = round_up_to_multiple<64>(offset + outputs_C_size);

		float* d_dequant_A = reinterpret_cast<float*>(static_cast<uint8_t*>(buffer.data()) + offset);

		const uint64_t required_size = offset + dequant_A_size;
		if (required_size > buffer.size()) {
			std::cerr << "❌ BUFFER OVERFLOW! Required: " << required_size << " Available: " << buffer.size() << std::endl;
			return 0;
		}

		if (( uint8_t* )d_dequant_A < ( uint8_t* )buffer.data() || ( uint8_t* )d_dequant_A >= ( uint8_t* )buffer.data() + buffer.size()) {
			std::cerr << "❌ INVALID POINTER! d_dequant_A is outside buffer bounds!" << std::endl;
			std::cerr << "   Buffer start: " << ( void* )buffer.data() << std::endl;
			std::cerr << "   Buffer end: " << ( void* )(( uint8_t* )buffer.data() + buffer.size()) << std::endl;
			std::cerr << "   d_dequant_A: " << ( void* )d_dequant_A << std::endl;
			return 0;
		}

		const uint64_t total_elements_A = matA_dim_00 * matA_dim_01;
		const dim3 dequant_grid((total_elements_A + 1023) / 1024);
		const dim3 dequant_block(1024);

		dequantize_q8_vectorized_kernel<<<dequant_grid, dequant_block>>>(d_blocks, d_dequant_A, total_elements_A);

		hipError_t dequant_err = hipDeviceSynchronize();
		if (dequant_err != hipSuccess) {
			std::cerr << "❌ Dequantization kernel failed: " << hipGetErrorString(dequant_err) << std::endl;
			return 0;
		}


		const float alpha = 1.0f;
		const float beta  = 0.0f;

		hipblasStatus_t status = hipblasSgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, matB_dim_01, matA_dim_00, matA_dim_01, &alpha, d_floats_B, matB_dim_01, d_dequant_A,
			matA_dim_01, &beta, d_outputs, matB_dim_01);

		if (status != HIPBLAS_STATUS_SUCCESS) {
			std::cerr << "❌ cuBLAS failed with status: " << status << std::endl;
		}

		hipError_t err = hipGetLastError();
		if (err != hipSuccess) {
			std::cerr << "❌ CUDA error after cuBLAS: " << hipGetErrorString(err) << std::endl;
		}

		err = hipDeviceSynchronize();
		if (err != hipSuccess) {
			std::cerr << "❌ CUDA synchronization failed: " << hipGetErrorString(err) << std::endl;
		}


		++current_index;
		return current_outputs.size() * sizeof(float);
	}
};

__global__ void float_to_half_kernel(const float* input, __half* output, uint64_t total_elements) {
	const uint64_t idx	  = blockIdx.x * blockDim.x + threadIdx.x;
	const uint64_t stride = blockDim.x * gridDim.x;

	for (uint64_t i = idx; i < total_elements; i += stride) {
		output[i] = __float2half(input[i]);
	}
}

template<bnch_swt::string_literal rhs> inline void compare_outputs(const std::vector<std::vector<float>>& outputs01, const std::vector<std::vector<float>>& outputs02,
	uint64_t total_iterations, uint64_t matC_total_elems) {
	static constexpr float relative_tolerance = 5e-2f;
	static constexpr float absolute_tolerance = 1e-30f;
	for (uint64_t x = 0; x < total_iterations; ++x) {
		for (uint64_t y = 0; y < matC_total_elems; ++y) {
			const float val1 = outputs01[x][y];
			const float val2 = outputs02[x][y];

			if (std::isnan(val1) && std::isnan(val2)) {
				continue;
			}
			if (std::isnan(val1) || std::isnan(val2)) {
				const float abs_diff = std::abs(val1 - val2);
				const float max_val	 = std::max(std::abs(val1), std::abs(val2));

				const bool values_match = (abs_diff <= absolute_tolerance) || (abs_diff <= relative_tolerance * max_val);
				std::cerr << rhs.operator std::string_view() << ": Value mismatch at [" << x << "," << y << "]: Reference Value: " << val1 << " vs Incorrect Value: " << val2
						  << std::endl;
				std::cerr << rhs.operator std::string_view() << ": Relative difference: " << (abs_diff / max_val) * 100.0f << "%" << std::endl;
				break;
			}
			if (std::isinf(val1) && std::isinf(val2)) {
				if ((val1 > 0) == (val2 > 0)) {
					continue;
				} else {
					const float abs_diff = std::abs(val1 - val2);
					const float max_val	 = std::max(std::abs(val1), std::abs(val2));

					const bool values_match = (abs_diff <= absolute_tolerance) || (abs_diff <= relative_tolerance * max_val);
					std::cerr << rhs.operator std::string_view() << ": Value mismatch at [" << x << "," << y << "]: Reference Value: " << val1 << " vs Incorrect Value: " << val2
							  << std::endl;
					std::cerr << rhs.operator std::string_view() << ": Relative difference: " << (abs_diff / max_val) * 100.0f << "%" << std::endl;
					break;
				}
			}
			if (std::isinf(val1) || std::isinf(val2)) {
				const float abs_diff = std::abs(val1 - val2);
				const float max_val	 = std::max(std::abs(val1), std::abs(val2));

				const bool values_match = (abs_diff <= absolute_tolerance) || (abs_diff <= relative_tolerance * max_val);
				std::cerr << rhs.operator std::string_view() << ": Value mismatch at [" << x << "," << y << "]: Reference Value: " << val1 << " vs Incorrect Value: " << val2
						  << std::endl;
				std::cerr << rhs.operator std::string_view() << ": Relative difference: " << (abs_diff / max_val) * 100.0f << "%" << std::endl;
				break;
			}

			const float abs_diff = std::abs(val1 - val2);
			const float max_val	 = std::max(std::abs(val1), std::abs(val2));

			const bool values_match = (abs_diff <= absolute_tolerance) || (abs_diff <= relative_tolerance * max_val);

			if (!values_match) {
				std::cerr << rhs.operator std::string_view() << ": Value mismatch at [" << x << "," << y << "]: Reference Value: " << val1 << " vs Incorrect Value: " << val2
						  << std::endl;
				std::cerr << rhs.operator std::string_view() << ": Relative difference: " << (abs_diff / max_val) * 100.0f << "%" << std::endl;
				break;
			}
		}
	}
}

template<uint64_t matA_dim_00, uint64_t matA_dim_01, uint64_t matB_dim_00, uint64_t matB_dim_01, mul_mat_types mul_mat_type> BNCH_SWT_INLINE void test_function() {
	static constexpr uint64_t matC_dim_00{ matA_dim_00 };
	static constexpr uint64_t matC_dim_01{ matB_dim_01 };
	static constexpr uint64_t matC_total_elems{ matC_dim_00 * matC_dim_01 };
	static constexpr uint64_t total_blocks_a{ static_cast<uint64_t>(static_cast<float>(matA_dim_00 * matA_dim_01) * static_cast<float>(sizeof(block_q8_0)) /
												  static_cast<float>(std::size(block_q8_0{}.quants))) /
		sizeof(block_q8_0) };
	static constexpr uint64_t total_floats_b{ matB_dim_00 * matB_dim_01 };
	auto blocks = generate_blocks_final<total_iterations, total_blocks_a>();
	auto floats = generate_floats_final<total_iterations, total_floats_b>();
	std::vector<std::vector<float>> outputs01{};
	std::vector<std::vector<float>> outputs02{};
	std::vector<std::vector<float>> outputs03{};
	outputs01.resize(total_iterations);
	outputs02.resize(total_iterations);
	outputs03.resize(total_iterations);
	for (uint64_t x = 0; x < total_iterations; ++x) {
		outputs01[x].resize(matC_total_elems);
		outputs02[x].resize(matC_total_elems);
		outputs03[x].resize(matC_total_elems);
	}

	static constexpr bnch_swt::string_literal stage_name{ "mul_mat: [" + bnch_swt::internal::toStringLiteral<matA_dim_00>() + "x" +
		bnch_swt::internal::toStringLiteral<matA_dim_01>() + " * " + bnch_swt::internal::toStringLiteral<matB_dim_00>() + "x" + bnch_swt::internal::toStringLiteral<matB_dim_01>() +
		"]" };
	static constexpr uint64_t total_elements_A = matA_dim_00 * matA_dim_01;
	static constexpr uint64_t total_blocks_A   = (total_elements_A + 32 - 1) / 32;
	static constexpr uint64_t blocks_size	   = total_blocks_A * sizeof(block_q8_0);
	static constexpr uint64_t floats_B_count   = matB_dim_00 * matB_dim_01;
	static constexpr uint64_t floats_B_size	   = floats_B_count * sizeof(float);
	static constexpr uint64_t outputs_C_count  = matA_dim_00 * matB_dim_01;
	static constexpr uint64_t outputs_C_size   = outputs_C_count * sizeof(float);

	uint64_t total_buffer_size = 0;
	total_buffer_size += round_up_to_multiple<64>(blocks_size);
	total_buffer_size += round_up_to_multiple<64>(floats_B_size);
	total_buffer_size += round_up_to_multiple<64>((matA_dim_00 * matA_dim_01) * sizeof(float));
	total_buffer_size += round_up_to_multiple<64>(outputs_C_size);

	cuda_buffer buffer{};
	buffer.init(total_buffer_size);

	uint64_t current_index{};
	bnch_swt::benchmark_stage<stage_name, total_iterations, measured_iterations>::template runBenchmarkWithPrep<"ggml_cuda_mul_mat",
		cuda_mul_mat_01_prep<matA_dim_00, matA_dim_01, 32>, ggml_cuda_mul_mat<matA_dim_00, matA_dim_01, 32>>(buffer, current_index, floats, blocks, outputs01, matB_dim_01);
	current_index = 0;

	//bnch_swt::benchmark_stage<stage_name, total_iterations, measured_iterations>::template runBenchmarkWithPrep<"reference_mul_mat",
	//cuda_mul_mat_01_prep<matA_dim_00, matA_dim_01, 32>, reference_mul_mat<matA_dim_00, matA_dim_01, 32>>(buffer, current_index, floats, blocks, outputs01, matB_dim_01);
	//current_index = 0;

	//bnch_swt::benchmark_stage<stage_name, total_iterations, measured_iterations>::template runBenchmarkWithPrep<"nvcuda_cublas_mul_mat",
	//		cuda_mul_mat_01_prep<matA_dim_00, matA_dim_01, 32>, nvcuda_cublas_mul_mat_simple<matA_dim_00, matA_dim_01, 32, mul_mat_type>>(buffer, current_index, floats, blocks,
	//		outputs03, matB_dim_01);
	current_index = 0;
	bnch_swt::benchmark_stage<stage_name, total_iterations, measured_iterations>::template runBenchmarkWithPrep<"rt_tm_cuda_mul_mat",
		cuda_mul_mat_01_prep<matA_dim_00, matA_dim_01, 32>, rt_tm_mul_mat<matA_dim_00, matA_dim_01, 32, mul_mat_type>>(buffer, current_index, floats, blocks, outputs02,
		matB_dim_01);

	bnch_swt::benchmark_stage<stage_name, total_iterations, measured_iterations>::printResults();
	compare_outputs<"outputs02">(outputs01, outputs02, total_iterations, matC_total_elems);
	//compare_outputs<"nvcuda_cublas_mul_mat_simple producing incorrect values">(outputs01, outputs03, total_iterations, matC_total_elems);
};

int main() {
	//test_function<14336, 4096, 4096, 16384, mul_mat_types::ffn_up>();
	//test_function<4096, 4096, 4096, 16384, mul_mat_types::q>();
	//test_function<14336, 4096, 4096, 1, mul_mat_types::ffn_up>();
	test_function<14336, 4096, 4096, 2, mul_mat_types::ffn_up>();
	test_function<14336, 4096, 4096, 4, mul_mat_types::ffn_up>();
	test_function<14336, 4096, 4096, 8, mul_mat_types::ffn_up>();

	test_function<14336, 4096, 4096, 16, mul_mat_types::ffn_up>();
	test_function<14336, 4096, 4096, 32, mul_mat_types::ffn_up>();
	test_function<14336, 4096, 4096, 64, mul_mat_types::ffn_up>();
	test_function<14336, 4096, 4096, 128, mul_mat_types::ffn_up>();
	test_function<14336, 4096, 4096, 256, mul_mat_types::ffn_up>();
	test_function<14336, 4096, 4096, 512, mul_mat_types::ffn_up>();
	test_function<14336, 4096, 4096, 1024, mul_mat_types::ffn_up>();
	test_function<14336, 4096, 4096, 2048, mul_mat_types::ffn_up>();
	test_function<14336, 4096, 4096, 4096, mul_mat_types::ffn_up>();
	test_function<14336, 4096, 4096, 8192, mul_mat_types::ffn_up>(); /*
	test_function<4096, 4096, 4096, 1, mul_mat_types::q>();
	test_function<4096, 4096, 4096, 2, mul_mat_types::q>();
	test_function<4096, 4096, 4096, 4, mul_mat_types::q>();
	test_function<4096, 4096, 4096, 8, mul_mat_types::q>();
	test_function<4096, 4096, 4096, 16, mul_mat_types::q>();
	test_function<4096, 4096, 4096, 32, mul_mat_types::q>();
	test_function<4096, 4096, 4096, 64, mul_mat_types::q>();
	test_function<4096, 4096, 4096, 128, mul_mat_types::q>();
	test_function<4096, 4096, 4096, 256, mul_mat_types::q>();
	test_function<4096, 4096, 4096, 512, mul_mat_types::q>();
	test_function<4096, 4096, 4096, 1024, mul_mat_types::q>();
	test_function<4096, 4096, 4096, 2048, mul_mat_types::q>();
	test_function<4096, 4096, 4096, 4096, mul_mat_types::q>();
	test_function<4096, 4096, 4096, 8192, mul_mat_types::q>();*/
	return 0;
}