#include "hip/hip_runtime.h"
#include <BnchSwt/BenchmarkSuite.hpp>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

static constexpr uint64_t total_iterations{ 2 };
static constexpr uint64_t measured_iterations{ 2 };

template<auto multiple, typename value_01_type = decltype(multiple)> BNCH_SWT_INLINE constexpr value_01_type round_up_to_multiple(value_01_type value) noexcept {
	if constexpr ((multiple & (multiple - 1)) == 0) {
		constexpr value_01_type mulSub1{ multiple - 1 };
		constexpr value_01_type notMulSub1{ static_cast<value_01_type>(~mulSub1) };
		return (value + (mulSub1)) & notMulSub1;
	} else {
		const value_01_type remainder = value % multiple;
		return remainder == 0 ? value : value + (multiple - remainder);
	}
}

struct cuda_buffer {
	using value_type													= uint8_t;
	using pointer														= value_type*;
	using size_type														= uint64_t;
	BNCH_SWT_INLINE cuda_buffer() noexcept								= default;
	BNCH_SWT_INLINE cuda_buffer& operator=(const cuda_buffer&) noexcept = delete;
	BNCH_SWT_INLINE cuda_buffer(const cuda_buffer&) noexcept			= delete;

	BNCH_SWT_INLINE cuda_buffer& operator=(cuda_buffer&& other) noexcept {
		if (this != &other) {
			std::swap(data_val, other.data_val);
			std::swap(size_val, other.size_val);
		}
		return *this;
	}

	BNCH_SWT_INLINE cuda_buffer(cuda_buffer&& other) noexcept {
		*this = std::move(other);
	}

	BNCH_SWT_INLINE void init(uint64_t size) {
		if (data_val) {
			clear();
		}

		hipError_t result = hipMalloc(&data_val, size);
		if (result != hipSuccess) {
			data_val = nullptr;
			throw std::runtime_error{ "cuda_buffer - failed to allocate GPU memory" };
		}

		size_val = size;
	}

	BNCH_SWT_INLINE void deinit() noexcept {
		clear();
	}

	BNCH_SWT_INLINE uint64_t size() noexcept {
		return size_val;
	}

	BNCH_SWT_INLINE void* data() noexcept {
		return data_val;
	}

	BNCH_SWT_INLINE void* claim_memory(uint64_t offset_to_claim) {
		uint64_t aligned_amount = round_up_to_multiple<64>(offset_to_claim);
		if (aligned_amount > size_val) {
			throw std::runtime_error{ "cuda_buffer - not enough memory allocated!" };
		}
		pointer return_value = data_val + aligned_amount;
		return return_value;
	}

	BNCH_SWT_INLINE ~cuda_buffer() noexcept {
		clear();
	}

  protected:
	value_type* data_val{};
	uint64_t size_val{};

	BNCH_SWT_INLINE void clear() noexcept {
		if (data_val) {
			hipError_t result = hipFree(data_val);
			data_val		   = nullptr;
			size_val		   = 0;
		}
	}
};

using q8_quant = int8_t;

struct block_q8_0 {
	q8_quant quants[32]{};
	uint16_t scale{};
};

inline static uint16_t fp32_to_fp16(float f) {
	return static_cast<uint16_t>(_mm_extract_epi16(_mm_cvtps_ph(_mm_set_ss(f), _MM_FROUND_TO_NEAREST_INT), 0));
}

inline block_q8_0 generate_block(const float* x) {
	block_q8_0 return_values{};

	float amax = 0.0f;

	for (int32_t j = 0; j < 32; j++) {
		const float v = x[j];
		amax		  = std::max(amax, fabsf(v));
	}

	const float d  = amax / ((1 << 7) - 1);
	const float id = d ? 1.0f / d : 0.0f;

	return_values.scale = fp32_to_fp16(d);

	for (int32_t j = 0; j < 32; ++j) {
		const float x0 = x[j] * id;

		return_values.quants[j] = roundf(x0);
	}
	return return_values;
}

inline std::vector<std::vector<block_q8_0>> generate_blocks(const std::vector<std::vector<float>>& floats) {
	std::vector<std::vector<block_q8_0>> result;
	result.reserve(floats.size());

	for (const auto& row: floats) {
		const uint64_t row_elements	 = row.size();
		const uint64_t blocks_needed = (row_elements + 31) / 32;

		std::vector<block_q8_0> row_blocks;
		row_blocks.reserve(blocks_needed);
		for (uint64_t x = 0; x < row_elements / 32; ++x) {
			row_blocks.emplace_back(generate_block(row.data() + x * 32));
		}

		result.emplace_back(std::move(row_blocks));
	}

	return result;
}

inline std::vector<std::vector<std::vector<block_q8_0>>> generate_blocks_final(const std::vector<std::vector<std::vector<float>>>& floats) {
	std::vector<std::vector<std::vector<block_q8_0>>> result;
	result.reserve(floats.size());

	for (const auto& values: floats) {
		result.emplace_back(generate_blocks(values));
	}

	return result;
}

inline float generate_llm_float() {
	static std::random_device rd;
	static std::mt19937 gen(rd());
	static std::normal_distribution<float> dist(0.0f, 0.02f);
	float value = dist(gen);
	return std::clamp(value, -0.5f, 0.5f);
}

template<uint64_t dim_00, uint64_t dim_01> inline std::vector<std::vector<float>> generate_floats() {
	std::vector<std::vector<float>> result;
	result.resize(dim_00);
	for (uint64_t x = 0; x < dim_00; ++x) {
		result[x].reserve(dim_01);
	}
	for (uint64_t i = 0; i < dim_00; ++i) {
		for (uint64_t j = 0; j < dim_01; ++j) {
			result[i].emplace_back(generate_llm_float());
		}
	}
	return result;
}

template<uint64_t iteration_count, uint64_t dim_00, uint64_t dim_01> inline std::vector<std::vector<std::vector<float>>> generate_floats_final() {
	std::vector<std::vector<std::vector<float>>> result;
	result.reserve(iteration_count);
	for (uint64_t x = 0; x < iteration_count; ++x) {
		result.emplace_back(generate_floats<dim_00, dim_01>());
	}
	return result;
}

template<typename value_type> inline std::vector<value_type> linearize_values(const std::vector<std::vector<value_type>>& values) {
	std::vector<value_type> return_values{};
	return_values.reserve(values.size() * values[0].size());
	for (uint64_t x = 0; x < values.size(); ++x) {
		for (uint64_t y = 0; y < values[x].size(); ++y) {
			return_values.emplace_back(values[x][y]);
		}
	}
	return return_values;
}

template<typename value_type> inline std::vector<std::vector<value_type>> transpose_values(const std::vector<std::vector<value_type>>& floats) {
	const uint64_t rows = floats.size();
	const uint64_t cols = floats.empty() ? 0 : floats[0].size();

	std::vector<std::vector<value_type>> result;
	result.resize(cols);
	for (uint64_t x = 0; x < cols; ++x) {
		result[x].reserve(rows);
	}

	for (uint64_t i = 0; i < rows; ++i) {
		for (uint64_t j = 0; j < cols; ++j) {
			result[j].emplace_back(floats[i][j]);
		}
	}
	return result;
}

template<typename value_type> inline std::vector<std::vector<std::vector<value_type>>> transpose_values_final(const std::vector<std::vector<std::vector<value_type>>>& floats) {
	std::vector<std::vector<std::vector<value_type>>> result;
	result.reserve(floats.size());
	for (uint64_t x = 0; x < floats.size(); ++x) {
		result.emplace_back(transpose_values(floats[x]));
	}
	return result;
}

template<typename value_type> inline std::vector<std::vector<value_type>> generate_values_final(const std::vector<std::vector<std::vector<value_type>>>& values) {
	std::vector<std::vector<value_type>> return_values{};
	for (uint64_t x = 0; x < values.size(); ++x) {
		return_values.emplace_back(linearize_values(values[x]));
	}
	return return_values;
}

BNCH_SWT_INLINE static constexpr float fp32_from_bits(uint32_t w) noexcept {
	return std::bit_cast<float>(w);
}

BNCH_SWT_INLINE static constexpr uint32_t fp32_to_bits(float f) noexcept {
	return std::bit_cast<uint32_t>(f);
}

BNCH_SWT_INLINE static float compute_fp16_to_fp32(half h) noexcept {
	const uint32_t w	 = static_cast<uint32_t>(h) << 16;
	const uint32_t sign	 = w & 0x80000000u;
	const uint32_t two_w = w + w;

	constexpr uint32_t exp_offset = 0xE0u << 23;
	constexpr float exp_scale	  = fp32_from_bits(0x7800000u);
	const float normalized_value  = fp32_from_bits((two_w >> 4) + exp_offset) * exp_scale;

	constexpr uint32_t magic_mask  = 126u << 23;
	constexpr float magic_bias	   = 0.5f;
	const float denormalized_value = fp32_from_bits((two_w >> 17) | magic_mask) - magic_bias;

	constexpr uint32_t denormalized_cutoff = 1u << 27;
	const uint32_t result				   = sign | (two_w < denormalized_cutoff ? fp32_to_bits(denormalized_value) : fp32_to_bits(normalized_value));
	return fp32_from_bits(result);
}

alignas(64) static float* __restrict fp16_to_fp32_array{ []() {
	alignas(64) static std::array<float, (1 << 16)> return_values_new{};
	for (uint64_t i = 0; i < (1 << 16); ++i) {
		return_values_new[i] = float{ compute_fp16_to_fp32(static_cast<uint16_t>(i)) };
	}
	return return_values_new.data();
}() };

BNCH_SWT_INLINE static float fp16_to_fp32(uint16_t f) {
	return fp16_to_fp32_array[f];
}

template<uint64_t M, uint64_t K> struct reference_mul_mat {
	BNCH_SWT_INLINE static uint64_t impl(cuda_buffer& buffer, uint64_t& current_index, std::vector<std::vector<float>>& floats, std::vector<std::vector<block_q8_0>>& blocks,
		std::vector<std::vector<float>>& outputs, uint64_t N) {
		const auto& current_blocks = blocks[current_index];
		const auto& current_floats = floats[current_index];
		auto& current_outputs	   = outputs[current_index];

		for (uint64_t row = 0; row < M; ++row) {
			for (uint64_t col = 0; col < N; ++col) {
				float sum = 0.0f;

				for (uint64_t k = 0; k < K; ++k) {
					const uint64_t block_idx	 = (row * K + k) / 32;
					const uint64_t elem_in_block = (row * K + k) % 32;

					const auto& block  = current_blocks[block_idx];
					const float scale  = __half2float(*reinterpret_cast<const __half*>(&block.scale));
					const float a_elem = scale * static_cast<float>(block.quants[elem_in_block]);

					const float b_elem = current_floats[k * N + col];

					sum += a_elem * b_elem;
				}

				current_outputs[row * N + col] = sum;
			}
		}

		return current_outputs.size() * sizeof(float);
	}
};

template<uint64_t M, uint64_t K> struct cuda_mul_mat_01_prep {
	BNCH_SWT_INLINE static uint64_t impl(cuda_buffer& buffer, uint64_t& current_index, std::vector<std::vector<float>>& floats, std::vector<std::vector<block_q8_0>>& blocks,
		std::vector<std::vector<float>>& outputs, uint64_t N) {
		constexpr uint64_t total_blocks_A = ((M * K) + 32 - 1) / 32;
		constexpr uint64_t blocks_size	  = total_blocks_A * sizeof(block_q8_0);
		const uint64_t floats_B_size	  = (K * N) * sizeof(float);
		const uint64_t outputs_C_size	  = (M * N) * sizeof(float);
		uint64_t offset					  = 0;
		block_q8_0* A_ptr_raw			  = reinterpret_cast<block_q8_0*>(static_cast<uint8_t*>(buffer.data()) + offset);
		offset							  = round_up_to_multiple<64>(offset + blocks_size);

		float* d_floats = reinterpret_cast<float*>(static_cast<uint8_t*>(buffer.data()) + offset);
		offset			= round_up_to_multiple<64>(offset + floats_B_size);

		float* d_outputs = reinterpret_cast<float*>(static_cast<uint8_t*>(buffer.data()) + offset);

		if (current_index > 0) {
			auto& previous_outputs = outputs[current_index - 1];
			hipError_t err		   = hipMemcpy(previous_outputs.data(), d_outputs, outputs_C_size, hipMemcpyDeviceToHost);
			if (err != hipSuccess) {
				std::cerr << "Failed to copy previous outputs from device: " + std::string(hipGetErrorString(err)) << std::endl;
			}
		}

		const auto& current_blocks = blocks[current_index];
		const auto& current_floats = floats[current_index];

		hipError_t err = hipMemcpy(A_ptr_raw, current_blocks.data(), blocks_size, hipMemcpyHostToDevice);
		if (err != hipSuccess) {
			std::cerr << "Failed to copy blocks to device: " + std::string(hipGetErrorString(err)) << std::endl;
		}

		err = hipMemcpy(d_floats, current_floats.data(), floats_B_size, hipMemcpyHostToDevice);
		if (err != hipSuccess) {
			std::cerr << "Failed to copy floats to device: " + std::string(hipGetErrorString(err)) << std::endl;
		}

		err = hipMemset(d_outputs, 0, outputs_C_size);
		if (err != hipSuccess) {
			std::cerr << "Failed to zero output buffer: " + std::string(hipGetErrorString(err)) << std::endl;
		}

		return 0;
	}

	BNCH_SWT_INLINE static uint64_t impl(cuda_buffer& buffer, uint64_t& current_index, std::vector<std::vector<float>>& floats_A, std::vector<std::vector<float>>& floats_B,
		std::vector<std::vector<float>>& outputs, uint64_t N) {
		const uint64_t floats_A_size  = (M * K) * sizeof(float);
		const uint64_t floats_B_size  = (K * N) * sizeof(float);
		const uint64_t outputs_C_size = (M * N) * sizeof(float);

		uint64_t offset = 0;

		float* d_floats_A = reinterpret_cast<float*>(static_cast<uint8_t*>(buffer.data()) + offset);
		offset			  = round_up_to_multiple<64>(offset + floats_A_size);

		float* d_floats_B = reinterpret_cast<float*>(static_cast<uint8_t*>(buffer.data()) + offset);
		offset			  = round_up_to_multiple<64>(offset + floats_B_size);

		float* d_outputs = reinterpret_cast<float*>(static_cast<uint8_t*>(buffer.data()) + offset);

		if (current_index > 0) {
			auto& previous_outputs = outputs[current_index - 1];
			hipError_t err		   = hipMemcpy(previous_outputs.data(), d_outputs, outputs_C_size, hipMemcpyDeviceToHost);
			if (err != hipSuccess) {
				std::cerr << "Failed to copy previous outputs from device: " + std::string(hipGetErrorString(err)) << std::endl;
			}
		}

		const auto& current_floats_A = floats_A[current_index];
		const auto& current_floats_B = floats_B[current_index];

		hipError_t err = hipMemcpy(d_floats_A, current_floats_A.data(), floats_A_size, hipMemcpyHostToDevice);
		if (err != hipSuccess) {
			std::cerr << "Failed to copy Matrix A floats to device: " + std::string(hipGetErrorString(err)) << std::endl;
		}

		err = hipMemcpy(d_floats_B, current_floats_B.data(), floats_B_size, hipMemcpyHostToDevice);
		if (err != hipSuccess) {
			std::cerr << "Failed to copy Matrix B floats to device: " + std::string(hipGetErrorString(err)) << std::endl;
		}

		// Optional: Zero output buffer (uncomment if needed)
		// err = hipMemset(d_outputs, 0, outputs_C_size);
		// if (err != hipSuccess) {
		//     std::cerr << "Failed to zero output buffer: " + std::string(hipGetErrorString(err)) << std::endl;
		// }

		return 0;
	}
};

template<uint64_t M, uint64_t K> struct cuda_mul_mat_01_prep_transposed {
	BNCH_SWT_INLINE static void transpose_blocks_to_column_major(const std::vector<block_q8_0>& src_blocks, std::vector<block_q8_0>& dst_blocks, uint64_t rows, uint64_t cols) {
		constexpr uint64_t elements_per_block = 32;
		const uint64_t blocks_per_row		  = (cols + elements_per_block - 1) / elements_per_block;
		const uint64_t blocks_per_col		  = (rows + elements_per_block - 1) / elements_per_block;

		dst_blocks.resize(src_blocks.size());

		for (uint64_t block_row = 0; block_row < blocks_per_col; ++block_row) {
			for (uint64_t block_col = 0; block_col < blocks_per_row; ++block_col) {
				uint64_t src_idx = block_row * blocks_per_row + block_col;
				uint64_t dst_idx = block_col * blocks_per_col + block_row;
				if (src_idx < src_blocks.size() && dst_idx < dst_blocks.size()) {
					dst_blocks[dst_idx] = src_blocks[src_idx];
				}
			}
		}
	}

	BNCH_SWT_INLINE static void transpose_floats_to_column_major(const std::vector<float>& src, std::vector<float>& dst, uint64_t rows, uint64_t cols) {
		dst.resize(src.size());
		for (uint64_t i = 0; i < rows; ++i) {
			for (uint64_t j = 0; j < cols; ++j) {
				dst[j * rows + i] = src[i * cols + j];
			}
		}
	}

	BNCH_SWT_INLINE static uint64_t impl(cuda_buffer& buffer, uint64_t& current_index, std::vector<std::vector<float>>& floats, std::vector<std::vector<block_q8_0>>& blocks,
		std::vector<std::vector<float>>& outputs, uint64_t N) {
		constexpr uint64_t total_blocks_A = ((M * K) + 32 - 1) / 32;
		constexpr uint64_t blocks_size	  = total_blocks_A * sizeof(block_q8_0);
		const uint64_t floats_B_size	  = (K * N) * sizeof(float);
		const uint64_t outputs_C_size	  = (M * N) * sizeof(float);
		uint64_t offset					  = 0;
		block_q8_0* A_ptr_raw			  = reinterpret_cast<block_q8_0*>(static_cast<uint8_t*>(buffer.data()) + offset);
		offset							  = round_up_to_multiple<64>(offset + blocks_size);

		float* d_floats = reinterpret_cast<float*>(static_cast<uint8_t*>(buffer.data()) + offset);
		offset			= round_up_to_multiple<64>(offset + floats_B_size);

		float* d_outputs = reinterpret_cast<float*>(static_cast<uint8_t*>(buffer.data()) + offset);

		if (current_index > 0) {
			auto& previous_outputs = outputs[current_index - 1];
			hipError_t err		   = hipMemcpy(previous_outputs.data(), d_outputs, outputs_C_size, hipMemcpyDeviceToHost);
			if (err != hipSuccess) {
				std::cerr << "Failed to copy previous outputs from device: " + std::string(hipGetErrorString(err)) << std::endl;
			}
		}

		const auto& current_blocks = blocks[current_index];
		const auto& current_floats = floats[current_index];

		std::vector<block_q8_0> transposed_blocks;
		std::vector<float> transposed_floats;

		transpose_blocks_to_column_major(current_blocks, transposed_blocks, M, K);
		transpose_floats_to_column_major(current_floats, transposed_floats, K, N);

		hipError_t err = hipMemcpy(A_ptr_raw, transposed_blocks.data(), blocks_size, hipMemcpyHostToDevice);
		if (err != hipSuccess) {
			std::cerr << "Failed to copy blocks to device: " + std::string(hipGetErrorString(err)) << std::endl;
		}

		err = hipMemcpy(d_floats, transposed_floats.data(), floats_B_size, hipMemcpyHostToDevice);
		if (err != hipSuccess) {
			std::cerr << "Failed to copy floats to device: " + std::string(hipGetErrorString(err)) << std::endl;
		}

		err = hipMemset(d_outputs, 0, outputs_C_size);
		if (err != hipSuccess) {
			std::cerr << "Failed to zero output buffer: " + std::string(hipGetErrorString(err)) << std::endl;
		}

		return 0;
	}

	BNCH_SWT_INLINE static uint64_t impl(cuda_buffer& buffer, uint64_t& current_index, std::vector<std::vector<float>>& floats_A, std::vector<std::vector<float>>& floats_B,
		std::vector<std::vector<float>>& outputs, uint64_t N) {
		const uint64_t floats_A_size  = (M * K) * sizeof(float);
		const uint64_t floats_B_size  = (K * N) * sizeof(float);
		const uint64_t outputs_C_size = (M * N) * sizeof(float);

		uint64_t offset = 0;

		float* d_floats_A = reinterpret_cast<float*>(static_cast<uint8_t*>(buffer.data()) + offset);
		offset			  = round_up_to_multiple<64>(offset + floats_A_size);

		float* d_floats_B = reinterpret_cast<float*>(static_cast<uint8_t*>(buffer.data()) + offset);
		offset			  = round_up_to_multiple<64>(offset + floats_B_size);

		float* d_outputs = reinterpret_cast<float*>(static_cast<uint8_t*>(buffer.data()) + offset);

		if (current_index > 0) {
			auto& previous_outputs = outputs[current_index - 1];
			hipError_t err		   = hipMemcpy(previous_outputs.data(), d_outputs, outputs_C_size, hipMemcpyDeviceToHost);
			if (err != hipSuccess) {
				std::cerr << "Failed to copy previous outputs from device: " + std::string(hipGetErrorString(err)) << std::endl;
			}
		}

		const auto& current_floats_A = floats_A[current_index];
		const auto& current_floats_B = floats_B[current_index];

		std::vector<float> transposed_A, transposed_B;

		transpose_floats_to_column_major(current_floats_A, transposed_A, M, K);
		transpose_floats_to_column_major(current_floats_B, transposed_B, K, N);

		hipError_t err = hipMemcpy(d_floats_A, transposed_A.data(), floats_A_size, hipMemcpyHostToDevice);
		if (err != hipSuccess) {
			std::cerr << "Failed to copy Matrix A floats to device: " + std::string(hipGetErrorString(err)) << std::endl;
		}

		err = hipMemcpy(d_floats_B, transposed_B.data(), floats_B_size, hipMemcpyHostToDevice);
		if (err != hipSuccess) {
			std::cerr << "Failed to copy Matrix B floats to device: " + std::string(hipGetErrorString(err)) << std::endl;
		}

		return 0;
	}
};

template<uint64_t M, uint64_t K> __global__ void ggml_cuda_mul_mat_kernel(const float* input01, const block_q8_0* input02, float* output, uint64_t N) {
	const uint64_t row = blockIdx.y * blockDim.y + threadIdx.y;
	const uint64_t col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row >= M || col >= N)
		return;

	float sum = 0.0f;

	const uint64_t k_end = K & ~3;

	uint64_t k = 0;
	for (; k < k_end; k += 4) {
#pragma unroll
		for (uint64_t i = 0; i < 4; ++i) {
			const uint64_t k_idx		 = k + i;
			const uint64_t linear_idx	 = row * K + k_idx;
			const uint64_t block_idx	 = linear_idx / 32;
			const uint64_t elem_in_block = linear_idx % 32;

			const block_q8_0& block = input02[block_idx];
			const float scale		= __half2float(*reinterpret_cast<const __half*>(&block.scale));
			const float a_elem		= scale * static_cast<float>(block.quants[elem_in_block]);
			const float b_elem		= input01[k_idx * N + col];

			sum += a_elem * b_elem;
		}
	}

	for (; k < K; ++k) {
		const uint64_t linear_idx	 = row * K + k;
		const uint64_t block_idx	 = linear_idx / 32;
		const uint64_t elem_in_block = linear_idx % 32;

		const block_q8_0& block = input02[block_idx];
		const float scale		= __half2float(*reinterpret_cast<const __half*>(&block.scale));
		const float a_elem		= scale * static_cast<float>(block.quants[elem_in_block]);
		const float b_elem		= input01[k * N + col];

		sum += a_elem * b_elem;
	}

	output[row * N + col] = sum;
}

template<uint64_t M, uint64_t K> __global__ void ggml_cuda_mul_mat_float_kernel(const float* input_A, const float* input_B, float* output, uint64_t N) {
	const uint64_t row = blockIdx.y * blockDim.y + threadIdx.y;
	const uint64_t col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row >= M || col >= N)
		return;

	float sum = 0.0f;

	const uint64_t k_end = K & ~3;
	uint64_t k			 = 0;

	for (; k < k_end; k += 4) {
#pragma unroll
		for (uint64_t i = 0; i < 4; ++i) {
			const uint64_t k_idx = k + i;
			const float a_elem	 = input_A[row * K + k_idx];
			const float b_elem	 = input_B[k_idx * N + col];
			sum += a_elem * b_elem;
		}
	}

	for (; k < K; ++k) {
		const float a_elem = input_A[row * K + k];
		const float b_elem = input_B[k * N + col];
		sum += a_elem * b_elem;
	}

	output[row * N + col] = sum;
}

template<uint64_t M, uint64_t K> struct ggml_cuda_mul_mat {
	BNCH_SWT_INLINE static uint64_t impl(cuda_buffer& buffer, uint64_t& current_index, std::vector<std::vector<float>>& floats, std::vector<std::vector<block_q8_0>>& blocks,
		std::vector<std::vector<float>>& outputs, uint64_t N) {
		auto& current_outputs = outputs[current_index];

		static constexpr uint64_t total_blocks_A = ((M * K) + 32 - 1) / 32;
		static constexpr uint64_t blocks_size	 = total_blocks_A * sizeof(block_q8_0);
		const uint64_t floats_B_size			 = (K * N) * sizeof(float);
		const uint64_t outputs_C_size			 = (M * N) * sizeof(float);

		uint64_t offset				= 0;
		const block_q8_0* A_ptr_raw = reinterpret_cast<const block_q8_0*>(static_cast<uint8_t*>(buffer.data()) + offset);
		offset						= round_up_to_multiple<64>(offset + blocks_size);

		const float* d_floats = reinterpret_cast<const float*>(static_cast<uint8_t*>(buffer.data()) + offset);
		offset				  = round_up_to_multiple<64>(offset + floats_B_size);

		float* d_outputs = reinterpret_cast<float*>(static_cast<uint8_t*>(buffer.data()) + offset);

		uint64_t block_dim_x, block_dim_y;
		if (N <= 4) {
			block_dim_x = N;
			block_dim_y = 256 / block_dim_x;
		} else if (M <= 16) {
			block_dim_x = 32;
			block_dim_y = 16;
		} else {
			block_dim_x = 16;
			block_dim_y = 32;
		}

		block_dim_x = std::min(block_dim_x, N);
		block_dim_y = std::min(block_dim_y, M);

		const uint64_t grid_dim_x = (N + block_dim_x - 1) / block_dim_x;
		const uint64_t grid_dim_y = (M + block_dim_y - 1) / block_dim_y;

		dim3 blockDim(static_cast<uint64_t>(block_dim_x), static_cast<uint64_t>(block_dim_y));
		dim3 gridDim(static_cast<uint64_t>(grid_dim_x), static_cast<uint64_t>(grid_dim_y));

		ggml_cuda_mul_mat_kernel<M, K><<<gridDim, blockDim>>>(d_floats, A_ptr_raw, d_outputs, N);

		hipError_t err = hipGetLastError();
		if (err != hipSuccess) {
			std::cerr << "GGML CUDA kernel launch failed: " + std::string(hipGetErrorString(err)) << std::endl;
		}

		err = hipDeviceSynchronize();
		if (err != hipSuccess) {
			std::cerr << "GGML CUDA kernel execution failed: " + std::string(hipGetErrorString(err)) << std::endl;
		}

		++current_index;
		return current_outputs.size() * sizeof(float);
	}

	BNCH_SWT_INLINE static uint64_t impl(cuda_buffer& buffer, uint64_t& current_index, std::vector<std::vector<float>>& floats_A, std::vector<std::vector<float>>& floats_B,
		std::vector<std::vector<float>>& outputs, uint64_t N) {
		auto& current_outputs = outputs[current_index];

		const uint64_t floats_A_size  = (M * K) * sizeof(float);
		const uint64_t floats_B_size  = (K * N) * sizeof(float);
		const uint64_t outputs_C_size = (M * N) * sizeof(float);

		uint64_t offset = 0;

		const float* d_floats_A = reinterpret_cast<const float*>(static_cast<uint8_t*>(buffer.data()) + offset);
		offset					= round_up_to_multiple<64>(offset + floats_A_size);

		const float* d_floats_B = reinterpret_cast<const float*>(static_cast<uint8_t*>(buffer.data()) + offset);
		offset					= round_up_to_multiple<64>(offset + floats_B_size);

		float* d_outputs = reinterpret_cast<float*>(static_cast<uint8_t*>(buffer.data()) + offset);

		uint64_t block_dim_x, block_dim_y;
		if (N <= 4) {
			block_dim_x = N;
			block_dim_y = 256 / block_dim_x;
		} else if (M <= 16) {
			block_dim_x = 32;
			block_dim_y = 16;
		} else {
			block_dim_x = 16;
			block_dim_y = 32;
		}

		block_dim_x = std::min(block_dim_x, N);
		block_dim_y = std::min(block_dim_y, M);

		const uint64_t grid_dim_x = (N + block_dim_x - 1) / block_dim_x;
		const uint64_t grid_dim_y = (M + block_dim_y - 1) / block_dim_y;

		dim3 blockDim(static_cast<uint64_t>(block_dim_x), static_cast<uint64_t>(block_dim_y));
		dim3 gridDim(static_cast<uint64_t>(grid_dim_x), static_cast<uint64_t>(grid_dim_y));

		ggml_cuda_mul_mat_float_kernel<M, K><<<gridDim, blockDim>>>(d_floats_A, d_floats_B, d_outputs, N);

		hipError_t err = hipGetLastError();
		if (err != hipSuccess) {
			std::cerr << "GGML CUDA float kernel launch failed: " + std::string(hipGetErrorString(err)) << std::endl;
		}

		err = hipDeviceSynchronize();
		if (err != hipSuccess) {
			std::cerr << "GGML CUDA float kernel execution failed: " + std::string(hipGetErrorString(err)) << std::endl;
		}

		++current_index;
		return current_outputs.size() * sizeof(float);
	}
};

#include <hipblas.h>
#include <hipblaslt.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

__global__ void cutlass_dequantize_q8_vectorized_kernel(const block_q8_0* input_blocks, float* output, uint64_t total_elements) {
	const uint64_t idx	  = blockIdx.x * blockDim.x + threadIdx.x;
	const uint64_t stride = blockDim.x * gridDim.x;

	for (uint64_t i = idx; i < total_elements; i += stride) {
		const uint64_t block_idx	 = i / 32;
		const uint64_t elem_in_block = i % 32;

		const block_q8_0& block = input_blocks[block_idx];
		const float scale		= __half2float(*reinterpret_cast<const __half*>(&block.scale));
		output[i]				= scale * static_cast<float>(block.quants[elem_in_block]);
	}
}

template<uint64_t M, uint64_t K> struct cublas_mul_mat {
	BNCH_SWT_INLINE static uint64_t impl(cuda_buffer& buffer, uint64_t& current_index, std::vector<std::vector<float>>& floats, std::vector<std::vector<block_q8_0>>& blocks,
		std::vector<std::vector<float>>& outputs, uint64_t N) {
		auto& current_outputs = outputs[current_index];

		static constexpr uint64_t total_blocks_A = ((M * K) + 32 - 1) / 32;
		static constexpr uint64_t blocks_size	 = total_blocks_A * sizeof(block_q8_0);
		const uint64_t floats_A_size			 = (M * K) * sizeof(float);
		const uint64_t floats_B_size			 = (K * N) * sizeof(float);
		const uint64_t outputs_C_size			 = (M * N) * sizeof(float);

		uint64_t offset				   = 0;
		const block_q8_0* A_blocks_raw = reinterpret_cast<const block_q8_0*>(static_cast<uint8_t*>(buffer.data()) + offset);
		offset						   = round_up_to_multiple<64>(offset + blocks_size);

		float* d_floats_A = reinterpret_cast<float*>(static_cast<uint8_t*>(buffer.data()) + offset);
		offset			  = round_up_to_multiple<64>(offset + floats_A_size);

		const float* d_floats_B = reinterpret_cast<const float*>(static_cast<uint8_t*>(buffer.data()) + offset);
		offset					= round_up_to_multiple<64>(offset + floats_B_size);

		float* d_outputs = reinterpret_cast<float*>(static_cast<uint8_t*>(buffer.data()) + offset);

		const uint64_t total_elements_A	 = M * K;
		const uint64_t threads_per_block = 256;
		const uint64_t blocks_needed	 = (total_elements_A + threads_per_block - 1) / threads_per_block;

		cutlass_dequantize_q8_vectorized_kernel<<<blocks_needed, threads_per_block>>>(A_blocks_raw, d_floats_A, total_elements_A);

		hipError_t err = hipDeviceSynchronize();
		if (err != hipSuccess) {
			std::cerr << "Dequantization kernel execution failed: " + std::string(hipGetErrorString(err)) << std::endl;
		}

		hipblasHandle_t handle;
		hipblasStatus_t status = hipblasCreate(&handle);
		if (status != HIPBLAS_STATUS_SUCCESS) {
			std::cerr << "cuBLAS initialization failed: " << status << std::endl;
			return 0;
		}

		const float alpha = 1.0f;
		const float beta  = 0.0f;

		status = hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, static_cast<int>(N), static_cast<int>(M), static_cast<int>(K), &alpha, d_floats_B, static_cast<int>(K), d_floats_A,
			static_cast<int>(M), &beta, d_outputs, static_cast<int>(N));

		if (status != HIPBLAS_STATUS_SUCCESS) {
			std::cerr << "cuBLAS SGEMM failed: " << status << std::endl;
		}

		hipblasDestroy(handle);

		err = hipDeviceSynchronize();
		if (err != hipSuccess) {
			std::cerr << "CUDA synchronization failed: " + std::string(hipGetErrorString(err)) << std::endl;
		}

		++current_index;
		return current_outputs.size() * sizeof(float);
	}

	BNCH_SWT_INLINE static uint64_t impl(cuda_buffer& buffer, uint64_t& current_index, std::vector<std::vector<float>>& floats_A, std::vector<std::vector<float>>& floats_B,
		std::vector<std::vector<float>>& outputs, uint64_t N) {
		auto& current_outputs = outputs[current_index];

		const uint64_t floats_A_size  = (M * K) * sizeof(float);
		const uint64_t floats_B_size  = (K * N) * sizeof(float);
		const uint64_t outputs_C_size = (M * N) * sizeof(float);

		uint64_t offset = 0;

		const float* d_floats_A = reinterpret_cast<const float*>(static_cast<uint8_t*>(buffer.data()) + offset);
		offset					= round_up_to_multiple<64>(offset + floats_A_size);

		const float* d_floats_B = reinterpret_cast<const float*>(static_cast<uint8_t*>(buffer.data()) + offset);
		offset					= round_up_to_multiple<64>(offset + floats_B_size);

		float* d_outputs = reinterpret_cast<float*>(static_cast<uint8_t*>(buffer.data()) + offset);

		hipblasHandle_t handle;
		hipblasStatus_t status = hipblasCreate(&handle);
		if (status != HIPBLAS_STATUS_SUCCESS) {
			std::cerr << "cuBLAS initialization failed: " << status << std::endl;
			return 0;
		}

		const float alpha = 1.0f;
		const float beta  = 0.0f;

		status = hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, static_cast<int>(N), static_cast<int>(M), static_cast<int>(K), &alpha, d_floats_B, static_cast<int>(K), d_floats_A,
			static_cast<int>(M), &beta, d_outputs, static_cast<int>(N));

		if (status != HIPBLAS_STATUS_SUCCESS) {
			std::cerr << "cuBLAS SGEMM failed: " << status << std::endl;
		}

		hipblasDestroy(handle);

		hipError_t err = hipDeviceSynchronize();
		if (err != hipSuccess) {
			std::cerr << "CUDA synchronization failed: " + std::string(hipGetErrorString(err)) << std::endl;
		}

		++current_index;
		return current_outputs.size() * sizeof(float);
	}
};

enum class kernel_type_profiles : uint8_t {
	fp16_mha,
	fp16_moe,
	bf16_mha,
	bf16_gqa,
	q4_mha,
	q4_gqa,
	q4_moe,
	q8_mha,
	q8_gqa,
	q8_moe,
	mixed_fp16_fp32,
	mixed_bf16_fp32,
	count,
};

enum class mul_mat_types {
	q,
	k,
	v,
	kq,
	kqv,
	kqv_out,
	ffn_gate,
	ffn_up,
	ffn_out,
};

#include <hipblas.h>
#include <hipblaslt.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

template<typename value_type> using base_type = std::remove_cvref_t<value_type>;

template<typename value_type> using x_type = decltype(base_type<value_type>::x);

template<typename value_type>
concept uint_cuda_types = std::is_unsigned_v<x_type<value_type>> && std::is_integral_v<x_type<value_type>>;

template<typename value_type>
concept int_cuda_types = std::is_signed_v<x_type<value_type>> && std::is_integral_v<x_type<value_type>> && !uint_cuda_types<value_type>;

template<typename value_type>
concept int8_cuda_types = int_cuda_types<x_type<value_type>> && sizeof(x_type<value_type>) == 1;

template<typename value_type>
concept int16_cuda_types = int_cuda_types<x_type<value_type>> && sizeof(x_type<value_type>) == 2;

template<typename value_type>
concept int32_cuda_types = int_cuda_types<x_type<value_type>> && sizeof(x_type<value_type>) == 4;

template<typename value_type>
concept int64_cuda_types = int_cuda_types<x_type<value_type>> && sizeof(x_type<value_type>) == 8;

template<typename value_type>
concept uint8_cuda_types = uint_cuda_types<x_type<value_type>> && sizeof(x_type<value_type>) == 1;

template<typename value_type>
concept uint16_cuda_types = uint_cuda_types<x_type<value_type>> && sizeof(x_type<value_type>) == 2;

template<typename value_type>
concept uint32_cuda_types = uint_cuda_types<x_type<value_type>> && sizeof(x_type<value_type>) == 4;

template<typename value_type>
concept uint64_cuda_types = uint_cuda_types<x_type<value_type>> && sizeof(x_type<value_type>) == 8;

template<typename value_type>
concept float_cuda_types = std::floating_point<x_type<value_type>>;

template<typename value_type>
concept float32_cuda_types = float_cuda_types<value_type> && sizeof(x_type<value_type>) == 4;

template<typename value_type>
concept float64_cuda_types = float_cuda_types<value_type> && sizeof(x_type<value_type>) == 8;

template<typename value_type>
concept r_value_reference_types = std::is_rvalue_reference_v<value_type>;

template<typename value_type> BNCH_SWT_INLINE __device__ constexpr value_type&& device_forward(value_type& arg) noexcept {
	return static_cast<value_type&&>(arg);
}

template<r_value_reference_types value_type> __device__ BNCH_SWT_INLINE constexpr value_type device_forward(value_type arg) noexcept {
	return arg;
}

enum class get_value_type_errors {
	invalid_type,
};

template<typename value_type>
concept dim04_types = requires() { base_type<value_type>::w; };

template<typename value_type>
concept dim03_types = requires() { base_type<value_type>::z; } && !dim04_types<value_type>;

template<typename value_type>
concept dim02_types = requires() { base_type<value_type>::y; } && !dim03_types<value_type> && !dim04_types<value_type>;

template<typename value_type>
concept dim01_types = requires() { base_type<value_type>::x; } && !dim02_types<value_type> && !dim03_types<value_type> && !dim04_types<value_type>;

template<typename value_type>
concept dim_types = requires() { base_type<value_type>::x; };

template<typename value_type> struct get_value_type {
	template<typename... value_types> BNCH_SWT_INLINE __device__ static constexpr decltype(auto) impl(value_types&&... args) = delete;
};

template<int8_cuda_types value_type> struct get_value_type<value_type> {
	template<typename... value_types> BNCH_SWT_INLINE __device__ static constexpr decltype(auto) impl(value_types&&... args) {
		if constexpr (dim01_types<value_type>) {
			return make_char1(device_forward<value_types>(args)...);
		} else if constexpr (dim02_types<value_type>) {
			return make_char2(device_forward<value_types>(args)...);
		} else if constexpr (dim03_types<value_type>) {
			return make_char3(device_forward<value_types>(args)...);
		} else if constexpr (dim04_types<value_type>) {
			return make_char4(device_forward<value_types>(args)...);
		}
	}
};

template<int16_cuda_types value_type> struct get_value_type<value_type> {
	template<typename... value_types> BNCH_SWT_INLINE __device__ static constexpr decltype(auto) impl(value_types&&... args) {
		if constexpr (dim01_types<value_type>) {
			return make_short1(device_forward<value_types>(args)...);
		} else if constexpr (dim02_types<value_type>) {
			return make_short2(device_forward<value_types>(args)...);
		} else if constexpr (dim03_types<value_type>) {
			return make_short3(device_forward<value_types>(args)...);
		} else if constexpr (dim04_types<value_type>) {
			return make_short4(device_forward<value_types>(args)...);
		}
	}
};

template<int32_cuda_types value_type> struct get_value_type<value_type> {
	template<typename... value_types> BNCH_SWT_INLINE __device__ static constexpr decltype(auto) impl(value_types&&... args) {
		if constexpr (dim01_types<value_type>) {
			return make_int1(device_forward<value_types>(args)...);
		} else if constexpr (dim02_types<value_type>) {
			return make_int2(device_forward<value_types>(args)...);
		} else if constexpr (dim03_types<value_type>) {
			return make_int3(device_forward<value_types>(args)...);
		} else if constexpr (dim04_types<value_type>) {
			return make_int4(device_forward<value_types>(args)...);
		}
	}
};

template<int64_cuda_types value_type> struct get_value_type<value_type> {
	template<typename... value_types> BNCH_SWT_INLINE __device__ static constexpr decltype(auto) impl(value_types&&... args) {
		if constexpr (dim01_types<value_type>) {
			return make_long1(device_forward<value_types>(args)...);
		} else if constexpr (dim02_types<value_type>) {
			return make_long2(device_forward<value_types>(args)...);
		} else if constexpr (dim03_types<value_type>) {
			return make_long3(device_forward<value_types>(args)...);
		} else if constexpr (dim04_types<value_type>) {
			return make_long4(device_forward<value_types>(args)...);
		}
	}
};

template<uint8_cuda_types value_type> struct get_value_type<value_type> {
	template<typename... value_types> BNCH_SWT_INLINE __device__ static constexpr decltype(auto) impl(value_types&&... args) {
		if constexpr (dim01_types<value_type>) {
			return make_uchar1(device_forward<value_types>(args)...);
		} else if constexpr (dim02_types<value_type>) {
			return make_uchar2(device_forward<value_types>(args)...);
		} else if constexpr (dim03_types<value_type>) {
			return make_uchar3(device_forward<value_types>(args)...);
		} else if constexpr (dim04_types<value_type>) {
			return make_uchar4(device_forward<value_types>(args)...);
		}
	}
};

template<uint16_cuda_types value_type> struct get_value_type<value_type> {
	template<typename... value_types> BNCH_SWT_INLINE __device__ static constexpr decltype(auto) impl(value_types&&... args) {
		if constexpr (dim01_types<value_type>) {
			return make_ushort1(device_forward<value_types>(args)...);
		} else if constexpr (dim02_types<value_type>) {
			return make_ushort2(device_forward<value_types>(args)...);
		} else if constexpr (dim03_types<value_type>) {
			return make_ushort3(device_forward<value_types>(args)...);
		} else if constexpr (dim04_types<value_type>) {
			return make_ushort4(device_forward<value_types>(args)...);
		}
	}
};

template<uint32_cuda_types value_type> struct get_value_type<value_type> {
	template<typename... value_types> BNCH_SWT_INLINE __device__ static constexpr decltype(auto) impl(value_types&&... args) {
		if constexpr (dim01_types<value_type>) {
			return make_uint1(device_forward<value_types>(args)...);
		} else if constexpr (dim02_types<value_type>) {
			return make_uint2(device_forward<value_types>(args)...);
		} else if constexpr (dim03_types<value_type>) {
			return make_uint3(device_forward<value_types>(args)...);
		} else if constexpr (dim04_types<value_type>) {
			return make_uint4(device_forward<value_types>(args)...);
		}
	}
};

template<uint64_cuda_types value_type> struct get_value_type<value_type> {
	template<typename... value_types> BNCH_SWT_INLINE __device__ static constexpr decltype(auto) impl(value_types&&... args) {
		if constexpr (dim01_types<value_type>) {
			return make_ulong1(device_forward<value_types>(args)...);
		} else if constexpr (dim02_types<value_type>) {
			return make_ulong2(device_forward<value_types>(args)...);
		} else if constexpr (dim03_types<value_type>) {
			return make_ulong3(device_forward<value_types>(args)...);
		} else if constexpr (dim04_types<value_type>) {
			return make_ulong4(device_forward<value_types>(args)...);
		}
	}
};

template<float32_cuda_types value_type> struct get_value_type<value_type> {
	template<typename... value_types> BNCH_SWT_INLINE __device__ static constexpr decltype(auto) impl(value_types&&... args) {
		if constexpr (dim01_types<value_type>) {
			return make_float1(device_forward<value_types>(args)...);
		} else if constexpr (dim02_types<value_type>) {
			return make_float2(device_forward<value_types>(args)...);
		} else if constexpr (dim03_types<value_type>) {
			return make_float3(device_forward<value_types>(args)...);
		} else if constexpr (dim04_types<value_type>) {
			return make_float4(device_forward<value_types>(args)...);
		}
	}
};

template<float64_cuda_types value_type> struct get_value_type<value_type> {
	template<typename... value_types> BNCH_SWT_INLINE __device__ static constexpr decltype(auto) impl(value_types&&... args) {
		if constexpr (dim01_types<value_type>) {
			return make_double1(device_forward<value_types>(args)...);
		} else if constexpr (dim02_types<value_type>) {
			return make_double2(device_forward<value_types>(args)...);
		} else if constexpr (dim03_types<value_type>) {
			return make_double3(device_forward<value_types>(args)...);
		} else if constexpr (dim04_types<value_type>) {
			return make_double4(device_forward<value_types>(args)...);
		}
	}
};

enum class binary_op_types {
	add,
	mul,
	sub,
	div,
};

template<binary_op_types> struct binary_op_core;

template<> struct binary_op_core<binary_op_types::add> {
	template<typename value_type01, typename value_type02> BNCH_SWT_INLINE static __device__ decltype(auto) impl(value_type01&& val01, value_type02&& val02) {
		return device_forward<value_type01>(val01) + static_cast<base_type<value_type01>>(device_forward<value_type02>(val02));
	}

	template<typename value_type01, typename value_type02> BNCH_SWT_INLINE static __device__ void impl_in_place(value_type01& val01, value_type02&& val02) {
		val01 += static_cast<base_type<value_type01>>(device_forward<value_type02>(val02));
	}
};

template<> struct binary_op_core<binary_op_types::mul> {
	template<typename value_type01, typename value_type02> BNCH_SWT_INLINE static __device__ decltype(auto) impl(value_type01&& val01, value_type02&& val02) {
		return device_forward<value_type01>(val01) * static_cast<base_type<value_type01>>(device_forward<value_type02>(val02));
	}

	template<typename value_type01, typename value_type02> BNCH_SWT_INLINE static __device__ void impl_in_place(value_type01& val01, value_type02&& val02) {
		val01 *= static_cast<base_type<value_type01>>(device_forward<value_type02>(val02));
	}
};

template<> struct binary_op_core<binary_op_types::sub> {
	template<typename value_type01, typename value_type02> BNCH_SWT_INLINE static __device__ decltype(auto) impl(value_type01&& val01, value_type02&& val02) {
		return device_forward<value_type01>(val01) - static_cast<base_type<value_type01>>(device_forward<value_type02>(val02));
	}

	template<typename value_type01, typename value_type02> BNCH_SWT_INLINE static __device__ void impl_in_place(value_type01& val01, value_type02&& val02) {
		val01 -= static_cast<base_type<value_type01>>(device_forward<value_type02>(val02));
	}
};

template<> struct binary_op_core<binary_op_types::div> {
	template<typename value_type01, typename value_type02> BNCH_SWT_INLINE static __device__ decltype(auto) impl(value_type01&& val01, value_type02&& val02) {
		return device_forward<value_type01>(val01) / static_cast<base_type<value_type01>>(device_forward<value_type02>(val02));
	}

	template<typename value_type01, typename value_type02> BNCH_SWT_INLINE static __device__ void impl_in_place(value_type01& val01, value_type02&& val02) {
		val01 /= static_cast<base_type<value_type01>>(device_forward<value_type02>(val02));
	}
};

template<typename value_type, binary_op_types binary_op_type> struct binary_op_base;

template<dim01_types value_type, binary_op_types binary_op_type> struct binary_op_base<value_type, binary_op_type> {
	template<typename value_type01, typename value_type02> BNCH_SWT_INLINE static __device__ decltype(auto) impl(value_type01&& val01, value_type02&& val02) {
		using op_core_type = binary_op_core<binary_op_type>;
		return get_value_type<value_type01>::impl(op_core_type::impl(device_forward<value_type01>(val01).x, device_forward<value_type02>(val02).x));
	}

	template<typename value_type01, typename value_type02> BNCH_SWT_INLINE static __device__ void impl_in_place(value_type01& val01, value_type02&& val02) {
		using op_core_type = binary_op_core<binary_op_type>;
		op_core_type::impl_in_place(val01.x, device_forward<value_type02>(val02).x);
	}
};

template<dim02_types value_type, binary_op_types binary_op_type> struct binary_op_base<value_type, binary_op_type> {
	template<typename value_type01, typename value_type02> BNCH_SWT_INLINE static __device__ decltype(auto) impl(value_type01&& val01, value_type02&& val02) {
		using op_core_type = binary_op_core<binary_op_type>;
		return get_value_type<value_type01>::impl(op_core_type::impl(device_forward<value_type01>(val01).x, device_forward<value_type02>(val02).x),
			op_core_type::impl(device_forward<value_type01>(val01).y, device_forward<value_type02>(val02).y));
	}

	template<typename value_type01, typename value_type02> BNCH_SWT_INLINE static __device__ void impl_in_place(value_type01& val01, value_type02&& val02) {
		using op_core_type = binary_op_core<binary_op_type>;
		op_core_type::impl_in_place(val01.x, device_forward<value_type02>(val02).x);
		op_core_type::impl_in_place(val01.y, device_forward<value_type02>(val02).y);
	}
};

template<dim03_types value_type, binary_op_types binary_op_type> struct binary_op_base<value_type, binary_op_type> {
	template<typename value_type01, typename value_type02> BNCH_SWT_INLINE static __device__ decltype(auto) impl(value_type01&& val01, value_type02&& val02) {
		using op_core_type = binary_op_core<binary_op_type>;
		return get_value_type<value_type01>::impl(op_core_type::impl(device_forward<value_type01>(val01).x, device_forward<value_type02>(val02).x),
			op_core_type::impl(device_forward<value_type01>(val01).y, device_forward<value_type02>(val02).y),
			op_core_type::impl(device_forward<value_type01>(val01).z, device_forward<value_type02>(val02).z));
	}

	template<typename value_type01, typename value_type02> BNCH_SWT_INLINE static __device__ void impl_in_place(value_type01& val01, value_type02&& val02) {
		using op_core_type = binary_op_core<binary_op_type>;
		op_core_type::impl_in_place(val01.x, device_forward<value_type02>(val02).x);
		op_core_type::impl_in_place(val01.y, device_forward<value_type02>(val02).y);
		op_core_type::impl_in_place(val01.z, device_forward<value_type02>(val02).z);
	}
};

template<dim04_types value_type, binary_op_types binary_op_type> struct binary_op_base<value_type, binary_op_type> {
	template<typename value_type01, typename value_type02> BNCH_SWT_INLINE static __device__ decltype(auto) impl(value_type01&& val01, value_type02&& val02) {
		using op_core_type = binary_op_core<binary_op_type>;
		return get_value_type<value_type01>::impl(op_core_type::impl(device_forward<value_type01>(val01).x, device_forward<value_type02>(val02).x),
			op_core_type::impl(device_forward<value_type01>(val01).y, device_forward<value_type02>(val02).y),
			op_core_type::impl(device_forward<value_type01>(val01).z, device_forward<value_type02>(val02).z),
			op_core_type::impl(device_forward<value_type01>(val01).w, device_forward<value_type02>(val02).w));
	}

	template<typename value_type01, typename value_type02> BNCH_SWT_INLINE static __device__ void impl_in_place(value_type01& val01, value_type02&& val02) {
		using op_core_type = binary_op_core<binary_op_type>;
		op_core_type::impl_in_place(val01.x, device_forward<value_type02>(val02).x);
		op_core_type::impl_in_place(val01.y, device_forward<value_type02>(val02).y);
		op_core_type::impl_in_place(val01.z, device_forward<value_type02>(val02).z);
		op_core_type::impl_in_place(val01.w, device_forward<value_type02>(val02).w);
	}
};

template<binary_op_types binary_op_type> struct binary_op {
	template<typename value_type01, typename value_type02> BNCH_SWT_INLINE static __device__ decltype(auto) impl(value_type01&& val01, value_type02&& val02) {
		return binary_op_base<value_type01, binary_op_type>::impl(device_forward<value_type01>(val01), device_forward<value_type02>(val02));
	}

	template<typename value_type01, typename value_type02> BNCH_SWT_INLINE static __device__ decltype(auto) impl_in_place(value_type01& val01, value_type02&& val02) {
		return binary_op_base<value_type01, binary_op_type>::impl_in_place(val01, device_forward<value_type02>(val02));
	}
};

template<dim_types value_type01, dim_types value_type02> BNCH_SWT_INLINE __device__ decltype(auto) operator+=(value_type01& val01, value_type02&& val02) {
	return binary_op<binary_op_types::add>::impl_in_place(val01, device_forward<value_type02>(val02));
}

template<dim_types value_type01, dim_types value_type02> BNCH_SWT_INLINE __device__ decltype(auto) operator+(value_type01&& val01, value_type02&& val02) {
	return binary_op<binary_op_types::add>::impl(device_forward<value_type01>(val01), device_forward<value_type02>(val02));
}

template<dim_types value_type01, dim_types value_type02> BNCH_SWT_INLINE __device__ decltype(auto) operator*=(value_type01& val01, value_type02&& val02) {
	return binary_op<binary_op_types::mul>::impl_in_place(val01, device_forward<value_type02>(val02));
}

template<dim_types value_type01, dim_types value_type02> BNCH_SWT_INLINE __device__ decltype(auto) operator*(value_type01&& val01, value_type02&& val02) {
	return binary_op<binary_op_types::mul>::impl(device_forward<value_type01>(val01), device_forward<value_type02>(val02));
}

template<dim_types value_type01, dim_types value_type02> BNCH_SWT_INLINE __device__ decltype(auto) operator-=(value_type01& val01, value_type02&& val02) {
	return binary_op<binary_op_types::sub>::impl_in_place(val01, device_forward<value_type02>(val02));
}

template<dim_types value_type01, dim_types value_type02> BNCH_SWT_INLINE __device__ decltype(auto) operator-(value_type01&& val01, value_type02&& val02) {
	return binary_op<binary_op_types::sub>::impl(device_forward<value_type01>(val01), device_forward<value_type02>(val02));
}

template<dim_types value_type01, dim_types value_type02> BNCH_SWT_INLINE __device__ decltype(auto) operator/=(value_type01& val01, value_type02&& val02) {
	return binary_op<binary_op_types::div>::impl_in_place(val01, device_forward<value_type02>(val02));
}

template<dim_types value_type01, dim_types value_type02> BNCH_SWT_INLINE __device__ decltype(auto) operator/(value_type01&& val01, value_type02&& val02) {
	return binary_op<binary_op_types::div>::impl(device_forward<value_type01>(val01), device_forward<value_type02>(val02));
}

struct gpu_properties {
	static constexpr uint64_t sm_count{ 70ull };
	static constexpr uint64_t max_threads_per_sm{ 1536ull };
	static constexpr uint64_t max_threads_per_block{ 1024ull };
	static constexpr uint64_t warp_size{ 32ull };
	static constexpr uint64_t l2_cache_size{ 50331648ull };
	static constexpr uint64_t shared_mem_per_block{ 49152ull };
	static constexpr uint64_t memory_bus_width{ 256ull };
	static constexpr uint64_t memory_clock_rate{ 14001000ull };
	static constexpr uint64_t major_compute_capability{ 12ull };
	static constexpr uint64_t minor_compute_capability{ 0ull };
	static constexpr uint64_t max_grid_size_x{ 2147483647ull };
	static constexpr uint64_t gpu_arch_index{ 4ull };
	static constexpr uint64_t total_threads{ 107520ull };
	static constexpr uint64_t optimal_block_size{ 512ull };
	static constexpr uint64_t optimal_grid_size{ 210ull };
};

template<uint64_t block_m, uint64_t block_n, uint64_t block_k, uint64_t warp_m_new, uint64_t warp_n_new, uint64_t thread_m_new, uint64_t thread_n_new> struct cuda_kernel_traits {
	static constexpr uint64_t block_tile_m		= block_m;
	static constexpr uint64_t block_tile_n		= block_n;
	static constexpr uint64_t block_tile_k		= block_k;
	static constexpr uint64_t warp_tile_m		= warp_m_new;
	static constexpr uint64_t warp_tile_n		= warp_n_new;
	static constexpr uint64_t thread_tile_m		= thread_m_new;
	static constexpr uint64_t thread_tile_n		= thread_n_new;
	static constexpr uint64_t warps_m			= block_m / warp_m_new;
	static constexpr uint64_t warps_n			= block_n / warp_n_new;
	static constexpr uint64_t threads_per_warp	= gpu_properties::warp_size;
	static constexpr uint64_t threads_per_block = warps_m * warps_n * threads_per_warp;

	static_assert(block_m > 0, "block_m must be greater than 0");
	static_assert(block_n > 0, "block_n must be greater than 0");
	static_assert(block_k > 0, "block_k must be greater than 0");
	static_assert(warp_m_new > 0, "warp_m must be greater than 0");
	static_assert(warp_n_new > 0, "warp_n must be greater than 0");
	static_assert(thread_m_new > 0, "thread_m must be greater than 0");
	static_assert(thread_n_new > 0, "thread_n must be greater than 0");

	static_assert(block_m % warp_m_new == 0, "block_m must be evenly divisible by warp_m");
	static_assert(block_n % warp_n_new == 0, "block_n must be evenly divisible by warp_n");

	static_assert(warp_m_new % thread_m_new == 0, "warp_m must be evenly divisible by thread_m");
	static_assert(warp_n_new % thread_n_new == 0, "warp_n must be evenly divisible by thread_n");

	static_assert((warp_m_new / thread_m_new) * (warp_n_new / thread_n_new) == gpu_properties::warp_size, "Warp configuration must result in exactly warp_size threads per warp");

	static_assert(threads_per_block <= gpu_properties::max_threads_per_block, "threads_per_block cannot exceed max_threads_per_block");
	static_assert(threads_per_block >= gpu_properties::warp_size, "threads_per_block must be at least warp_size");

	static_assert(block_m <= 512, "block_m should not exceed 512 for reasonable shared memory usage");
	static_assert(block_n <= 512, "block_n should not exceed 512 for reasonable shared memory usage");
	static_assert(block_k <= 64, "block_k should not exceed 64 for reasonable register usage");

	static_assert(block_k % 4 == 0, "block_k should be a multiple of 4 for vectorized loads");

	static_assert(thread_m_new <= 8, "thread_m should not exceed 8 for reasonable register usage");
	static_assert(thread_n_new <= 8, "thread_n should not exceed 8 for reasonable register usage");

	static_assert(warps_m > 0 && warps_n > 0, "Must have at least one warp in each dimension");
	static_assert(warps_m * warps_n <= 32, "Total warps per block should not exceed 32");

	static constexpr uint64_t shared_mem_usage = 2 * (block_m * block_k + block_k * block_n) * sizeof(float);
	static_assert(shared_mem_usage <= gpu_properties::shared_mem_per_block, "Estimated shared memory usage exceeds shared_mem_per_block limit");

	static_assert(threads_per_block % gpu_properties::warp_size == 0, "threads_per_block must be a multiple of warp_size");
};

template<uint64_t M, uint64_t K, typename traits>
__device__ __forceinline__ void load_smem_tile_A(float* smem_A, const block_q8_0* A_global, uint64_t N, uint64_t k_offset, uint64_t block_row) {
	constexpr uint64_t block_m			 = traits::block_tile_m;
	constexpr uint64_t block_k			 = traits::block_tile_k;
	constexpr uint64_t threads_per_block = traits::threads_per_block;
	const uint64_t tid					 = threadIdx.x;
	const uint64_t k_blocks				 = (K + 31) / 32;
	const uint64_t elements_per_block	 = block_m * block_k;
	const uint64_t vec4_elements		 = elements_per_block / 4;
	const uint64_t vec4_per_thread		 = (vec4_elements + threads_per_block - 1) / threads_per_block;
	for (uint64_t i = 0; i < vec4_per_thread; ++i) {
		const uint64_t vec4_idx							 = tid + i * threads_per_block;
		const uint64_t linear_idx						 = vec4_idx * 4;
		const uint64_t row								 = linear_idx / block_k;
		const uint64_t col								 = linear_idx % block_k;
		const uint64_t global_row						 = block_row + row;
		const uint64_t global_col						 = k_offset + col;
		const uint64_t q8_block_row						 = global_row;
		const uint64_t q8_block_col						 = global_col / 32;
		const uint64_t q8_elem_idx						 = global_col % 32;
		const uint64_t q8_block_idx						 = q8_block_row * k_blocks + q8_block_col;
		const block_q8_0& q8_block						 = A_global[q8_block_idx];
		const float scale_raw							 = __half2float(*reinterpret_cast<const __half*>(&q8_block.scale));
		const uint64_t smem_offset						 = row * block_k + col;
		*reinterpret_cast<float4*>(&smem_A[smem_offset]) = make_float4(static_cast<float>(q8_block.quants[q8_elem_idx]), static_cast<float>(q8_block.quants[q8_elem_idx + 1]),
															   static_cast<float>(q8_block.quants[q8_elem_idx + 2]), static_cast<float>(q8_block.quants[q8_elem_idx + 3])) *
			make_float4(scale_raw, scale_raw, scale_raw, scale_raw);
	}
}

template<uint64_t M, uint64_t K, typename traits>
__device__ __forceinline__ void load_smem_tile_B(float* smem_B, const float* B_global, uint64_t N, uint64_t k_offset, uint64_t block_col) {
	constexpr uint64_t block_n			 = traits::block_tile_n;
	constexpr uint64_t block_k			 = traits::block_tile_k;
	constexpr uint64_t threads_per_block = traits::threads_per_block;

	const uint64_t tid					 = threadIdx.x;
	const uint64_t vec4_cols_per_row	 = block_n / 4;
	const uint64_t total_vec4_loads		 = block_k * vec4_cols_per_row;
	const uint64_t vec4_loads_per_thread = (total_vec4_loads + threads_per_block - 1) / threads_per_block;

	for (uint64_t i = 0; i < vec4_loads_per_thread; ++i) {
		const uint64_t vec4_idx = tid + i * threads_per_block;
		if (vec4_idx < total_vec4_loads) {
			const uint64_t row		= vec4_idx / vec4_cols_per_row;
			const uint64_t vec4_col = vec4_idx % vec4_cols_per_row;
			const uint64_t col		= vec4_col * 4;

			const uint64_t global_row = k_offset + row;
			const uint64_t global_col = block_col + col;

			if (global_row < K && global_col + 3 < N) {
				const uint64_t global_offset					 = global_row * N + global_col;
				const uint64_t smem_offset						 = row * block_n + col;
				*reinterpret_cast<float4*>(&smem_B[smem_offset]) = *reinterpret_cast<const float4*>(&B_global[global_offset]);
			} else {
				for (uint64_t elem = 0; elem < 4; ++elem) {
					const uint64_t elem_global_col = global_col + elem;
					const uint64_t elem_col		   = col + elem;
					if (global_row < K && elem_global_col < N && elem_col < block_n) {
						smem_B[row * block_n + elem_col] = B_global[global_row * N + elem_global_col];
					}
				}
			}
		}
	}
}

template<typename traits> __device__ __forceinline__ void compute_warp_tile(float* smem_A, float* smem_B, float accumulator[traits::thread_tile_m][traits::thread_tile_n],
	uint64_t warp_row, uint64_t warp_col) {
	constexpr uint64_t warp_m	= traits::warp_tile_m;
	constexpr uint64_t warp_n	= traits::warp_tile_n;
	constexpr uint64_t thread_m = traits::thread_tile_m;
	constexpr uint64_t thread_n = traits::thread_tile_n;
	constexpr uint64_t block_k	= traits::block_tile_k;
	constexpr uint64_t block_n	= traits::block_tile_n;
	constexpr uint64_t block_m	= traits::block_tile_m;

	const uint64_t lane_id		   = threadIdx.x % 32;
	const uint64_t threads_per_row = warp_n / thread_n;
	const uint64_t thread_row	   = lane_id / threads_per_row;
	const uint64_t thread_col	   = lane_id % threads_per_row;

	if constexpr (thread_m % 4 == 0 && thread_n % 4 == 0) {
		float4 frag_A[thread_m / 4];
		float4 frag_B[thread_n / 4];

		for (uint64_t k = 0; k < block_k; ++k) {
			for (uint64_t tm_vec = 0; tm_vec < thread_m / 4; ++tm_vec) {
				const uint64_t base_row	   = warp_row + thread_row * thread_m + tm_vec * 4;
				const uint64_t smem_offset = base_row * block_k + k;

				frag_A[tm_vec] = make_float4(smem_A[smem_offset], smem_A[smem_offset + block_k], smem_A[smem_offset + 2 * block_k], smem_A[smem_offset + 3 * block_k]);
			}

			for (uint64_t tn_vec = 0; tn_vec < thread_n / 4; ++tn_vec) {
				const uint64_t base_col	   = warp_col + thread_col * thread_n + tn_vec * 4;
				const uint64_t smem_offset = k * block_n + base_col;

				frag_B[tn_vec] = *reinterpret_cast<const float4*>(&smem_B[smem_offset]);
			}

			for (uint64_t tm_vec = 0; tm_vec < thread_m / 4; ++tm_vec) {
				for (uint64_t tn_vec = 0; tn_vec < thread_n / 4; ++tn_vec) {
					const float4& a_vec = frag_A[tm_vec];
					const float4& b_vec = frag_B[tn_vec];

					accumulator[tm_vec * 4][tn_vec * 4] += a_vec.x * b_vec.x;
					accumulator[tm_vec * 4][tn_vec * 4 + 1] += a_vec.x * b_vec.y;
					accumulator[tm_vec * 4][tn_vec * 4 + 2] += a_vec.x * b_vec.z;
					accumulator[tm_vec * 4][tn_vec * 4 + 3] += a_vec.x * b_vec.w;

					accumulator[tm_vec * 4 + 1][tn_vec * 4] += a_vec.y * b_vec.x;
					accumulator[tm_vec * 4 + 1][tn_vec * 4 + 1] += a_vec.y * b_vec.y;
					accumulator[tm_vec * 4 + 1][tn_vec * 4 + 2] += a_vec.y * b_vec.z;
					accumulator[tm_vec * 4 + 1][tn_vec * 4 + 3] += a_vec.y * b_vec.w;

					accumulator[tm_vec * 4 + 2][tn_vec * 4] += a_vec.z * b_vec.x;
					accumulator[tm_vec * 4 + 2][tn_vec * 4 + 1] += a_vec.z * b_vec.y;
					accumulator[tm_vec * 4 + 2][tn_vec * 4 + 2] += a_vec.z * b_vec.z;
					accumulator[tm_vec * 4 + 2][tn_vec * 4 + 3] += a_vec.z * b_vec.w;

					accumulator[tm_vec * 4 + 3][tn_vec * 4] += a_vec.w * b_vec.x;
					accumulator[tm_vec * 4 + 3][tn_vec * 4 + 1] += a_vec.w * b_vec.y;
					accumulator[tm_vec * 4 + 3][tn_vec * 4 + 2] += a_vec.w * b_vec.z;
					accumulator[tm_vec * 4 + 3][tn_vec * 4 + 3] += a_vec.w * b_vec.w;
				}
			}
		}
	} else {
		float frag_A[thread_m];
		float frag_B[thread_n];

		for (uint64_t k = 0; k < block_k; ++k) {
			for (uint64_t tm = 0; tm < thread_m; ++tm) {
				const uint64_t smem_row = warp_row + thread_row * thread_m + tm;
				if (smem_row < block_m) {
					frag_A[tm] = smem_A[smem_row * block_k + k];
				}
			}

			for (uint64_t tn = 0; tn < thread_n; ++tn) {
				const uint64_t smem_col = warp_col + thread_col * thread_n + tn;
				if (smem_col < block_n) {
					frag_B[tn] = smem_B[k * block_n + smem_col];
				}
			}

			for (uint64_t tm = 0; tm < thread_m; ++tm) {
				for (uint64_t tn = 0; tn < thread_n; ++tn) {
					accumulator[tm][tn] += frag_A[tm] * frag_B[tn];
				}
			}
		}
	}
}

template<typename traits> __device__ __forceinline__ void store_output_tile(float* C_global, float accumulator[traits::thread_tile_m][traits::thread_tile_n], uint64_t M,
	uint64_t N, uint64_t block_row, uint64_t block_col, uint64_t warp_row, uint64_t warp_col) {
	constexpr uint64_t thread_m = traits::thread_tile_m;
	constexpr uint64_t thread_n = traits::thread_tile_n;
	constexpr uint64_t warp_n	= traits::warp_tile_n;

	const uint64_t lane_id		   = threadIdx.x % 32;
	const uint64_t threads_per_row = warp_n / thread_n;
	const uint64_t thread_row	   = lane_id / threads_per_row;
	const uint64_t thread_col	   = lane_id % threads_per_row;

#pragma unroll
	for (uint64_t tm = 0; tm < thread_m; ++tm) {
#pragma unroll
		for (uint64_t tn = 0; tn < thread_n; ++tn) {
			const uint64_t global_row = block_row + warp_row + thread_row * thread_m + tm;
			const uint64_t global_col = block_col + warp_col + thread_col * thread_n + tn;

			if (global_row < M && global_col < N) {
				C_global[global_row * N + global_col] = accumulator[tm][tn];
			}
		}
	}
}

template<uint64_t M, uint64_t K, typename traits> __launch_bounds__(256, 2) __global__ void rt_tm_gemm_kernel(const block_q8_0* A, const float* B, float* C, uint64_t N) {
	constexpr uint64_t block_m	= traits::block_tile_m;
	constexpr uint64_t block_n	= traits::block_tile_n;
	constexpr uint64_t block_k	= traits::block_tile_k;
	constexpr uint64_t warp_m	= traits::warp_tile_m;
	constexpr uint64_t warp_n	= traits::warp_tile_n;
	constexpr uint64_t thread_m = traits::thread_tile_m;
	constexpr uint64_t thread_n = traits::thread_tile_n;
	constexpr uint64_t warps_m	= traits::warps_m;
	constexpr uint64_t warps_n	= traits::warps_n;

	__shared__ float smem_A[2][block_m * block_k];
	__shared__ float smem_B[2][block_k * block_n];

	const uint64_t block_row = blockIdx.y * block_m;
	const uint64_t block_col = blockIdx.x * block_n;

	const uint64_t warp_id	= threadIdx.x / 32;
	const uint64_t warp_row = (warp_id / warps_n) * warp_m;
	const uint64_t warp_col = (warp_id % warps_n) * warp_n;

	float accumulator[thread_m][thread_n];
#pragma unroll
	for (uint64_t tm = 0; tm < thread_m; ++tm) {
#pragma unroll
		for (uint64_t tn = 0; tn < thread_n; ++tn) {
			accumulator[tm][tn] = 0.0f;
		}
	}

	uint64_t smem_write_stage = 0;
	uint64_t smem_read_stage  = 0;

	load_smem_tile_A<M, K, traits>(smem_A[smem_write_stage], A, N, 0, block_row);
	load_smem_tile_B<M, K, traits>(smem_B[smem_write_stage], B, N, 0, block_col);
	__syncthreads();

	for (uint64_t k_tile = 0; k_tile < K; k_tile += block_k) {
		smem_read_stage	 = smem_write_stage;
		smem_write_stage = 1 - smem_write_stage;

		if (k_tile + block_k < K) {
			load_smem_tile_A<M, K, traits>(smem_A[smem_write_stage], A, N, k_tile + block_k, block_row);
			load_smem_tile_B<M, K, traits>(smem_B[smem_write_stage], B, N, k_tile + block_k, block_col);
		}

		compute_warp_tile<traits>(smem_A[smem_read_stage], smem_B[smem_read_stage], accumulator, warp_row, warp_col);

		__syncthreads();
	}

	store_output_tile<traits>(C, accumulator, M, N, block_row, block_col, warp_row, warp_col);
}

using mul_mat_1_to_128	 = cuda_kernel_traits<32, 64, 16, 16, 32, 4, 4>;
using mul_mat_129_to_512 = cuda_kernel_traits<128, 64, 32, 32, 32, 8, 4>;

template<uint64_t M, uint64_t K, typename traits>
__device__ __forceinline__ void load_smem_tile_A_float(float* smem_A, const float* A_global, uint64_t N, uint64_t k_offset, uint64_t block_row) {
	constexpr uint64_t block_m			 = traits::block_tile_m;
	constexpr uint64_t block_k			 = traits::block_tile_k;
	constexpr uint64_t threads_per_block = traits::threads_per_block;
	const uint64_t tid					 = threadIdx.x;
	const uint64_t elements_per_block	 = block_m * block_k;
	const uint64_t vec4_elements		 = elements_per_block / 4;
	const uint64_t vec4_per_thread		 = (vec4_elements + threads_per_block - 1) / threads_per_block;

	for (uint64_t i = 0; i < vec4_per_thread; ++i) {
		const uint64_t vec4_idx	  = tid + i * threads_per_block;
		const uint64_t linear_idx = vec4_idx * 4;

		if (linear_idx < elements_per_block) {
			const uint64_t row		  = linear_idx / block_k;
			const uint64_t col		  = linear_idx % block_k;
			const uint64_t global_row = block_row + row;
			const uint64_t global_col = k_offset + col;

			if (global_row < M && global_col + 3 < K) {
				const uint64_t global_offset					 = global_row * K + global_col;
				const uint64_t smem_offset						 = row * block_k + col;
				*reinterpret_cast<float4*>(&smem_A[smem_offset]) = *reinterpret_cast<const float4*>(&A_global[global_offset]);
			} else {
				for (uint64_t elem = 0; elem < 4; ++elem) {
					const uint64_t elem_global_col = global_col + elem;
					const uint64_t elem_col		   = col + elem;
					if (global_row < M && elem_global_col < K && elem_col < block_k) {
						smem_A[row * block_k + elem_col] = A_global[global_row * K + elem_global_col];
					}
				}
			}
		}
	}
}

template<uint64_t M, uint64_t K, typename traits>
__device__ __forceinline__ void load_smem_tile_B_float(float* smem_B, const float* B_global, uint64_t N, uint64_t k_offset, uint64_t block_col) {
	constexpr uint64_t block_n			 = traits::block_tile_n;
	constexpr uint64_t block_k			 = traits::block_tile_k;
	constexpr uint64_t threads_per_block = traits::threads_per_block;

	const uint64_t tid					 = threadIdx.x;
	const uint64_t vec4_cols_per_row	 = block_n / 4;
	const uint64_t total_vec4_loads		 = block_k * vec4_cols_per_row;
	const uint64_t vec4_loads_per_thread = (total_vec4_loads + threads_per_block - 1) / threads_per_block;

	for (uint64_t i = 0; i < vec4_loads_per_thread; ++i) {
		const uint64_t vec4_idx = tid + i * threads_per_block;
		if (vec4_idx < total_vec4_loads) {
			const uint64_t row		= vec4_idx / vec4_cols_per_row;
			const uint64_t vec4_col = vec4_idx % vec4_cols_per_row;
			const uint64_t col		= vec4_col * 4;

			const uint64_t global_row = k_offset + row;
			const uint64_t global_col = block_col + col;

			if (global_row < K && global_col + 3 < N) {
				const uint64_t global_offset					 = global_row * N + global_col;
				const uint64_t smem_offset						 = row * block_n + col;
				*reinterpret_cast<float4*>(&smem_B[smem_offset]) = *reinterpret_cast<const float4*>(&B_global[global_offset]);
			} else {
				for (uint64_t elem = 0; elem < 4; ++elem) {
					const uint64_t elem_global_col = global_col + elem;
					const uint64_t elem_col		   = col + elem;
					if (global_row < K && elem_global_col < N && elem_col < block_n) {
						smem_B[row * block_n + elem_col] = B_global[global_row * N + elem_global_col];
					}
				}
			}
		}
	}
}

template<typename traits> __device__ __forceinline__ void compute_warp_tile_float(float* smem_A, float* smem_B, float accumulator[traits::thread_tile_m][traits::thread_tile_n],
	uint64_t warp_row, uint64_t warp_col) {
	constexpr uint64_t warp_m	= traits::warp_tile_m;
	constexpr uint64_t warp_n	= traits::warp_tile_n;
	constexpr uint64_t thread_m = traits::thread_tile_m;
	constexpr uint64_t thread_n = traits::thread_tile_n;
	constexpr uint64_t block_k	= traits::block_tile_k;
	constexpr uint64_t block_n	= traits::block_tile_n;
	constexpr uint64_t block_m	= traits::block_tile_m;

	const uint64_t lane_id		   = threadIdx.x % 32;
	const uint64_t threads_per_row = warp_n / thread_n;
	const uint64_t thread_row	   = lane_id / threads_per_row;
	const uint64_t thread_col	   = lane_id % threads_per_row;

	if constexpr (thread_m % 4 == 0 && thread_n % 4 == 0) {
		float4 frag_A[thread_m / 4];
		float4 frag_B[thread_n / 4];

		for (uint64_t k = 0; k < block_k; ++k) {
			for (uint64_t tm_vec = 0; tm_vec < thread_m / 4; ++tm_vec) {
				const uint64_t base_row	   = warp_row + thread_row * thread_m + tm_vec * 4;
				const uint64_t smem_offset = base_row * block_k + k;

				frag_A[tm_vec] = make_float4(smem_A[smem_offset], smem_A[smem_offset + block_k], smem_A[smem_offset + 2 * block_k], smem_A[smem_offset + 3 * block_k]);
			}

			for (uint64_t tn_vec = 0; tn_vec < thread_n / 4; ++tn_vec) {
				const uint64_t base_col	   = warp_col + thread_col * thread_n + tn_vec * 4;
				const uint64_t smem_offset = k * block_n + base_col;

				frag_B[tn_vec] = *reinterpret_cast<const float4*>(&smem_B[smem_offset]);
			}

			for (uint64_t tm_vec = 0; tm_vec < thread_m / 4; ++tm_vec) {
				for (uint64_t tn_vec = 0; tn_vec < thread_n / 4; ++tn_vec) {
					const float4& a_vec = frag_A[tm_vec];
					const float4& b_vec = frag_B[tn_vec];

					accumulator[tm_vec * 4][tn_vec * 4] += a_vec.x * b_vec.x;
					accumulator[tm_vec * 4][tn_vec * 4 + 1] += a_vec.x * b_vec.y;
					accumulator[tm_vec * 4][tn_vec * 4 + 2] += a_vec.x * b_vec.z;
					accumulator[tm_vec * 4][tn_vec * 4 + 3] += a_vec.x * b_vec.w;

					accumulator[tm_vec * 4 + 1][tn_vec * 4] += a_vec.y * b_vec.x;
					accumulator[tm_vec * 4 + 1][tn_vec * 4 + 1] += a_vec.y * b_vec.y;
					accumulator[tm_vec * 4 + 1][tn_vec * 4 + 2] += a_vec.y * b_vec.z;
					accumulator[tm_vec * 4 + 1][tn_vec * 4 + 3] += a_vec.y * b_vec.w;

					accumulator[tm_vec * 4 + 2][tn_vec * 4] += a_vec.z * b_vec.x;
					accumulator[tm_vec * 4 + 2][tn_vec * 4 + 1] += a_vec.z * b_vec.y;
					accumulator[tm_vec * 4 + 2][tn_vec * 4 + 2] += a_vec.z * b_vec.z;
					accumulator[tm_vec * 4 + 2][tn_vec * 4 + 3] += a_vec.z * b_vec.w;

					accumulator[tm_vec * 4 + 3][tn_vec * 4] += a_vec.w * b_vec.x;
					accumulator[tm_vec * 4 + 3][tn_vec * 4 + 1] += a_vec.w * b_vec.y;
					accumulator[tm_vec * 4 + 3][tn_vec * 4 + 2] += a_vec.w * b_vec.z;
					accumulator[tm_vec * 4 + 3][tn_vec * 4 + 3] += a_vec.w * b_vec.w;
				}
			}
		}
	} else {
		float frag_A[thread_m];
		float frag_B[thread_n];

		for (uint64_t k = 0; k < block_k; ++k) {
			for (uint64_t tm = 0; tm < thread_m; ++tm) {
				const uint64_t smem_row = warp_row + thread_row * thread_m + tm;
				if (smem_row < block_m) {
					frag_A[tm] = smem_A[smem_row * block_k + k];
				}
			}

			for (uint64_t tn = 0; tn < thread_n; ++tn) {
				const uint64_t smem_col = warp_col + thread_col * thread_n + tn;
				if (smem_col < block_n) {
					frag_B[tn] = smem_B[k * block_n + smem_col];
				}
			}

			for (uint64_t tm = 0; tm < thread_m; ++tm) {
				for (uint64_t tn = 0; tn < thread_n; ++tn) {
					accumulator[tm][tn] += frag_A[tm] * frag_B[tn];
				}
			}
		}
	}
}

template<typename traits> __device__ __forceinline__ void store_output_tile_float(float* C_global, float accumulator[traits::thread_tile_m][traits::thread_tile_n], uint64_t M,
	uint64_t N, uint64_t block_row, uint64_t block_col, uint64_t warp_row, uint64_t warp_col) {
	constexpr uint64_t thread_m = traits::thread_tile_m;
	constexpr uint64_t thread_n = traits::thread_tile_n;
	constexpr uint64_t warp_n	= traits::warp_tile_n;

	const uint64_t lane_id		   = threadIdx.x % 32;
	const uint64_t threads_per_row = warp_n / thread_n;
	const uint64_t thread_row	   = lane_id / threads_per_row;
	const uint64_t thread_col	   = lane_id % threads_per_row;

#pragma unroll
	for (uint64_t tm = 0; tm < thread_m; ++tm) {
#pragma unroll
		for (uint64_t tn = 0; tn < thread_n; ++tn) {
			const uint64_t global_row = block_row + warp_row + thread_row * thread_m + tm;
			const uint64_t global_col = block_col + warp_col + thread_col * thread_n + tn;

			if (global_row < M && global_col < N) {
				C_global[global_row * N + global_col] = accumulator[tm][tn];
			}
		}
	}
}

template<uint64_t M, uint64_t K> __launch_bounds__(256, 2) __global__ void rt_tm_gemm_float_kernel(const float* A, const float* B, float* C, uint64_t N) {
	using traits = mul_mat_1_to_128;

	constexpr uint64_t block_m	= traits::block_tile_m;
	constexpr uint64_t block_n	= traits::block_tile_n;
	constexpr uint64_t block_k	= traits::block_tile_k;
	constexpr uint64_t warp_m	= traits::warp_tile_m;
	constexpr uint64_t warp_n	= traits::warp_tile_n;
	constexpr uint64_t thread_m = traits::thread_tile_m;
	constexpr uint64_t thread_n = traits::thread_tile_n;
	constexpr uint64_t warps_m	= traits::warps_m;
	constexpr uint64_t warps_n	= traits::warps_n;

	__shared__ float smem_A[2][block_m * block_k];
	__shared__ float smem_B[2][block_k * block_n];

	const uint64_t block_row = blockIdx.y * block_m;
	const uint64_t block_col = blockIdx.x * block_n;

	const uint64_t warp_id	= threadIdx.x / 32;
	const uint64_t warp_row = (warp_id / warps_n) * warp_m;
	const uint64_t warp_col = (warp_id % warps_n) * warp_n;

	float accumulator[thread_m][thread_n];
#pragma unroll
	for (uint64_t tm = 0; tm < thread_m; ++tm) {
#pragma unroll
		for (uint64_t tn = 0; tn < thread_n; ++tn) {
			accumulator[tm][tn] = 0.0f;
		}
	}

	uint64_t smem_write_stage = 0;
	uint64_t smem_read_stage  = 0;

	load_smem_tile_A_float<M, K, traits>(smem_A[smem_write_stage], A, N, 0, block_row);
	load_smem_tile_B_float<M, K, traits>(smem_B[smem_write_stage], B, N, 0, block_col);
	__syncthreads();

	for (uint64_t k_tile = 0; k_tile < K; k_tile += block_k) {
		smem_read_stage	 = smem_write_stage;
		smem_write_stage = 1 - smem_write_stage;

		if (k_tile + block_k < K) {
			load_smem_tile_A_float<M, K, traits>(smem_A[smem_write_stage], A, N, k_tile + block_k, block_row);
			load_smem_tile_B_float<M, K, traits>(smem_B[smem_write_stage], B, N, k_tile + block_k, block_col);
		}

		compute_warp_tile_float<traits>(smem_A[smem_read_stage], smem_B[smem_read_stage], accumulator, warp_row, warp_col);

		__syncthreads();
	}

	store_output_tile_float<traits>(C, accumulator, M, N, block_row, block_col, warp_row, warp_col);
}

#include <cutlass_rt_tm/gemm/device/gemm.h>
#include <cutlass_rt_tm/cuda_host_adapter.hpp>

using element_a = float;
using element_b = float;
using element_c = float;
using layout_a	= cutlass_rt_tm::layout::RowMajor;
using layout_b	= cutlass_rt_tm::layout::RowMajor;
using layout_c	= cutlass_rt_tm::layout::RowMajor;

__global__ void dequantize_a_matrix_kernel(const block_q8_0* input_blocks, float* output, uint64_t total_elements) {
	const uint64_t idx	  = blockIdx.x * blockDim.x + threadIdx.x;
	const uint64_t stride = blockDim.x * gridDim.x;

	for (uint64_t i = idx; i < total_elements; i += stride) {
		const uint64_t block_idx	 = i >> 5;
		const uint64_t elem_in_block = i & 31;

		const block_q8_0& block = input_blocks[block_idx];
		const float scale		= __half2float(*reinterpret_cast<const __half*>(&block.scale));
		output[i]				= scale * static_cast<float>(block.quants[elem_in_block]);
	}
}

template<uint64_t M, uint64_t K> struct nihilus_gemm {
	BNCH_SWT_INLINE static uint64_t impl(cuda_buffer& buffer, uint64_t& current_index, std::vector<std::vector<float>>& floats, std::vector<std::vector<block_q8_0>>& blocks,
		std::vector<std::vector<float>>& outputs, uint64_t N) {
		static constexpr uint64_t total_blocks_A = ((M * K) + 32 - 1) >> 5;
		static constexpr uint64_t blocks_size	 = total_blocks_A * sizeof(block_q8_0);
		const uint64_t floats_B_size			 = (K * N) * sizeof(float);
		const uint64_t outputs_C_size			 = (M * N) * sizeof(float);
		static constexpr uint64_t dequant_A_size = (M * K) * sizeof(float);

		uint64_t offset				= 0;
		const block_q8_0* A_ptr_raw = reinterpret_cast<const block_q8_0*>(static_cast<uint8_t*>(buffer.data()) + offset);
		offset						= round_up_to_multiple<64>(offset + blocks_size);

		const float* B_ptr = reinterpret_cast<const float*>(static_cast<uint8_t*>(buffer.data()) + offset);
		offset			   = round_up_to_multiple<64>(offset + floats_B_size);

		float* C_ptr = reinterpret_cast<float*>(static_cast<uint8_t*>(buffer.data()) + offset);
		offset		 = round_up_to_multiple<64>(offset + outputs_C_size);

		float* A_ptr = reinterpret_cast<float*>(static_cast<uint8_t*>(buffer.data()) + offset);
		hipError_t dequant_err;

		if (N <= 256) {
			using policy = mul_mat_1_to_128;
			dim3 block(policy::threads_per_block);
			dim3 grid((N + policy::block_tile_n - 1) / policy::block_tile_n, (M + policy::block_tile_m - 1) / policy::block_tile_m);
			rt_tm_gemm_kernel<M, K, policy><<<grid, block>>>(A_ptr_raw, B_ptr, C_ptr, N);
		} else {
			constexpr uint64_t total_elements_A = M * K;
			const dim3 dequant_grid((total_blocks_A + 1023) / 1024);
			const dim3 dequant_block(1024);
			dequantize_a_matrix_kernel<<<dequant_grid, dequant_block>>>(A_ptr_raw, A_ptr, total_elements_A);

			dequant_err = hipGetLastError();
			if (dequant_err != hipSuccess) {
				std::cerr << "Dequantization kernel failed: " << hipGetErrorString(dequant_err) << std::endl;
			}

			hipDeviceSynchronize();

			using index_type		= cutlass_rt_tm::gemm::GemmCoord::Index;
			using nihilus_gemm_type = cutlass_rt_tm::gemm::device::Gemm<M, K, element_a, layout_a, element_b, layout_b, element_c, layout_c, element_c>;
			nihilus_gemm_type gemm_op;
			cutlass_rt_tm::Status status = gemm_op({ { static_cast<index_type>(M), static_cast<index_type>(N), static_cast<index_type>(K) }, { A_ptr, static_cast<index_type>(K) },
				{ B_ptr, static_cast<index_type>(N) }, { C_ptr, static_cast<index_type>(N) }, { C_ptr, static_cast<index_type>(N) }, { 1.0f, 0.0f } });

			if (status != cutlass_rt_tm::Status::kSuccess) {
				std::cerr << "CUTLASS GEMM failed: " << cutlass_rt_tm::cutlass_rt_tmGetStatusString(status) << std::endl;
			}
		}

		dequant_err = hipDeviceSynchronize();
		if (dequant_err != hipSuccess) {
			std::cerr << "hipDeviceSynchronize failed: " << hipGetErrorString(dequant_err) << std::endl;
		}

		++current_index;
		return outputs[current_index - 1].size() * sizeof(float);
	}

	BNCH_SWT_INLINE static uint64_t impl(cuda_buffer& buffer, uint64_t& current_index, std::vector<std::vector<float>>& floats_A, std::vector<std::vector<float>>& floats_B,
		std::vector<std::vector<float>>& outputs, uint64_t N) {
		const uint64_t floats_A_size  = (M * K) * sizeof(float);
		const uint64_t floats_B_size  = (K * N) * sizeof(float);
		const uint64_t outputs_C_size = (M * N) * sizeof(float);

		uint64_t offset = 0;

		const float* A_ptr = reinterpret_cast<const float*>(static_cast<uint8_t*>(buffer.data()) + offset);
		offset			   = round_up_to_multiple<64>(offset + floats_A_size);

		const float* B_ptr = reinterpret_cast<const float*>(static_cast<uint8_t*>(buffer.data()) + offset);
		offset			   = round_up_to_multiple<64>(offset + floats_B_size);

		float* C_ptr = reinterpret_cast<float*>(static_cast<uint8_t*>(buffer.data()) + offset);
		using index_type		= cutlass_rt_tm::gemm::GemmCoord::Index;
		using nihilus_gemm_type = cutlass_rt_tm::gemm::device::Gemm<M, K, element_a, layout_a, element_b, layout_b, element_c, layout_c, element_c>;
		nihilus_gemm_type gemm_op;
		cutlass_rt_tm::Status status = gemm_op({ { static_cast<index_type>(M), static_cast<index_type>(N), static_cast<index_type>(K) }, { A_ptr, static_cast<index_type>(K) },
			{ B_ptr, static_cast<index_type>(N) }, { C_ptr, static_cast<index_type>(N) }, { C_ptr, static_cast<index_type>(N) }, { 1.0f, 0.0f } });

		if (status != cutlass_rt_tm::Status::kSuccess) {
			std::cerr << "CUTLASS GEMM failed: " << cutlass_rt_tm::cutlass_rt_tmGetStatusString(status) << std::endl;
		}

		hipError_t err = hipGetLastError();
		if (err != hipSuccess) {
			std::cerr << "❌ CUDA error after CUTLASS: " << hipGetErrorString(err) << std::endl;
		}

		err = hipDeviceSynchronize();
		if (err != hipSuccess) {
			std::cerr << "❌ CUDA synchronization failed: " << hipGetErrorString(err) << std::endl;
		}

		++current_index;
		return outputs[current_index - 1].size() * sizeof(float);
	}
};

#include <cutlass/gemm/device/gemm.h>
#include <cutlass/cuda_host_adapter.hpp>

using cutless_element_a = float;
using cutless_element_b = float;
using cutless_element_c = float;
using cutless_layout_a	= cutlass::layout::RowMajor;
using cutless_layout_b	= cutlass::layout::RowMajor;
using cutless_layout_c	= cutlass::layout::RowMajor;

using Cutlass_Gemm = cutlass::gemm::device::Gemm<cutless_element_a, cutless_layout_a, cutless_element_b, cutless_layout_b, cutless_element_c, cutless_layout_c, cutless_element_c>;

template<uint64_t M, uint64_t K> struct cutlass_baseline_gemm {
	BNCH_SWT_INLINE static uint64_t impl(cuda_buffer& buffer, uint64_t& current_index, std::vector<std::vector<float>>& floats, std::vector<std::vector<block_q8_0>>& blocks,
		std::vector<std::vector<float>>& outputs, uint64_t N) {
		static constexpr uint64_t total_blocks_A = ((M * K) + 32 - 1) >> 5;
		static constexpr uint64_t blocks_size	 = total_blocks_A * sizeof(block_q8_0);
		const uint64_t floats_B_size			 = (K * N) * sizeof(float);
		const uint64_t outputs_C_size			 = (M * N) * sizeof(float);
		static constexpr uint64_t dequant_A_size = (M * K) * sizeof(float);

		uint64_t offset				= 0;
		const block_q8_0* A_ptr_raw = reinterpret_cast<const block_q8_0*>(static_cast<uint8_t*>(buffer.data()) + offset);
		offset						= round_up_to_multiple<64>(offset + blocks_size);

		const float* B_ptr = reinterpret_cast<const float*>(static_cast<uint8_t*>(buffer.data()) + offset);
		offset			   = round_up_to_multiple<64>(offset + floats_B_size);

		float* C_ptr = reinterpret_cast<float*>(static_cast<uint8_t*>(buffer.data()) + offset);
		offset		 = round_up_to_multiple<64>(offset + outputs_C_size);

		float* A_ptr = reinterpret_cast<float*>(static_cast<uint8_t*>(buffer.data()) + offset);

		constexpr uint64_t total_elements_A = M * K;
		const dim3 dequant_grid((total_elements_A + 1023) / 1024);
		const dim3 dequant_block(1024);

		cutlass_dequantize_q8_vectorized_kernel<<<dequant_grid, dequant_block>>>(A_ptr_raw, A_ptr, total_elements_A);

		hipError_t dequant_err = hipGetLastError();
		if (dequant_err != hipSuccess) {
			std::cerr << "Dequantization kernel failed: " << hipGetErrorString(dequant_err) << std::endl;
		}

		hipDeviceSynchronize();

		dequant_err = hipGetLastError();
		if (dequant_err != hipSuccess) {
			std::cerr << "hipDeviceSynchronize failed: " << hipGetErrorString(dequant_err) << std::endl;
		}

		using index_type = cutlass::gemm::GemmCoord::Index;
		Cutlass_Gemm gemm_op;
		cutlass::Status status = gemm_op({ { static_cast<index_type>(M), static_cast<index_type>(N), static_cast<index_type>(K) }, { A_ptr, static_cast<index_type>(K) },
			{ B_ptr, static_cast<index_type>(N) }, { C_ptr, static_cast<index_type>(N) }, { C_ptr, static_cast<index_type>(N) }, { 1.0f, 0.0f } });

		if (status != cutlass::Status::kSuccess) {
			std::cerr << "CUTLASS GEMM failed: " << cutlass::cutlassGetStatusString(status) << std::endl;
		}

		hipError_t err = hipGetLastError();
		if (err != hipSuccess) {
			std::cerr << "❌ CUDA error after CUTLASS: " << hipGetErrorString(err) << std::endl;
		}

		err = hipDeviceSynchronize();
		if (err != hipSuccess) {
			std::cerr << "❌ CUDA synchronization failed: " << hipGetErrorString(err) << std::endl;
		}

		++current_index;
		return outputs[current_index - 1].size() * sizeof(float);
	}

	BNCH_SWT_INLINE static uint64_t impl(cuda_buffer& buffer, uint64_t& current_index, std::vector<std::vector<float>>& floats_A, std::vector<std::vector<float>>& floats_B,
		std::vector<std::vector<float>>& outputs, uint64_t N) {
		const uint64_t floats_A_size  = (M * K) * sizeof(float);
		const uint64_t floats_B_size  = (K * N) * sizeof(float);
		const uint64_t outputs_C_size = (M * N) * sizeof(float);

		uint64_t offset = 0;

		const float* A_ptr = reinterpret_cast<const float*>(static_cast<uint8_t*>(buffer.data()) + offset);
		offset			   = round_up_to_multiple<64>(offset + floats_A_size);

		const float* B_ptr = reinterpret_cast<const float*>(static_cast<uint8_t*>(buffer.data()) + offset);
		offset			   = round_up_to_multiple<64>(offset + floats_B_size);

		float* C_ptr = reinterpret_cast<float*>(static_cast<uint8_t*>(buffer.data()) + offset);

		using index_type = cutlass::gemm::GemmCoord::Index;
		Cutlass_Gemm gemm_op;
		cutlass::Status status = gemm_op({ { static_cast<index_type>(M), static_cast<index_type>(N), static_cast<index_type>(K) }, { A_ptr, static_cast<index_type>(K) },
			{ B_ptr, static_cast<index_type>(N) }, { C_ptr, static_cast<index_type>(N) }, { C_ptr, static_cast<index_type>(N) }, { 1.0f, 0.0f } });

		if (status != cutlass::Status::kSuccess) {
			std::cerr << "CUTLASS GEMM failed: " << cutlass::cutlassGetStatusString(status) << std::endl;
		}

		hipError_t err = hipGetLastError();
		if (err != hipSuccess) {
			std::cerr << "❌ CUDA error after CUTLASS: " << hipGetErrorString(err) << std::endl;
		}

		err = hipDeviceSynchronize();
		if (err != hipSuccess) {
			std::cerr << "❌ CUDA synchronization failed: " << hipGetErrorString(err) << std::endl;
		}

		++current_index;
		return outputs[current_index - 1].size() * sizeof(float);
	}
};

template<bnch_swt::string_literal rhs> inline void compare_outputs(const std::vector<std::vector<float>>& outputs01, const std::vector<std::vector<float>>& outputs02) {
	static constexpr float relative_tolerance = 1e-1f;
	static constexpr float absolute_tolerance = 1e-30f;
	if (outputs01.size() != outputs02.size()) {
		std::cerr << "Unequal output sizes!" << std::endl;
		return;
	}
	for (uint64_t x = 0; x < outputs02.size(); ++x) {
		if (outputs01[x].size() != outputs02[x].size()) {
			std::cerr << "Unequal output sizes!" << std::endl;
			return;
		}
		for (uint64_t y = 0; y < outputs01[x].size(); ++y) {
			const float val1 = outputs01[x][y];
			const float val2 = outputs02[x][y];

			const float abs_diff = std::abs(val1 - val2);
			const float max_val	 = std::max(std::abs(val1), std::abs(val2));

			if (std::isinf(val1) || std::isinf(val2) || std::isnan(val1) || std::isnan(val2) || !((abs_diff <= absolute_tolerance) || (abs_diff <= relative_tolerance * max_val))) {
				std::cerr << rhs.operator std::string_view() << ": Mismatch at [" << x << "," << y << "]: Ref Val: " << val1 << " vs Incorrect Val: " << val2 << std::endl;
				std::cerr << "Relative difference: " << (abs_diff / max_val) * 100.0f << "%" << std::endl;
				return;
			}
		}
	}
}

template<uint64_t M, uint64_t K, uint64_t matB_dim_00, uint64_t N> BNCH_SWT_INLINE void test_function() {
	static constexpr uint64_t matC_dim_00{ M };
	static constexpr uint64_t matC_dim_01{ N };
	static constexpr uint64_t total_elements_C{ matC_dim_00 * matC_dim_01 };
	static constexpr uint64_t total_blocks_a{
		static_cast<uint64_t>(static_cast<float>(M * K) * static_cast<float>(sizeof(block_q8_0)) / static_cast<float>(std::size(block_q8_0{}.quants))) / sizeof(block_q8_0)
	};
	static constexpr uint64_t total_floats_b{ matB_dim_00 * N };
	std::vector<std::vector<std::vector<float>>> block_floats{ generate_floats_final<total_iterations, M, K>() };
	std::vector<std::vector<float>> floats{ generate_values_final(generate_floats_final<total_iterations, K, N>()) };
	std::vector<std::vector<block_q8_0>> blocks{ generate_values_final(generate_blocks_final(block_floats)) };
	std::vector<std::vector<float>> outputs01{};
	std::vector<std::vector<float>> outputs02{};
	std::vector<std::vector<float>> outputs03{};
	outputs01.resize(total_iterations);
	outputs02.resize(total_iterations);
	outputs03.resize(total_iterations);
	for (uint64_t x = 0; x < total_iterations; ++x) {
		outputs01[x].resize(total_elements_C);
		outputs02[x].resize(total_elements_C);
		outputs03[x].resize(total_elements_C);
	}

	static constexpr bnch_swt::string_literal stage_name{ "(Q8_0 * F32) mul_mat: [" + bnch_swt::internal::toStringLiteral<M>() + "x" + bnch_swt::internal::toStringLiteral<K>() +
		" * " + bnch_swt::internal::toStringLiteral<matB_dim_00>() + "x" + bnch_swt::internal::toStringLiteral<N>() + "]" };
	static constexpr uint64_t total_elements_A = M * K;
	static constexpr uint64_t total_blocks_A   = (total_elements_A + 32 - 1) / 32;
	static constexpr uint64_t blocks_size	   = total_blocks_A * sizeof(block_q8_0);
	static constexpr uint64_t floats_B_count   = matB_dim_00 * N;
	static constexpr uint64_t floats_B_size	   = floats_B_count * sizeof(float);
	static constexpr uint64_t outputs_C_count  = M * N;
	static constexpr uint64_t outputs_C_size   = outputs_C_count * sizeof(float);
	static constexpr uint64_t dequant_A_size   = M * K * sizeof(float);

	uint64_t total_buffer_size = 0;
	total_buffer_size += round_up_to_multiple<64>(blocks_size);
	total_buffer_size += round_up_to_multiple<64>(floats_B_size);
	total_buffer_size += round_up_to_multiple<64>(outputs_C_size);
	total_buffer_size += round_up_to_multiple<64>(dequant_A_size);

	cuda_buffer buffer{};
	buffer.init(total_buffer_size);

	uint64_t current_index{};
	bnch_swt::benchmark_stage<stage_name, total_iterations, measured_iterations>::template runBenchmarkWithPrep<"ggml_cuda_mul_mat", cuda_mul_mat_01_prep<M, K>,
		ggml_cuda_mul_mat<M, K>>(buffer, current_index, floats, blocks, outputs01, N);
	current_index = 0;

	bnch_swt::benchmark_stage<stage_name, total_iterations, measured_iterations>::template runBenchmarkWithPrep<"cutlass_baseline_gemm", cuda_mul_mat_01_prep<M, K>,
		cutlass_baseline_gemm<M, K>>(buffer, current_index, floats, blocks, outputs02, N);
	current_index = 0;

	bnch_swt::benchmark_stage<stage_name, total_iterations, measured_iterations>::template runBenchmarkWithPrep<"nihilus_gemm", cuda_mul_mat_01_prep<M, K>,
		nihilus_gemm<M, K>>(buffer, current_index, floats, blocks, outputs03, N);
	current_index = 0;

	bnch_swt::benchmark_stage<stage_name, total_iterations, measured_iterations>::printResults();
	compare_outputs<"cutlass_baseline_gemm Incorrect Value">(outputs01, outputs02);
	compare_outputs<"nihilus_gemm Incorrect Value">(outputs01, outputs03);
};

template<uint64_t M, uint64_t K, uint64_t matB_dim_00, uint64_t N> BNCH_SWT_INLINE void test_function_floats() {
	static constexpr uint64_t matC_dim_00{ M };
	static constexpr uint64_t matC_dim_01{ N };
	static constexpr uint64_t total_elements_C{ matC_dim_00 * matC_dim_01 };
	std::vector<std::vector<float>> floats_a{ generate_values_final(generate_floats_final<total_iterations, M, K>()) };
	std::vector<std::vector<float>> floats_b{ generate_values_final(generate_floats_final<total_iterations, K, N>()) };
	std::vector<std::vector<float>> outputs01{};
	std::vector<std::vector<float>> outputs02{};
	std::vector<std::vector<float>> outputs03{};
	std::vector<std::vector<float>> outputs04{};
	std::vector<std::vector<float>> outputs05{};
	outputs01.resize(total_iterations);
	outputs02.resize(total_iterations);
	outputs03.resize(total_iterations);
	outputs04.resize(total_iterations);
	outputs05.resize(total_iterations);
	for (uint64_t x = 0; x < total_iterations; ++x) {
		outputs01[x].resize(total_elements_C);
		outputs02[x].resize(total_elements_C);
		outputs03[x].resize(total_elements_C);
		outputs04[x].resize(total_elements_C);
		outputs05[x].resize(total_elements_C);
	}

	static constexpr bnch_swt::string_literal stage_name{ "(F32 * F32) mul_mat: [" + bnch_swt::internal::toStringLiteral<M>() + "x" + bnch_swt::internal::toStringLiteral<K>() +
		" * " + bnch_swt::internal::toStringLiteral<matB_dim_00>() + "x" + bnch_swt::internal::toStringLiteral<N>() + "]" };
	constexpr uint64_t total_elements_A = M * K;
	constexpr uint64_t total_elements_B = K * N;
	constexpr uint64_t floats_A_size	= total_elements_A * sizeof(float);
	constexpr uint64_t floats_B_size	= total_elements_B * sizeof(float);
	constexpr uint64_t floats_C_size	= total_elements_C * sizeof(float);

	uint64_t total_buffer_size = 0;
	total_buffer_size += round_up_to_multiple<64>(floats_A_size);
	total_buffer_size += round_up_to_multiple<64>(floats_B_size);
	total_buffer_size += round_up_to_multiple<64>(floats_C_size);

	cuda_buffer buffer{};
	buffer.init(total_buffer_size);

	uint64_t current_index{};
	bnch_swt::benchmark_stage<stage_name, total_iterations, measured_iterations>::template runBenchmarkWithPrep<"ggml_cuda_mul_mat", cuda_mul_mat_01_prep<M, K>,
		ggml_cuda_mul_mat<M, K>>(buffer, current_index, floats_a, floats_b, outputs01, N);
	current_index = 0;

	bnch_swt::benchmark_stage<stage_name, total_iterations, measured_iterations>::template runBenchmarkWithPrep<"nihilus_gemm", cuda_mul_mat_01_prep<M, K>, nihilus_gemm<M, K>>(
		buffer, current_index, floats_a, floats_b, outputs04, N);
	current_index = 0;

	bnch_swt::benchmark_stage<stage_name, total_iterations, measured_iterations>::template runBenchmarkWithPrep<"cutlass_baseline_gemm", cuda_mul_mat_01_prep<M, K>,
		cutlass_baseline_gemm<M, K>>(buffer, current_index, floats_a, floats_b, outputs02, N);
	current_index = 0;

	bnch_swt::benchmark_stage<stage_name, total_iterations, measured_iterations>::template runBenchmarkWithPrep<"nvcuda_cublas_mul_mat_simple",
		cuda_mul_mat_01_prep_transposed<M, K>, cublas_mul_mat<M, K>>(buffer, current_index, floats_a, floats_b, outputs03, N);
	current_index = 0;

	bnch_swt::benchmark_stage<stage_name, total_iterations, measured_iterations>::printResults();
	compare_outputs<"cutlass_baseline_gemm Incorrect Value">(outputs01, outputs02);
	compare_outputs<"nvcuda_cublas_mul_mat_simple Incorrect Value">(outputs01, outputs03);
	compare_outputs<"nihilus_gemm Incorrect Value">(outputs01, outputs04);
};

int32_t main() {
	test_function_floats<4096, 4096, 4096, 1>();
	test_function_floats<4096, 4096, 4096, 2>();
	test_function_floats<4096, 4096, 4096, 4>();
	test_function_floats<4096, 4096, 4096, 8>();
	test_function_floats<4096, 4096, 4096, 16>();
	test_function_floats<4096, 4096, 4096, 32>();
	test_function_floats<4096, 4096, 4096, 64>();
	test_function_floats<4096, 4096, 4096, 128>();
	test_function_floats<4096, 4096, 4096, 256>();
	test_function_floats<4096, 4096, 4096, 512>();
	test_function_floats<4096, 4096, 4096, 1024>();
	test_function_floats<4096, 4096, 4096, 2048>();
	test_function_floats<4096, 4096, 4096, 4096>();
	test_function_floats<4096, 4096, 4096, 8192>();
	test_function_floats<4096, 4096, 4096, 16384>();
	test_function_floats<14336, 4096, 4096, 1>();
	test_function_floats<14336, 4096, 4096, 2>();
	test_function_floats<14336, 4096, 4096, 4>();
	test_function_floats<14336, 4096, 4096, 8>();
	test_function_floats<14336, 4096, 4096, 16>();
	test_function_floats<14336, 4096, 4096, 32>();
	test_function_floats<14336, 4096, 4096, 64>();
	test_function_floats<14336, 4096, 4096, 128>();
	test_function_floats<14336, 4096, 4096, 256>();
	test_function_floats<14336, 4096, 4096, 512>();
	test_function_floats<14336, 4096, 4096, 1024>();
	test_function_floats<14336, 4096, 4096, 2048>();
	test_function_floats<14336, 4096, 4096, 4096>();
	test_function_floats<14336, 4096, 4096, 8192>();
	test_function_floats<14336, 4096, 4096, 16384>();
	test_function<4096, 4096, 4096, 1>();
	test_function<4096, 4096, 4096, 2>();
	test_function<4096, 4096, 4096, 4>();
	test_function<4096, 4096, 4096, 8>();
	test_function<4096, 4096, 4096, 16>();
	test_function<4096, 4096, 4096, 32>();
	test_function<4096, 4096, 4096, 64>();
	test_function<4096, 4096, 4096, 128>();
	test_function<4096, 4096, 4096, 256>();
	test_function<4096, 4096, 4096, 512>();
	test_function<4096, 4096, 4096, 1024>();
	test_function<4096, 4096, 4096, 2048>();
	test_function<4096, 4096, 4096, 4096>();
	test_function<4096, 4096, 4096, 8192>();
	test_function<4096, 4096, 4096, 16384>();
	test_function<14336, 4096, 4096, 1>();
	test_function<14336, 4096, 4096, 2>();
	test_function<14336, 4096, 4096, 4>();
	test_function<14336, 4096, 4096, 8>();
	test_function<14336, 4096, 4096, 16>();
	test_function<14336, 4096, 4096, 32>();
	test_function<14336, 4096, 4096, 64>();
	test_function<14336, 4096, 4096, 128>();
	test_function<14336, 4096, 4096, 256>();
	test_function<14336, 4096, 4096, 512>();
	test_function<14336, 4096, 4096, 1024>();
	test_function<14336, 4096, 4096, 2048>();
	test_function<14336, 4096, 4096, 4096>();
	test_function<14336, 4096, 4096, 8192>();
	test_function<14336, 4096, 4096, 16384>();
	return 0;
}