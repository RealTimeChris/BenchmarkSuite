#include "hip/hip_runtime.h"
#include <BnchSwt/BenchmarkSuite.hpp>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

static constexpr uint64_t total_iterations{ 8 };
static constexpr uint64_t measured_iterations{ 1 };

template<auto multiple, typename value_01_type = decltype(multiple)> BNCH_SWT_INLINE constexpr value_01_type round_up_to_multiple(value_01_type value) noexcept {
	if constexpr ((multiple & (multiple - 1)) == 0) {
		constexpr value_01_type mulSub1{ multiple - 1 };
		constexpr value_01_type notMulSub1{ static_cast<value_01_type>(~mulSub1) };
		return (value + (mulSub1)) & notMulSub1;
	} else {
		const value_01_type remainder = value % multiple;
		return remainder == 0 ? value : value + (multiple - remainder);
	}
}

struct cuda_buffer {
	using value_type													= uint8_t;
	using pointer														= value_type*;
	using size_type														= uint64_t;
	BNCH_SWT_INLINE cuda_buffer() noexcept								= default;
	BNCH_SWT_INLINE cuda_buffer& operator=(const cuda_buffer&) noexcept = delete;
	BNCH_SWT_INLINE cuda_buffer(const cuda_buffer&) noexcept			= delete;

	BNCH_SWT_INLINE cuda_buffer& operator=(cuda_buffer&& other) noexcept {
		if (this != &other) {
			std::swap(data_val, other.data_val);
			std::swap(size_val, other.size_val);
		}
		return *this;
	}

	BNCH_SWT_INLINE cuda_buffer(cuda_buffer&& other) noexcept {
		*this = std::move(other);
	}

	BNCH_SWT_INLINE void init(uint64_t size) noexcept {
		if (data_val) {
			clear();
		}

		hipError_t result = hipMalloc(&data_val, size);
		if (result != hipSuccess) {
			data_val = nullptr;
			throw std::runtime_error{ "cuda_buffer - failed to allocate GPU memory" };
		}

		size_val = size;
	}

	BNCH_SWT_INLINE void deinit() noexcept {
		clear();
	}

	BNCH_SWT_INLINE uint64_t size() noexcept {
		return size_val;
	}

	BNCH_SWT_INLINE void* data() noexcept {
		return data_val;
	}

	BNCH_SWT_INLINE void* claim_memory(uint64_t offset_to_claim) noexcept {
		uint64_t aligned_amount = round_up_to_multiple<64>(offset_to_claim);
		if (aligned_amount > size_val) {
			throw std::runtime_error{ "cuda_buffer - not enough memory allocated!" };
		}
		pointer return_value = data_val + aligned_amount;
		return return_value;
	}

	BNCH_SWT_INLINE ~cuda_buffer() noexcept {
		clear();
	}

  protected:
	value_type* data_val{};
	uint64_t size_val{};

	BNCH_SWT_INLINE void clear() noexcept {
		if (data_val) {
			hipError_t result = hipFree(data_val);
			data_val		   = nullptr;
			size_val		   = 0;
		}
	}
};

using q8_quant = int8_t;

struct block_q8_0 {
	q8_quant quants[32]{};
	uint16_t scale{};
};

inline static uint16_t fp32_to_fp16(float f) {
	return static_cast<uint16_t>(_mm_extract_epi16(_mm_cvtps_ph(_mm_set_ss(f), _MM_FROUND_TO_NEAREST_INT), 0));
}

template<std::size_t count> inline std::vector<block_q8_0> generate_blocks() {
	std::vector<block_q8_0> result;
	result.reserve(count);

	for (std::size_t i = 0; i < count; ++i) {
		block_q8_0 block{};

		for (auto& q: block.quants) {
			q = static_cast<int8_t>((bnch_swt::random_generator::generateValue<uint8_t>() % 13) - 6);
		}

		float scale_float = (bnch_swt::random_generator::generateValue<float>() / std::numeric_limits<float>::max()) * 0.1f;

		block.scale = static_cast<uint16_t>(fp32_to_fp16(scale_float));

		result.emplace_back(block);
	}
	return result;
}

template<uint64_t iteration_count, std::size_t count> inline std::vector<std::vector<block_q8_0>> generate_blocks_final() {
	std::vector<std::vector<block_q8_0>> return_values{};
	for (uint64_t x = 0; x < iteration_count; ++x) {
		return_values.emplace_back(generate_blocks<count>());
	}
	return return_values;
}

template<std::size_t count> inline std::vector<float> generate_floats() {
	std::vector<float> result;
	result.reserve(count);

	for (std::size_t i = 0; i < count; ++i) {
		result.emplace_back(bnch_swt::random_generator::generateValue<float>());
	}
	return result;
}

template<uint64_t iteration_count, std::size_t count> inline std::vector<std::vector<float>> generate_floats_final() {
	std::vector<std::vector<float>> return_values{};
	for (uint64_t x = 0; x < iteration_count; ++x) {
		return_values.emplace_back(generate_floats<count>());
	}
	return return_values;
}

BNCH_SWT_INLINE static constexpr float fp32_from_bits(uint32_t w) noexcept {
	return std::bit_cast<float>(w);
}

BNCH_SWT_INLINE static constexpr uint32_t fp32_to_bits(float f) noexcept {
	return std::bit_cast<uint32_t>(f);
}

BNCH_SWT_INLINE static float compute_fp16_to_fp32(half h) noexcept {
	const uint32_t w	 = static_cast<uint32_t>(h) << 16;
	const uint32_t sign	 = w & 0x80000000u;
	const uint32_t two_w = w + w;

	constexpr uint32_t exp_offset = 0xE0u << 23;
	constexpr float exp_scale	  = fp32_from_bits(0x7800000u);
	const float normalized_value  = fp32_from_bits((two_w >> 4) + exp_offset) * exp_scale;

	constexpr uint32_t magic_mask  = 126u << 23;
	constexpr float magic_bias	   = 0.5f;
	const float denormalized_value = fp32_from_bits((two_w >> 17) | magic_mask) - magic_bias;

	constexpr uint32_t denormalized_cutoff = 1u << 27;
	const uint32_t result				   = sign | (two_w < denormalized_cutoff ? fp32_to_bits(denormalized_value) : fp32_to_bits(normalized_value));
	return fp32_from_bits(result);
}

alignas(64) static float* __restrict fp16_to_fp32_array{ []() {
	alignas(64) static std::array<float, (1 << 16)> return_values_new{};
	for (uint64_t i = 0; i < (1 << 16); ++i) {
		return_values_new[i] = float{ compute_fp16_to_fp32(static_cast<uint16_t>(i)) };
	}
	return return_values_new.data();
}() };

BNCH_SWT_INLINE static float fp16_to_fp32(uint16_t f) {
	return fp16_to_fp32_array[f];
}

template<uint64_t matA_rows, uint64_t matA_cols, uint64_t matB_cols, uint64_t block_size> struct reference_mul_mat {
	BNCH_SWT_INLINE static uint64_t impl(uint64_t& current_index, std::vector<std::vector<float>>& floats, std::vector<std::vector<block_q8_0>>& blocks,
		std::vector<std::vector<float>>& outputs) {
		const auto& current_blocks = blocks[current_index];
		const auto& current_floats = floats[current_index];
		auto& current_outputs	   = outputs[current_index];

		for (uint64_t row = 0; row < matA_rows; ++row) {
			for (uint64_t col = 0; col < matB_cols; ++col) {
				float sum = 0.0f;

				for (uint64_t k = 0; k < matA_cols; ++k) {
					const uint64_t block_idx	 = (row * matA_cols + k) / block_size;
					const uint64_t elem_in_block = (row * matA_cols + k) % block_size;

					const auto& block  = current_blocks[block_idx];
					const float scale  = __half2float(*reinterpret_cast<const __half*>(&block.scale));
					const float a_elem = scale * static_cast<float>(block.quants[elem_in_block]);

					const float b_elem = current_floats[k * matB_cols + col];

					sum += a_elem * b_elem;
				}

				current_outputs[row * matB_cols + col] = sum;
			}
		}

		return current_outputs.size() * sizeof(float);
	}
};

template<uint64_t matA_rows, uint64_t matA_cols, uint64_t matB_cols, uint64_t block_size> struct cuda_mul_mat_01_prep {
	BNCH_SWT_INLINE static uint64_t impl(cuda_buffer& buffer, uint64_t& current_index, std::vector<std::vector<float>>& floats, std::vector<std::vector<block_q8_0>>& blocks,
		std::vector<std::vector<float>>& outputs) {
		constexpr uint64_t total_blocks_A = ((matA_rows * matA_cols) + block_size - 1) / block_size;
		constexpr uint64_t blocks_size	  = total_blocks_A * sizeof(block_q8_0);
		constexpr uint64_t floats_B_size  = (matA_cols * matB_cols) * sizeof(float);
		constexpr uint64_t outputs_C_size = (matA_rows * matB_cols) * sizeof(float);
		uint64_t offset					  = 0;
		block_q8_0* d_blocks			  = reinterpret_cast<block_q8_0*>(static_cast<uint8_t*>(buffer.data()) + offset);
		offset							  = round_up_to_multiple<64>(offset + blocks_size);

		float* d_floats = reinterpret_cast<float*>(static_cast<uint8_t*>(buffer.data()) + offset);
		offset			= round_up_to_multiple<64>(offset + floats_B_size);

		float* d_outputs = reinterpret_cast<float*>(static_cast<uint8_t*>(buffer.data()) + offset);

		if (current_index > 0) {
			auto& previous_outputs = outputs[current_index - 1];
			hipError_t err		   = hipMemcpy(previous_outputs.data(), d_outputs, outputs_C_size, hipMemcpyDeviceToHost);
			if (err != hipSuccess) {
				std::cerr << "Failed to copy previous outputs from device: " + std::string(hipGetErrorString(err)) << std::endl;
			}
		}

		const auto& current_blocks = blocks[current_index];
		const auto& current_floats = floats[current_index];

		hipError_t err = hipMemcpy(d_blocks, current_blocks.data(), blocks_size, hipMemcpyHostToDevice);
		if (err != hipSuccess) {
			std::cerr << "Failed to copy blocks to device: " + std::string(hipGetErrorString(err)) << std::endl;
		}

		err = hipMemcpy(d_floats, current_floats.data(), floats_B_size, hipMemcpyHostToDevice);
		if (err != hipSuccess) {
			std::cerr << "Failed to copy floats to device: " + std::string(hipGetErrorString(err)) << std::endl;
		}

		err = hipMemset(d_outputs, 0, outputs_C_size);
		if (err != hipSuccess) {
			std::cerr << "Failed to zero output buffer: " + std::string(hipGetErrorString(err)) << std::endl;
		}

		return 0;
	}
};

template<uint64_t matA_rows, uint64_t matA_cols, uint64_t matB_cols, uint64_t block_size>
__global__ void ggml_cuda_mul_mat_kernel(const float* input01, const block_q8_0* input02, float* output) {
	const uint64_t row = blockIdx.y * blockDim.y + threadIdx.y;
	const uint64_t col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row >= matA_rows || col >= matB_cols)
		return;

	float sum = 0.0f;

	const uint64_t k_end = matA_cols & ~3;

	uint64_t k = 0;
	for (; k < k_end; k += 4) {
#pragma unroll
		for (int i = 0; i < 4; ++i) {
			const uint64_t k_idx		 = k + i;
			const uint64_t linear_idx	 = row * matA_cols + k_idx;
			const uint64_t block_idx	 = linear_idx / block_size;
			const uint64_t elem_in_block = linear_idx % block_size;

			const block_q8_0& block = input02[block_idx];
			const float scale		= __half2float(*reinterpret_cast<const __half*>(&block.scale));
			const float a_elem		= scale * static_cast<float>(block.quants[elem_in_block]);
			const float b_elem		= input01[k_idx * matB_cols + col];

			sum += a_elem * b_elem;
		}
	}

	for (; k < matA_cols; ++k) {
		const uint64_t linear_idx	 = row * matA_cols + k;
		const uint64_t block_idx	 = linear_idx / block_size;
		const uint64_t elem_in_block = linear_idx % block_size;

		const block_q8_0& block = input02[block_idx];
		const float scale		= __half2float(*reinterpret_cast<const __half*>(&block.scale));
		const float a_elem		= scale * static_cast<float>(block.quants[elem_in_block]);
		const float b_elem		= input01[k * matB_cols + col];

		sum += a_elem * b_elem;
	}

	output[row * matB_cols + col] = sum;
}

template<uint64_t matA_rows, uint64_t matA_cols, uint64_t matB_cols, uint64_t block_size> struct ggml_cuda_mul_mat {
	BNCH_SWT_INLINE static uint64_t impl(cuda_buffer& buffer, uint64_t& current_index, std::vector<std::vector<float>>& floats, std::vector<std::vector<block_q8_0>>& blocks,
		std::vector<std::vector<float>>& outputs) {
		auto& current_outputs = outputs[current_index];

		static constexpr uint64_t total_blocks_A = ((matA_rows * matA_cols) + block_size - 1) / block_size;
		static constexpr uint64_t blocks_size	 = total_blocks_A * sizeof(block_q8_0);
		static constexpr uint64_t floats_B_size	 = (matA_cols * matB_cols) * sizeof(float);
		static constexpr uint64_t outputs_C_size = (matA_rows * matB_cols) * sizeof(float);

		uint64_t offset			   = 0;
		const block_q8_0* d_blocks = reinterpret_cast<const block_q8_0*>(static_cast<uint8_t*>(buffer.data()) + offset);
		offset					   = round_up_to_multiple<64>(offset + blocks_size);

		const float* d_floats = reinterpret_cast<const float*>(static_cast<uint8_t*>(buffer.data()) + offset);
		offset				  = round_up_to_multiple<64>(offset + floats_B_size);

		float* d_outputs = reinterpret_cast<float*>(static_cast<uint8_t*>(buffer.data()) + offset);

		uint64_t block_dim_x, block_dim_y;
		if (matB_cols <= 4) {
			block_dim_x = matB_cols;
			block_dim_y = 256 / block_dim_x;
		} else if (matA_rows <= 16) {
			block_dim_x = 32;
			block_dim_y = 16;
		} else {
			block_dim_x = 16;
			block_dim_y = 32;
		}

		block_dim_x = std::min(block_dim_x, matB_cols);
		block_dim_y = std::min(block_dim_y, matA_rows);

		const uint64_t grid_dim_x = (matB_cols + block_dim_x - 1) / block_dim_x;
		const uint64_t grid_dim_y = (matA_rows + block_dim_y - 1) / block_dim_y;

		dim3 blockDim(static_cast<unsigned int>(block_dim_x), static_cast<unsigned int>(block_dim_y));
		dim3 gridDim(static_cast<unsigned int>(grid_dim_x), static_cast<unsigned int>(grid_dim_y));

		ggml_cuda_mul_mat_kernel<matA_rows, matA_cols, matB_cols, block_size><<<gridDim, blockDim>>>(d_floats, d_blocks, d_outputs);

		hipError_t err = hipGetLastError();
		if (err != hipSuccess) {
			std::cerr << "GGML CUDA kernel launch failed: " + std::string(hipGetErrorString(err)) << std::endl;
		}

		err = hipDeviceSynchronize();
		if (err != hipSuccess) {
			std::cerr << "GGML CUDA kernel execution failed: " + std::string(hipGetErrorString(err)) << std::endl;
		}

		++current_index;
		return current_outputs.size() * sizeof(float);
	}
};

enum class kernel_type_profiles : uint8_t {
	fp16_mha,
	fp16_moe,
	bf16_mha,
	bf16_gqa,
	q4_mha,
	q4_gqa,
	q4_moe,
	q8_mha,
	q8_gqa,
	q8_moe,
	mixed_fp16_fp32,
	mixed_bf16_fp32,
	count,
};

enum class mul_mat_types {
	q,
	k,
	v,
	kq,
	kqv,
	kqv_out,
	ffn_gate,
	ffn_up,
	ffn_out,
};

template<mul_mat_types core_type, kernel_type_profiles kernel_type_profile> struct mul_mat_params;

template<> struct mul_mat_params<mul_mat_types::q, kernel_type_profiles::q8_gqa> {
	static constexpr uint64_t block_size{ 32 };
	static constexpr uint64_t tile_size{ 32 };
};

template<> struct mul_mat_params<mul_mat_types::ffn_up, kernel_type_profiles::q8_gqa> {
	static constexpr uint64_t block_size{ 32 };
	static constexpr uint64_t tile_size{ 16 };
};

struct gpu_properties {
	static constexpr uint64_t sm_count{ 70ull };
	static constexpr uint64_t max_threads_per_sm{ 1536ull };
	static constexpr uint64_t max_threads_per_block{ 1024ull };
	static constexpr uint64_t warp_size{ 32ull };
	static constexpr uint64_t l2_cache_size{ 50331648ull };
	static constexpr uint64_t shared_mem_per_block{ 49152ull };
	static constexpr uint64_t memory_bus_width{ 256ull };
	static constexpr uint64_t memory_clock_rate{ 14001000ull };
	static constexpr uint64_t major_compute_capability{ 12ull };
	static constexpr uint64_t minor_compute_capability{ 0ull };
	static constexpr uint64_t max_grid_size_x{ 2147483647ull };
	static constexpr uint64_t gpu_arch_index{ 4ull };
	static constexpr uint64_t total_threads{ 107520ull };
	static constexpr uint64_t optimal_block_size{ 512ull };
	static constexpr uint64_t optimal_grid_size{ 210ull };
};

struct cuda_launch_params {
	uint64_t block_chunk_size;
	uint64_t grid_chunk_size;
	uint64_t blocks_per_grid;
	uint64_t threads_per_block;
	uint64_t warp_aligned_size;
};

template<typename output_type> BNCH_SWT_INLINE static constexpr cuda_launch_params calculate_gpu_launch_params(uint64_t total_required_bytes) {
	cuda_launch_params params;

	bool fits_in_l2 = total_required_bytes <= static_cast<uint64_t>(static_cast<float>(gpu_properties::l2_cache_size) * 0.6f);

	bool fits_in_shared = total_required_bytes <= static_cast<uint64_t>(static_cast<float>(gpu_properties::shared_mem_per_block) * 0.8f);

	if (fits_in_l2) {
		params.blocks_per_grid	 = gpu_properties::optimal_grid_size;
		params.threads_per_block = gpu_properties::optimal_block_size;
		params.grid_chunk_size	 = std::numeric_limits<uint64_t>::max();

		if (fits_in_shared) {
			params.block_chunk_size = gpu_properties::optimal_block_size * gpu_properties::warp_size;
		} else {
			params.block_chunk_size = gpu_properties::optimal_block_size;
		}

	} else {
		const uint64_t usable_l2 = static_cast<uint64_t>(static_cast<float>(gpu_properties::l2_cache_size) * 0.5f);

		const uint64_t chunks_needed = (total_required_bytes + usable_l2 - 1) / usable_l2;

		params.blocks_per_grid = std::min(gpu_properties::optimal_grid_size, gpu_properties::total_threads / gpu_properties::optimal_block_size / chunks_needed);

		params.threads_per_block = gpu_properties::optimal_block_size;
		params.grid_chunk_size	 = usable_l2;
		params.block_chunk_size	 = usable_l2 / params.blocks_per_grid;
	}
	params.warp_aligned_size = ((params.block_chunk_size + gpu_properties::warp_size - 1) / gpu_properties::warp_size) * gpu_properties::warp_size;

	return params;
}

template<uint64_t matA_rows, uint64_t matA_cols, uint64_t matB_cols, uint64_t block_size>
__global__ void cuda_mul_mat_kernel(const float* input01, const block_q8_0* input02, float* output) {
	const uint64_t row = blockIdx.y * blockDim.y + threadIdx.y;
	const uint64_t col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row >= matA_rows || col >= matB_cols)
		return;

	float sum = 0.0f;

	for (uint64_t k = 0; k < matA_cols; ++k) {
		const uint64_t linear_idx	 = row * matA_cols + k;
		const uint64_t block_idx	 = linear_idx / block_size;
		const uint64_t elem_in_block = linear_idx % block_size;

		const block_q8_0& block = input02[block_idx];
		const float scale		= __half2float(*reinterpret_cast<const __half*>(&block.scale));
		const float a_elem		= scale * static_cast<float>(block.quants[elem_in_block]);

		const float b_elem = input01[k * matB_cols + col];

		sum += a_elem * b_elem;
	}

	output[row * matB_cols + col] = sum;
}

template<uint64_t matA_rows, uint64_t matA_cols, uint64_t matB_cols, uint64_t block_size> struct cuda_mul_mat_01 {
	BNCH_SWT_INLINE static uint64_t impl(cuda_buffer& buffer, uint64_t& current_index, std::vector<std::vector<float>>& floats, std::vector<std::vector<block_q8_0>>& blocks,
		std::vector<std::vector<float>>& outputs) {
		auto& current_outputs = outputs[current_index];

		static constexpr uint64_t total_blocks_A = ((matA_rows * matA_cols) + block_size - 1) / block_size;
		static constexpr uint64_t blocks_size	 = total_blocks_A * sizeof(block_q8_0);
		static constexpr uint64_t floats_B_size	 = (matA_cols * matB_cols) * sizeof(float);
		static constexpr uint64_t outputs_C_size = (matA_rows * matB_cols) * sizeof(float);

		uint64_t offset			   = 0;
		const block_q8_0* d_blocks = reinterpret_cast<const block_q8_0*>(static_cast<uint8_t*>(buffer.data()) + offset);
		offset					   = round_up_to_multiple<64>(offset + blocks_size);

		const float* d_floats = reinterpret_cast<const float*>(static_cast<uint8_t*>(buffer.data()) + offset);
		offset				  = round_up_to_multiple<64>(offset + floats_B_size);

		float* d_outputs = reinterpret_cast<float*>(static_cast<uint8_t*>(buffer.data()) + offset);

		static constexpr uint64_t total_operation_bytes = outputs_C_size + blocks_size + floats_B_size;

		static constexpr auto launch_params = calculate_gpu_launch_params<float>(total_operation_bytes);

		const uint64_t threads_per_block = launch_params.threads_per_block;

		uint64_t block_dim_x, block_dim_y;

		if (threads_per_block >= 256) {
			block_dim_x = 16;
			block_dim_y = threads_per_block / block_dim_x;
		} else if (threads_per_block >= 128) {
			block_dim_x = 16;
			block_dim_y = threads_per_block / block_dim_x;
		} else {
			block_dim_x = 8;
			block_dim_y = threads_per_block / block_dim_x;
		}
		block_dim_x = std::min(block_dim_x, matB_cols);
		block_dim_y = std::min(block_dim_y, matA_rows);

		const uint64_t grid_dim_x = (matB_cols + block_dim_x - 1) / block_dim_x;
		const uint64_t grid_dim_y = (matA_rows + block_dim_y - 1) / block_dim_y;

		const uint64_t total_blocks = grid_dim_x * grid_dim_y;

		if (total_blocks > launch_params.blocks_per_grid) {
			if (threads_per_block < gpu_properties::max_threads_per_block) {
				const uint64_t scale_factor			 = (total_blocks + launch_params.blocks_per_grid - 1) / launch_params.blocks_per_grid;
				const uint64_t new_threads_per_block = std::min(threads_per_block * scale_factor, gpu_properties::max_threads_per_block);

				if (new_threads_per_block >= 256) {
					block_dim_x = 16;
					block_dim_y = new_threads_per_block / block_dim_x;
				} else {
					block_dim_x = 8;
					block_dim_y = new_threads_per_block / block_dim_x;
				}

				block_dim_x = std::min(block_dim_x, matB_cols);
				block_dim_y = std::min(block_dim_y, matA_rows);
			}
		}

		dim3 blockDim(static_cast<unsigned int>(block_dim_x), static_cast<unsigned int>(block_dim_y));
		dim3 gridDim(static_cast<unsigned int>(grid_dim_x), static_cast<unsigned int>(grid_dim_y));

		cuda_mul_mat_kernel<matA_rows, matA_cols, matB_cols, block_size><<<gridDim, blockDim>>>(d_floats, d_blocks, d_outputs);

		hipError_t err = hipGetLastError();
		if (err != hipSuccess) {
			std::cerr << "CUDA kernel launch failed: " + std::string(hipGetErrorString(err)) << std::endl;
		}

		//err = hipDeviceSynchronize();
		if (err != hipSuccess) {
			std::cerr << "CUDA kernel execution failed: " + std::string(hipGetErrorString(err)) << std::endl;
		}

		return current_outputs.size() * sizeof(float);
	}
};

template<uint64_t matA_rows, uint64_t matA_cols, uint64_t matB_cols, uint64_t block_size, mul_mat_types mul_mat_type>
__global__ void cuda_mul_mat_kernel_optimized(const float* input01, const block_q8_0* input02, float* output) {
	__shared__ float tile_A[mul_mat_params<mul_mat_type, kernel_type_profiles::q8_gqa>::tile_size][mul_mat_params<mul_mat_type, kernel_type_profiles::q8_gqa>::tile_size];
	__shared__ float tile_B[mul_mat_params<mul_mat_type, kernel_type_profiles::q8_gqa>::tile_size][mul_mat_params<mul_mat_type, kernel_type_profiles::q8_gqa>::tile_size];

	const uint64_t row = blockIdx.y * mul_mat_params<mul_mat_type, kernel_type_profiles::q8_gqa>::tile_size + threadIdx.y;
	const uint64_t col = blockIdx.x * mul_mat_params<mul_mat_type, kernel_type_profiles::q8_gqa>::tile_size + threadIdx.x;

	float sum = 0.0f;

	for (uint64_t tile = 0;
		tile < (matA_cols + mul_mat_params<mul_mat_type, kernel_type_profiles::q8_gqa>::tile_size - 1) / mul_mat_params<mul_mat_type, kernel_type_profiles::q8_gqa>::tile_size;
		++tile) {
		const uint64_t a_col = tile * mul_mat_params<mul_mat_type, kernel_type_profiles::q8_gqa>::tile_size + threadIdx.x;
		const uint64_t b_row = tile * mul_mat_params<mul_mat_type, kernel_type_profiles::q8_gqa>::tile_size + threadIdx.y;

		if (row < matA_rows && a_col < matA_cols) {
			const uint64_t linear_idx	 = row * matA_cols + a_col;
			const uint64_t block_idx	 = linear_idx / block_size;
			const uint64_t elem_in_block = linear_idx % block_size;

			const block_q8_0& block			 = input02[block_idx];
			const float scale				 = __half2float(*reinterpret_cast<const __half*>(&block.scale));
			tile_A[threadIdx.y][threadIdx.x] = scale * static_cast<float>(block.quants[elem_in_block]);
		} else {
			tile_A[threadIdx.y][threadIdx.x] = 0.0f;
		}

		if (b_row < matA_cols && col < matB_cols) {
			tile_B[threadIdx.y][threadIdx.x] = input01[b_row * matB_cols + col];
		} else {
			tile_B[threadIdx.y][threadIdx.x] = 0.0f;
		}

		__syncthreads();

#pragma unroll
		for (uint64_t k = 0; k < mul_mat_params<mul_mat_type, kernel_type_profiles::q8_gqa>::tile_size; ++k) {
			sum += tile_A[threadIdx.y][k] * tile_B[k][threadIdx.x];
		}

		__syncthreads();
	}

	if (row < matA_rows && col < matB_cols) {
		output[row * matB_cols + col] = sum;
	}
}

template<uint64_t matA_rows, uint64_t matA_cols, uint64_t matB_cols, uint64_t block_size, mul_mat_types mul_mat_type> struct nihilus_cuda_mul_mat {
	BNCH_SWT_INLINE static uint64_t impl(cuda_buffer& buffer, uint64_t& current_index, std::vector<std::vector<float>>& floats, std::vector<std::vector<block_q8_0>>& blocks,
		std::vector<std::vector<float>>& outputs) {
		auto& current_outputs = outputs[current_index];

		static constexpr uint64_t total_blocks_A = ((matA_rows * matA_cols) + block_size - 1) / block_size;
		static constexpr uint64_t blocks_size	 = total_blocks_A * sizeof(block_q8_0);
		static constexpr uint64_t floats_B_size	 = (matA_cols * matB_cols) * sizeof(float);
		static constexpr uint64_t outputs_C_size = (matA_rows * matB_cols) * sizeof(float);

		uint64_t offset			   = 0;
		const block_q8_0* d_blocks = reinterpret_cast<const block_q8_0*>(static_cast<uint8_t*>(buffer.data()) + offset);
		offset					   = round_up_to_multiple<64>(offset + blocks_size);

		const float* d_floats = reinterpret_cast<const float*>(static_cast<uint8_t*>(buffer.data()) + offset);
		offset				  = round_up_to_multiple<64>(offset + floats_B_size);

		float* d_outputs = reinterpret_cast<float*>(static_cast<uint8_t*>(buffer.data()) + offset);
		if constexpr (matB_cols <= 4) {
			cuda_mul_mat_01<matA_rows, matA_cols, matB_cols, block_size>::impl(buffer, current_index, floats, blocks, outputs);
		} else {
			const dim3 blockDim(mul_mat_params<mul_mat_type, kernel_type_profiles::q8_gqa>::tile_size, mul_mat_params<mul_mat_type, kernel_type_profiles::q8_gqa>::tile_size);
			const dim3 gridDim((matB_cols + mul_mat_params<mul_mat_type, kernel_type_profiles::q8_gqa>::tile_size - 1) /
					mul_mat_params<mul_mat_type, kernel_type_profiles::q8_gqa>::tile_size,
				(matA_rows + mul_mat_params<mul_mat_type, kernel_type_profiles::q8_gqa>::tile_size - 1) / mul_mat_params<mul_mat_type, kernel_type_profiles::q8_gqa>::tile_size);
			cuda_mul_mat_kernel_optimized<matA_rows, matA_cols, matB_cols, block_size, mul_mat_type><<<gridDim, blockDim>>>(d_floats, d_blocks, d_outputs);
		}

		hipError_t err = hipGetLastError();
		if (err != hipSuccess) {
			std::cerr << "CUDA kernel launch failed: " + std::string(hipGetErrorString(err)) << std::endl;
		}

		err = hipDeviceSynchronize();
		if (err != hipSuccess) {
			std::cerr << "CUDA kernel execution failed: " + std::string(hipGetErrorString(err)) << std::endl;
		}

		++current_index;
		return current_outputs.size() * sizeof(float);
	}
};

template<bnch_swt::string_literal rhs> inline void compare_outputs(const std::vector<std::vector<float>>& outputs01, const std::vector<std::vector<float>>& outputs02,
	uint64_t total_iterations, uint64_t matC_total_elems) {
	static constexpr float relative_tolerance = 3e-2f;
	static constexpr float absolute_tolerance = 1e-30f;
	for (uint64_t x = 0; x < total_iterations; ++x) {
		for (uint64_t y = 0; y < matC_total_elems; ++y) {
			const float val1 = outputs01[x][y];
			const float val2 = outputs02[x][y];

			if (std::isnan(val1) && std::isnan(val2)) {
				continue;
			}
			if (std::isnan(val1) || std::isnan(val2)) {
				std::cerr << rhs.operator std::string_view() << ": NaN mismatch at [" << x << "," << y << "]: " << val1 << " vs " << val2 << std::endl;
				break;
			}
			if (std::isinf(val1) && std::isinf(val2)) {
				if ((val1 > 0) == (val2 > 0)) {
					continue;
				} else {
					std::cerr << rhs.operator std::string_view() << ": Infinity sign mismatch at [" << x << "," << y << "]: " << val1 << " vs " << val2 << std::endl;
					break;
				}
			}
			if (std::isinf(val1) || std::isinf(val2)) {
				std::cerr << rhs.operator std::string_view() << ": Infinity mismatch at [" << x << "," << y << "]: " << val1 << " vs " << val2 << std::endl;
				break;
			}

			const float abs_diff = std::abs(val1 - val2);
			const float max_val	 = std::max(std::abs(val1), std::abs(val2));

			const bool values_match = (abs_diff <= absolute_tolerance) || (abs_diff <= relative_tolerance * max_val);

			if (!values_match) {
				std::cerr << rhs.operator std::string_view() << ": Value mismatch at [" << x << "," << y << "]: " << val1 << " vs " << val2 << std::endl;
				std::cerr << rhs.operator std::string_view() << ": Relative difference: " << (abs_diff / max_val) * 100.0f << "%" << std::endl;
				break;
			}
		}
	}
}

template<uint64_t matA_dim_00, uint64_t matA_dim_01, uint64_t matB_dim_00, uint64_t matB_dim_01, mul_mat_types mul_mat_type> BNCH_SWT_INLINE void test_function() {
	static constexpr uint64_t matC_dim_00{ matA_dim_00 };
	static constexpr uint64_t matC_dim_01{ matB_dim_01 };
	static constexpr uint64_t matC_total_elems{ matC_dim_00 * matC_dim_01 };
	static constexpr uint64_t total_blocks_a{ static_cast<uint64_t>(static_cast<float>(matA_dim_00 * matA_dim_01) * static_cast<float>(sizeof(block_q8_0)) /
												  static_cast<float>(std::size(block_q8_0{}.quants))) /
		sizeof(block_q8_0) };
	static constexpr uint64_t total_floats_b{ matB_dim_00 * matB_dim_01 };
	auto blocks = generate_blocks_final<total_iterations, total_blocks_a>();
	auto floats = generate_floats_final<total_iterations, total_floats_b>();
	std::vector<std::vector<float>> outputs01{};
	std::vector<std::vector<float>> outputs02{};
	outputs01.resize(total_iterations);
	outputs02.resize(total_iterations);
	for (uint64_t x = 0; x < total_iterations; ++x) {
		outputs01[x].resize(matC_total_elems);
		outputs02[x].resize(matC_total_elems);
	}

	static constexpr bnch_swt::string_literal stage_name{ "mul_mat: [" + bnch_swt::internal::toStringLiteral<matA_dim_00>() + "x" +
		bnch_swt::internal::toStringLiteral<matA_dim_01>() + " * " + bnch_swt::internal::toStringLiteral<matB_dim_00>() + "x" + bnch_swt::internal::toStringLiteral<matB_dim_01>() +
		"]" };
	static constexpr uint64_t total_elements_A = matA_dim_00 * matA_dim_01;
	static constexpr uint64_t total_blocks_A   = (total_elements_A + 32 - 1) / 32;
	static constexpr uint64_t blocks_size	   = total_blocks_A * sizeof(block_q8_0);
	static constexpr uint64_t floats_B_count   = matB_dim_00 * matB_dim_01;
	static constexpr uint64_t floats_B_size	   = floats_B_count * sizeof(float);
	static constexpr uint64_t outputs_C_count  = matA_dim_00 * matB_dim_01;
	static constexpr uint64_t outputs_C_size   = outputs_C_count * sizeof(float);

	uint64_t total_buffer_size = 0;
	total_buffer_size += round_up_to_multiple<64>(blocks_size);
	total_buffer_size += round_up_to_multiple<64>(floats_B_size);
	total_buffer_size += round_up_to_multiple<64>(outputs_C_size);

	cuda_buffer buffer{};
	buffer.init(total_buffer_size);

	uint64_t current_index{};
	bnch_swt::benchmark_stage<stage_name, total_iterations, measured_iterations>::template runBenchmarkWithPrep<"ggml_cuda_mul_mat",
		cuda_mul_mat_01_prep<matA_dim_00, matA_dim_01, matB_dim_01, 32>, ggml_cuda_mul_mat<matA_dim_00, matA_dim_01, matB_dim_01, 32>>(buffer, current_index, floats, blocks,
		outputs01);
	current_index = 0;
	bnch_swt::benchmark_stage<stage_name, total_iterations, measured_iterations>::template runBenchmarkWithPrep<"nihilus_cuda_mul_mat",
		cuda_mul_mat_01_prep<matA_dim_00, matA_dim_01, matB_dim_01, 32>, nihilus_cuda_mul_mat<matA_dim_00, matA_dim_01, matB_dim_01, 32, mul_mat_type>>(buffer, current_index,
		floats, blocks, outputs02);

	bnch_swt::benchmark_stage<stage_name, total_iterations, measured_iterations>::printResults();
	compare_outputs<"outputs02">(outputs01, outputs02, total_iterations, matC_total_elems);
};

int main() {
	test_function<14336, 4096, 4096, 1, mul_mat_types::ffn_up>();
	test_function<14336, 4096, 4096, 2, mul_mat_types::ffn_up>();
	test_function<14336, 4096, 4096, 4, mul_mat_types::ffn_up>();
	test_function<14336, 4096, 4096, 8, mul_mat_types::ffn_up>();
	test_function<14336, 4096, 4096, 16, mul_mat_types::ffn_up>();
	test_function<14336, 4096, 4096, 32, mul_mat_types::ffn_up>();
	test_function<14336, 4096, 4096, 64, mul_mat_types::ffn_up>();
	test_function<14336, 4096, 4096, 128, mul_mat_types::ffn_up>();
	test_function<14336, 4096, 4096, 256, mul_mat_types::ffn_up>();
	test_function<14336, 4096, 4096, 512, mul_mat_types::ffn_up>();
	test_function<14336, 4096, 4096, 1024, mul_mat_types::ffn_up>();
	test_function<14336, 4096, 4096, 2048, mul_mat_types::ffn_up>();
	test_function<14336, 4096, 4096, 4096, mul_mat_types::ffn_up>();
	test_function<4096, 4096, 4096, 1, mul_mat_types::q>();
	test_function<4096, 4096, 4096, 2, mul_mat_types::q>();
	test_function<4096, 4096, 4096, 4, mul_mat_types::q>();
	test_function<4096, 4096, 4096, 8, mul_mat_types::q>();
	test_function<4096, 4096, 4096, 16, mul_mat_types::q>();
	test_function<4096, 4096, 4096, 32, mul_mat_types::q>();
	test_function<4096, 4096, 4096, 64, mul_mat_types::q>();
	test_function<4096, 4096, 4096, 128, mul_mat_types::q>();
	test_function<4096, 4096, 4096, 256, mul_mat_types::q>();
	test_function<4096, 4096, 4096, 512, mul_mat_types::q>();
	test_function<4096, 4096, 4096, 1024, mul_mat_types::q>();
	test_function<4096, 4096, 4096, 2048, mul_mat_types::q>();
	test_function<4096, 4096, 4096, 4096, mul_mat_types::q>();
	return 0;
}
